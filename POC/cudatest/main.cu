#include "hip/hip_runtime.h"
#include <cstdio>
#include "impl_cuda.h"

using namespace fdb;

__global__ void a() { printf("a\n"); }

int main() {
    parallelFor(vec3S(1, 1, 1), vec3S(1, 1, 4), [=] FDB_DEVICE (vec3S block_idx, vec3S thread_idx) {
        printf("hello, world! %d\n", thread_idx[2]);
    });

    checkCudaErrors(hipDeviceSynchronize());
    return 0;
}
