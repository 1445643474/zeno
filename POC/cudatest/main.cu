#include "hip/hip_runtime.h"
#include <cstdio>
#include "impl_host.h"
#include "Vector.h"

using namespace fdb;

__global__ void a() { printf("a\n"); }

int main() {
    Vector<int> a;
    a.resize(5, 40);
    auto av = a.view();
    parallelFor(a.size(), [=] FDB_DEVICE (size_t i) {
        printf("- %ld %d\n", i, av[i]);
        av[i] = 42;
    });
    a.resize(8, 4);
    parallelFor(a.size(), [=] FDB_DEVICE (size_t i) {
        printf("+ %ld %d\n", i, av[i]);
    });
    synchronize();
    return 0;
}
