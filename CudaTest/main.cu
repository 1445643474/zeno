#include "hip/hip_runtime.h"
#if 0
#include <stdio.h>
__global__ void test() { printf("FuCK U NVIDIA!\n"); } int main(void) { test<<<1, 1>>>(); hipDeviceSynchronize(); }
#else

#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
#include "ndarray.cuh"
#include <cassert>
#include <cstdio>
#include <cmath>

int main(void)
{
  NDArray arr({16}, {sizeof(int)});

  for (ssize_t ix = 0; ix < 16; ix++) {
    *(int *)arr({ix}) = 3;
  }

  blur<<<4, 4>>>(arr);
  checkCudaErrors(hipDeviceSynchronize());
  for (ssize_t ix = 0; ix < 16; ix++) {
    printf("%d\n", *(int *)arr({ix}));
  }

  return 0;
}
#endif
