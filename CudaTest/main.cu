#include "hip/hip_runtime.h"
#if 0
#include <stdio.h>
__global__ void test() { printf("FuCK U NVIDIA!\n"); } int main(void) { test<<<1, 1>>>(); hipDeviceSynchronize(); }
#else

#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
#include "ndarray.cuh"
#include <cassert>
#include <cstdio>
#include <cmath>

__global__ void blur(NDTypedView<int> arr)
{
  size_t ix = blockIdx.x * blockDim.x + threadIdx.x;
  size_t iy = blockIdx.y * blockDim.y + threadIdx.y;

  arr({ix, iy}) = ix + 1;

  __shared__ char tmpData[16 * sizeof(int)];
  NDTypedView<int> tmp({16}, {sizeof(int)}, tmpData);
}

int main(void)
{
  size_t nx = 32, ny = 32;
  NDArray arr({nx, ny}, {sizeof(int), sizeof(int) * nx});

  for (size_t iy = 0; iy < ny; iy++) {
    for (size_t ix = 0; ix < nx; ix++) {
      *(int *)arr({ix, iy}) = 233;
    }
  }

  dim3 griddim{(unsigned)nx / 8, (unsigned)ny / 8, 1};
  dim3 blockdim{8, 8, 1};
  blur<<<griddim, blockdim>>>(arr);
  checkCudaErrors(hipDeviceSynchronize());

  for (size_t iy = 0; iy < ny; iy++) {
    for (size_t ix = 0; ix < nx; ix++) {
      printf("%d\n", *(int *)arr({ix, iy}));
    }
  }

  return 0;
}
#endif
