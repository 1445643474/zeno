#include "hip/hip_runtime.h"
#if 0
#include <stdio.h>
__global__ void test() { printf("FuCK U NVIDIA!\n"); } int main(void) { test<<<1, 1>>>(); hipDeviceSynchronize(); }
#else

#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
#include "managed.cuh"
#include <cassert>
#include <cstdio>
#include <cmath>

const size_t Nx = 128;

template <class T>
__global__ void blur(T *arr)
{
  size_t ix = blockIdx.x * blockDim.x + threadIdx.x;
  if (ix < Nx) {
    *Subscriptor(*arr, ix).get() = ix + 1;
  }
}

int main(void)
{
  Field<Dense<Pointer<Dense<Place<int>, 4, 0>>, Nx / 4, 0>> arr;

  for (size_t ix = 0; ix < Nx; ix++) {
    arr.subscript(ix).activate();
    arr.subscript(ix).get() = 3;
  }
  blur<<<(Nx + 1023) / 1024, (Nx < 1024 ? Nx : 1024)>>>(arr);
  checkCudaErrors(hipDeviceSynchronize());
  for (size_t ix = 0; ix < Nx; ix++) {
    printf("%d\n", arr.subscript(ix).get());
  }

  delete arr;
  return 0;
}
#endif
