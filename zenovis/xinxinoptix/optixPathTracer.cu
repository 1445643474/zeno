#include "hip/hip_runtime.h"
#include <optix.h>
#include <cuda/random.h>
#include <sutil/vec_math.h>
#include <cuda/helpers.h>
#include "optixPathTracer.h"

extern "C" {
__constant__ Params params;
}



//------------------------------------------------------------------------------
//
//
//
//------------------------------------------------------------------------------

struct RadiancePRD
{
    // TODO: move some state directly into payload registers?
    float3       emitted;
    float3       radiance;
    float3       attenuation;
    float3       origin;
    float3       direction;
    float        opacity;
    unsigned int seed;
    unsigned int flags = 0;
    int          countEmitted;
    int          done;
    int          pad;
};


//------------------------------------------------------------------------------
//
//
//
//------------------------------------------------------------------------------


static __forceinline__ __device__ void traceRadiance(
        OptixTraversableHandle handle,
        float3                 ray_origin,
        float3                 ray_direction,
        float                  tmin,
        float                  tmax,
        RadiancePRD*           prd
        )
{
    // TODO: deduce stride from num ray-types passed in params

    unsigned int u0, u1;
    packPointer( prd, u0, u1 );
    optixTrace(
            handle,
            ray_origin,
            ray_direction,
            tmin,
            tmax,
            0.0f,                // rayTime
            OptixVisibilityMask( 1 ),
            OPTIX_RAY_FLAG_NONE,
            RAY_TYPE_RADIANCE,        // SBT offset
            RAY_TYPE_COUNT,           // SBT stride
            RAY_TYPE_RADIANCE,        // missSBTIndex
            u0, u1 );
}


static __forceinline__ __device__ bool traceOcclusion(
        OptixTraversableHandle handle,
        float3                 ray_origin,
        float3                 ray_direction,
        float                  tmin,
        float                  tmax
        )
{
    unsigned int occluded = 0u;
    optixTrace(
            handle,
            ray_origin,
            ray_direction,
            tmin,
            tmax,
            0.0f,                    // rayTime
            OptixVisibilityMask( 1 ),
            OPTIX_RAY_FLAG_NONE,
            RAY_TYPE_OCCLUSION,      // SBT offset
            RAY_TYPE_COUNT,          // SBT stride
            RAY_TYPE_OCCLUSION       // missSBTIndex
            );
        return occluded;//???
}


//------------------------------------------------------------------------------
//
//
//
//------------------------------------------------------------------------------

extern "C" __global__ void __raygen__rg()
{
    const int    w   = params.width;
    const int    h   = params.height;
    //const float3 eye = params.eye;
    const uint3  idx = optixGetLaunchIndex();
    const int    subframe_index = params.subframe_index;
    const CameraInfo cam = params.cam;

    unsigned int seed = tea<4>( idx.y*w + idx.x, subframe_index );

    float3 result = make_float3( 0.0f );
    int i = params.samples_per_launch;
    do
    {
        // The center of each pixel is at fraction (0.5,0.5)
        const float2 subpixel_jitter = make_float2( rnd( seed ), rnd( seed ) );

        const float2 d = 2.0f * make_float2(
                ( static_cast<float>( idx.x ) + subpixel_jitter.x ) / static_cast<float>( w ),
                ( static_cast<float>( idx.y ) + subpixel_jitter.y ) / static_cast<float>( h )
                ) - 1.0f;
        float3 ray_direction = normalize(cam.right * -d.x + cam.up * d.y + cam.front);
        float3 ray_origin    = cam.eye;

        RadiancePRD prd;
        prd.emitted      = make_float3(0.f);
        prd.radiance     = make_float3(0.f);
        prd.attenuation  = make_float3(1.f);
        prd.countEmitted = true;
        prd.done         = false;
        prd.seed         = seed;
        prd.opacity      = 0;
        int depth = 0;
        for( ;; )
        {
            traceRadiance(
                    params.handle,
                    ray_origin,
                    ray_direction,
                    0.01f,  // tmin       // TODO: smarter offset
                    1e16f,  // tmax
                    &prd );

            result += prd.emitted;
            result += prd.radiance * prd.attenuation;

            if( prd.done  || depth >= 3 ) // TODO RR, variable for depth
                break;

            ray_origin    = prd.origin;
            ray_direction = prd.direction;
            if(prd.opacity<0.99)
                ++depth;
        }
    }
    while( --i );

    const uint3    launch_index = optixGetLaunchIndex();
    const unsigned int image_index  = launch_index.y * params.width + launch_index.x;
    float3         accum_color  = result / static_cast<float>( params.samples_per_launch );

    if( subframe_index > 0 )
    {
        const float                 a = 1.0f / static_cast<float>( subframe_index+1 );
        const float3 accum_color_prev = make_float3( params.accum_buffer[ image_index ]);
        accum_color = lerp( accum_color_prev, accum_color, a );
    }
    params.accum_buffer[ image_index ] = make_float4( accum_color, 1.0f);
    params.frame_buffer[ image_index ] = make_color ( accum_color );
}
