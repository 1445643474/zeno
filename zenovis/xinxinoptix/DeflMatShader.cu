#include "hip/hip_runtime.h"
#include <optix.h>
#include <cuda/random.h>
#include <sutil/vec_math.h>
#include <cuda/helpers.h>
#include "optixPathTracer.h"
#include "TraceStuff.h"
#include "MaterialStuff.h"



/*
extern "C" __global__ void __closesthit__occlusion()
{
    setPayloadOcclusion( true );
}
extern "C" __global__ void __anyhit__shadow_cutout()
{
    HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();

    const int    prim_idx        = optixGetPrimitiveIndex();
    const vec3 ray_dir         = (optixGetWorldRayDirection());
    const int    vert_idx_offset = prim_idx*3;

    const vec3 v0   = make_float3( rt_data->vertices[ vert_idx_offset+0 ] );
    const vec3 v1   = make_float3( rt_data->vertices[ vert_idx_offset+1 ] );
    const vec3 v2   = make_float3( rt_data->vertices[ vert_idx_offset+2 ] );
    const vec3 N_0  = normalize( cross( v1-v0, v2-v0 ) );

    const vec3 N    = faceforward( N_0, -ray_dir, N_0 );
    const vec3 P    = vec3(optixGetWorldRayOrigin()) + optixGetRayTmax()*ray_dir;

    RadiancePRD* prd = getPRD();
    float opacity = 0.0;//sin(P.y)>0?1.0:0.0;
    prd->opacity = opacity;
    // Stochastic alpha test to get an alpha blend effect.
    if (opacity >0.99 ) // No need to calculate an expensive random number if the test is going to fail anyway.
    {
        optixIgnoreIntersection();
    }
    else
    {
        prd->flags |= 1;
        optixTerminateRay();
    }
}


extern "C" __global__ void __closesthit__radiance()
{
    HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();

    const int    prim_idx        = optixGetPrimitiveIndex();
    const vec3 ray_dir         = (optixGetWorldRayDirection());
    const int    vert_idx_offset = prim_idx*3;

    const vec3 v0   = make_float3( rt_data->vertices[ vert_idx_offset+0 ] );
    const vec3 v1   = make_float3( rt_data->vertices[ vert_idx_offset+1 ] );
    const vec3 v2   = make_float3( rt_data->vertices[ vert_idx_offset+2 ] );
    const vec3 N_0  = normalize( cross( v1-v0, v2-v0 ) );

    const vec3 N    = faceforward( N_0, -ray_dir, N_0 );
    const vec3 P    = vec3(optixGetWorldRayOrigin()) + optixGetRayTmax()*ray_dir;

    RadiancePRD* prd = getPRD();

    if( prd->countEmitted )
        prd->emitted = rt_data->emission_color;
    else
        prd->emitted = vec3( 0.0f );

    
    float opacity = 0.0;//sin(P.y)>0?1.0:0.0;
    prd->opacity = opacity;
    if(opacity>0.99)
    {
        prd->radiance += vec3(0.0f);
        prd->origin = P;
        prd->direction = ray_dir;
        return;
    }

    unsigned int seed = prd->seed;

    {
        const float z1 = rnd(seed);
        const float z2 = rnd(seed);

        vec3 w_in;
        cosine_sample_hemisphere( z1, z2, w_in );
        Onb onb( N );
        onb.inverse_transform( w_in );
        prd->direction = w_in;
        prd->origin    = P;

        prd->attenuation *= rt_data->diffuse_color;
        prd->countEmitted = false;
    }

    const float z1 = rnd(seed);
    const float z2 = rnd(seed);
    prd->seed = seed;

    ParallelogramLight light = params.light;
    const vec3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

    // Calculate properties of light sample (for area based pdf)
    const float  Ldist = length(light_pos - P );
    const vec3 L     = normalize(light_pos - P );
    const float  nDl   = dot( N, L );
    const float  LnDl  = -dot( vec3(light.normal), L );

    float weight = 0.0f;
    if( nDl > 0.0f && LnDl > 0.0f )
    {
        prd->flags = 0;
        traceOcclusion(
            params.handle,
            P,
            L,
            1e-5f,         // tmin
            Ldist - 1e-5f  // tmax
            );
        unsigned int occluded = prd->flags;
        if( !occluded )
        {
            const float A = length(cross(light.v1, light.v2));
            weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
        }
    }

    prd->radiance += light.emission * weight;
}
*/


extern "C" __global__ void __closesthit__occlusion()
{
    setPayloadOcclusion( true );
}
extern "C" __global__ void __anyhit__shadow_cutout()
{
    HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();

    const int    prim_idx        = optixGetPrimitiveIndex();
    const vec3 ray_dir         = optixGetWorldRayDirection();
    const int    vert_idx_offset = prim_idx*3;

    const vec3 v0   = make_float3( rt_data->vertices[ vert_idx_offset+0 ] );
    const vec3 v1   = make_float3( rt_data->vertices[ vert_idx_offset+1 ] );
    const vec3 v2   = make_float3( rt_data->vertices[ vert_idx_offset+2 ] );
    const vec3 N_0  = normalize( cross( v1-v0, v2-v0 ) );

    const vec3 N    = faceforward( N_0, -ray_dir, N_0 );
    const vec3 P    = vec3(optixGetWorldRayOrigin()) + optixGetRayTmax()*ray_dir;

    RadiancePRD* prd = getPRD();

    vec3  mat_baseColor = vec3(1.0,0.766,0.336);
    float mat_metallic = 1;
    float mat_roughness = 0.1;
    float mat_subsurface = 0.0;
    float mat_specular = 0;
    float mat_specularTint = 0.0;
    float mat_anisotropic = 0.0;
    float mat_sheen = 0.0;
    float mat_sheenTint = 0.0;
    float mat_clearCoat = 0.0;
    float mat_clearCoatGloss = 0.0;
    float mat_opacity = 0.0;
    vec3 attr_pos = vec3(P.x, P.y, P.z);
    vec3 attr_norm = vec3(0,0,1);
    vec3 attr_uv = vec3(0,0,0);//todo later
    vec3 attr_clr = vec3(rt_data->diffuse_color.x, rt_data->diffuse_color.y, rt_data->diffuse_color.z);
    vec3 attr_tang = vec3(0,0,0);
///////here injecting of material code in GLSL style///////////////////////////////


    float pnoise = perlin(1, 3, attr_pos*0.02);
    pnoise = clamp(pnoise, 0.0f, 1.0f);

    float pnoise2 = perlin(1, 4, attr_pos*0.02);
    mat_metallic = pnoise;

    mat_roughness = pnoise2;
    mat_roughness = clamp(mat_roughness, 0.01f,0.99f)*0.5f;

    float pnoise3 = perlin(10.0, 5, attr_pos*0.005);
    mat_opacity = clamp(pnoise3, 0.0f,1.0f);

////////////end of GLSL material code injection///////////////////////////////////////////////
    vec3 baseColor = mat_baseColor;
    float metallic = mat_metallic;;
    float roughness = mat_roughness;
    float subsurface = mat_subsurface;
    float specular = mat_specular;
    float specularTint = mat_specularTint;
    float anisotropic = mat_anisotropic;
    float sheen = mat_sheen;
    float sheenTint = mat_sheenTint;
    float clearCoat = mat_clearCoat;
    float clearCoatGloss = mat_clearCoatGloss;
    float opacity = mat_opacity;
    // Stochastic alpha test to get an alpha blend effect.
    if (opacity >0.99 ) // No need to calculate an expensive random number if the test is going to fail anyway.
    {
        optixIgnoreIntersection();
    }
    else
    {
        prd->flags |= 1;
        optixTerminateRay();
    }
}


extern "C" __global__ void __closesthit__radiance()
{
    HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();

    const int    prim_idx        = optixGetPrimitiveIndex();
    const float3 ray_dir         = optixGetWorldRayDirection();
    const int    vert_idx_offset = prim_idx*3;

    const float3 v0   = make_float3( rt_data->vertices[ vert_idx_offset+0 ] );
    const float3 v1   = make_float3( rt_data->vertices[ vert_idx_offset+1 ] );
    const float3 v2   = make_float3( rt_data->vertices[ vert_idx_offset+2 ] );
    const float3 N_0  = normalize( cross( v1-v0, v2-v0 ) );

    const float3 N    = faceforward( N_0, -ray_dir, N_0 );
    const float3 P    = optixGetWorldRayOrigin() + optixGetRayTmax()*ray_dir;

    RadiancePRD* prd = getPRD();

    if( prd->countEmitted )
        prd->emitted = rt_data->emission_color;
    else
        prd->emitted = make_float3( 0.0f );


    vec3  mat_baseColor = vec3(1.0,0.766,0.336);
    float mat_metallic = 1;
    float mat_roughness = 0.1;
    float mat_subsurface = 0.0;
    float mat_specular = 0;
    float mat_specularTint = 0.0;
    float mat_anisotropic = 0.0;
    float mat_sheen = 0.0;
    float mat_sheenTint = 0.0;
    float mat_clearCoat = 0.0;
    float mat_clearCoatGloss = 0.0;
    float mat_opacity = 0.0;
    vec3 attr_pos = vec3(P.x, P.y, P.z);
    vec3 attr_norm = vec3(0,0,1);
    vec3 attr_uv = vec3(0,0,0);//todo later
    vec3 attr_clr = vec3(rt_data->diffuse_color.x, rt_data->diffuse_color.y, rt_data->diffuse_color.z);
    vec3 attr_tang = vec3(0,0,0);
///////here injecting of material code in GLSL style///////////////////////////////


    float pnoise = perlin(1, 3, attr_pos*0.02);
    pnoise = clamp(pnoise, 0.0f, 1.0f);

    float pnoise2 = perlin(1, 4, attr_pos*0.02);
    mat_metallic = pnoise;

    mat_roughness = pnoise2;
    mat_roughness = clamp(mat_roughness, 0.01f,0.99f)*0.5f;

    float pnoise3 = perlin(10.0, 5, attr_pos*0.005);
    mat_opacity = clamp(pnoise3, 0.0f,1.0f);

////////////end of GLSL code injection///////////////////////////////////////////////
    vec3 baseColor = mat_baseColor;
    float metallic = mat_metallic;;
    float roughness = mat_roughness;
    float subsurface = mat_subsurface;
    float specular = mat_specular;
    float specularTint = mat_specularTint;
    float anisotropic = mat_anisotropic;
    float sheen = mat_sheen;
    float sheenTint = mat_sheenTint;
    float clearCoat = mat_clearCoat;
    float clearCoatGloss = mat_clearCoatGloss;
    float opacity = mat_opacity;
    //todo normal mapping TBN*N;



    //end of material computation
    metallic = clamp(metallic,0.01, 0.99);
    roughness = clamp(roughness, 0.01,0.99);
    //discard fully opacity pixels
    prd->opacity = opacity;
    if(opacity>0.99)
    {
        prd->radiance += make_float3(0.0f);
        prd->origin = P;
        prd->direction = ray_dir;
        return;
    }

    //{
    unsigned int seed = prd->seed;
    float is_refl;
    float3 wi = DisneyBRDF::sample_f(
                                seed,
                                baseColor,
                                metallic,
                                subsurface,
                                specular,
                                roughness,
                                specularTint,
                                anisotropic,
                                sheen,
                                sheenTint,
                                clearCoat,
                                clearCoatGloss,
                                N,
                                make_float3(0,0,0),
                                make_float3(0,0,0),
                                -normalize(ray_dir),
                                is_refl);

    float pdf = DisneyBRDF::pdf(baseColor,
                                metallic,
                                subsurface,
                                specular,
                                roughness,
                                specularTint,
                                anisotropic,
                                sheen,
                                sheenTint,
                                clearCoat,
                                clearCoatGloss,
                                N,
                                make_float3(0,0,0),
                                make_float3(0,0,0),
                                wi,
                                -normalize(ray_dir)
                                );
    float3 f = DisneyBRDF::eval(baseColor,
                                metallic,
                                subsurface,
                                specular,
                                roughness,
                                specularTint,
                                anisotropic,
                                sheen,
                                sheenTint,
                                clearCoat,
                                clearCoatGloss,
                                N,
                                make_float3(0,0,0),
                                make_float3(0,0,0),
                                wi,
                                -normalize(ray_dir)
                                );
    prd->prob2 = prd->prob;
    prd->prob *= pdf;
    prd->origin = P;
    prd->direction = wi;
    prd->countEmitted = false;
    if(is_refl)
        prd->attenuation *= f * clamp(dot(wi, N),0.0f,1.0f);
    else
        prd->attenuation *= f * clamp(dot(wi, N),0.0f,1.0f);
    //}

    // {
    //     const float z1 = rnd(seed);
    //     const float z2 = rnd(seed);

    //     float3 w_in;
    //     cosine_sample_hemisphere( z1, z2, w_in );
    //     Onb onb( N );
    //     onb.inverse_transform( w_in );
    //     prd->direction = w_in;
    //     prd->origin    = P;

    //     prd->attenuation *= rt_data->diffuse_color;
    //     prd->countEmitted = false;
    // }

    const float z1 = rnd(seed);
    const float z2 = rnd(seed);
    prd->seed = seed;

    ParallelogramLight light = params.light;
    const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

    // Calculate properties of light sample (for area based pdf)
    const float  Ldist = length(light_pos - P );
    const float3 L     = normalize(light_pos - P );
    const float  nDl   = dot( N, L );
    const float  LnDl  = -dot( light.normal, L );

    float weight = 0.0f;
    if( nDl > 0.0f && LnDl > 0.0f )
    {
        prd->flags = 0;
        traceOcclusion(
            params.handle,
            P,
            L,
            0.01f,         // tmin
            Ldist - 0.01f  // tmax
            );
        unsigned int occluded = prd->flags;
        if( !occluded )
        {
            const float A = length(cross(light.v1, light.v2));
            weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
        }
    }

    prd->radiance += light.emission * weight;
}
