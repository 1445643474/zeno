#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
#include <cassert>
#include <cstdio>
#include <cmath>


const float niu = 0.005f;
const float tau = 3.f * niu + 0.5f;
const float inv_tau = 1.f / tau;

__constant__ int directions[][3] = {{0,0,0},{1,0,0},{-1,0,0},{0,1,0},{0,-1,0},{0,0,1},{0,0,-1},{1,1,1},{-1,-1,-1},{1,1,-1},{-1,-1,1},{1,-1,1},{-1,1,-1},{-1,1,1},{1,-1,-1}};
__constant__ float weights[] = {2.f/9.f, 1.f/9.f, 1.f/9.f, 1.f/9.f, 1.f/9.f, 1.f/9.f, 1.f/9.f,1.f/72.f, 1.f/72.f, 1.f/72.f, 1.f/72.f, 1.f/72.f, 1.f/72.f, 1.f/72.f, 1.f/72.f};


#define N 8

template <class T>
struct volume {
    T *grid;

    void allocate() {
        size_t size = N * N * N;
        checkCudaErrors(hipMallocManaged(&grid, size * sizeof(T)));
    }

    void free() {
        checkCudaErrors(hipFree(grid));
    }

    __host__ __device__ T &at(int i, int j, int k) const {
        return grid[i + j * N + k * N * N];
    }
};

#define GSL(x, nx) \
    int x = blockDim.x * blockIdx.x + threadIdx.x; \
    x < nx; x += blockDim.x * gridDim.x


__device__ float f_eq(volume<float> rho,
    volume<float4> vel, int q, int x, int y, int z) {
    float m = rho.at(x, y, z);
    float4 v = vel.at(x, y, z);
    float eu = v.x * directions[q][0]
        + v.y * directions[q][1] + v.z * directions[q][2];
    float uv = v.x * v.x + v.y * v.y + v.z * v.z;
    float term = 1.f + 3.f * eu + 4.5f * eu * eu - 1.5f * uv;
    float feq = weights[q] * m * term;
    return feq;
}


__global__ void fill(volume<float> vol) {
    for (GSL(z, N)) {
        for (GSL(y, N)) {
            for (GSL(x, N)) {
                vol.at(x, y, z) = float(x) / N;
            }
        }
    }
}


int main(void)
{
    volume<float> vol;
    vol.allocate();

    fill<<<dim3(1, 1, 1), dim3(1, 1, 1)>>>(vol);

    checkCudaErrors(hipDeviceSynchronize());

    for (int i = 0; i < N; i++) {
        printf("%f\n", vol.at(i, 0, 0));
    }

    return 0;
}


/*
void displayFunc() {
    glClear(GL_COLOR_BUFFER_BIT);
    glDrawPixels(N, N, GL_RED, GL_FLOAT, pixels);
    glFlush();
}

void timerFunc(int unused) {
    stepFunc();
    renderFunc();
    glutPostRedisplay();
    glutTimerFunc(ITV, timerFunc, 0);
}

void keyboardFunc(unsigned char key, int x, int y) {
    if (key == 27)
        exit(0);
}

int main(int argc, char **argv) {
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DEPTH | GLUT_SINGLE | GLUT_RGBA);
    glutInitWindowPosition(100, 100);
    glutInitWindowSize(N, N);
    glutCreateWindow("GLUT Window");
    glutDisplayFunc(displayFunc);
    glutKeyboardFunc(keyboardFunc);
    initFunc();
    renderFunc();
    glutTimerFunc(ITV, timerFunc, 0);
    glutMainLoop();
}
*/
