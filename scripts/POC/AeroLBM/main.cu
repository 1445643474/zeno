#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
#include <cassert>
#include <cstdio>
#include <cmath>
#include <GL/glut.h>

#define N 64

template <class T>
struct volume {
    T *grid;

    void allocate() {
        size_t size = N * N * N;
        checkCudaErrors(hipMallocManaged(&grid, size * sizeof(T)));
    }

    void free() {
        checkCudaErrors(hipFree(grid));
    }

    __host__ __device__ T &at(int i, int j, int k) const {
        return grid[i + j * N + k * N * N];
    }

    __host__ __device__ auto &at(int c, int i, int j, int k) const {
        return at(i, j, k)[c];
    }
};

#define GSL(x, start, end) \
    int x = (start) + blockDim.x * blockIdx.x + threadIdx.x; \
    x < (end); x += blockDim.x * gridDim.x


static inline __constant__ const int directions[][3] = {{0,0,0},{1,0,0},{-1,0,0},{0,1,0},{0,-1,0},{0,0,1},{0,0,-1},{1,1,1},{-1,-1,-1},{1,1,-1},{-1,-1,1},{1,-1,1},{-1,1,-1},{-1,1,1},{1,-1,-1}};
static inline __constant__ const float weights[] = {2.f/9.f, 1.f/9.f, 1.f/9.f, 1.f/9.f, 1.f/9.f, 1.f/9.f, 1.f/9.f,1.f/72.f, 1.f/72.f, 1.f/72.f, 1.f/72.f, 1.f/72.f, 1.f/72.f, 1.f/72.f, 1.f/72.f};

static inline const float niu = 0.005f;
static inline const float tau = 3.f * niu + 0.5f;
static inline const float inv_tau = 1.f / tau;

struct LBM {
    volume<float4> vel;
    volume<float[16]> f_new;
    volume<float[16]> f_old;

    void allocate() {
        vel.allocate();
        f_new.allocate();
        f_old.allocate();
    }

    __device__ float f_eq(int q, int x, int y, int z) {
        float4 v = vel.at(x, y, z);
        float eu = v.x * directions[q][0]
            + v.y * directions[q][1] + v.z * directions[q][2];
        float uv = v.x * v.x + v.y * v.y + v.z * v.z;
        float term = 1.f + 3.f * eu + 4.5f * eu * eu - 1.5f * uv;
        float feq = weights[q] * v.w * term;
        return feq;
    }
};

__global__ void initialize(LBM lbm) {
    for (GSL(z, 0, N)) for (GSL(y, 0, N)) for (GSL(x, 0, N)) {
        lbm.vel.at(x, y, z) = make_float4(0.f, 0.f, 0.f, 1.f);
    }
}

__global__ void substep1(LBM lbm) {
    for (GSL(z, 1, N - 1)) for (GSL(y, 1, N - 1)) for (GSL(x, 1, N - 1)) {
        for (int q = 0; q < 15; q++) {
            int mdx = (x - directions[q][0] + N) % N;
            int mdy = (y - directions[q][1] + N) % N;
            int mdz = (z - directions[q][2] + N) % N;
            lbm.f_new.at(q, x, y, z) = lbm.f_old.at(q, mdx, mdy, mdz)
                * (1.f - inv_tau) + lbm.f_eq(q, mdx, mdy, mdz) * inv_tau;
        }
    }
}

__global__ void substep2(LBM lbm) {
    for (GSL(z, 1, N - 1)) for (GSL(y, 1, N - 1)) for (GSL(x, 1, N - 1)) {
        float m = 0.f;
        float vx = 0.f, vy = 0.f, vz = 0.f;
        for (int q = 0; q < 15; q++) {
            float f = lbm.f_new.at(q, x, y, z);
            lbm.f_old.at(q, x, y, z) = f;
            vx += f * directions[q][0];
            vy += f * directions[q][1];
            vz += f * directions[q][2];
            m += f;
        }
        float mscale = 1.f / fmaxf(m, 1e-6f);
        vx /= mscale; vy /= mscale; vz /= mscale;
        lbm.vel.at(x, y, z) = make_float4(vx, vy, vz, m);
    }
}

template <class T>
struct xyz {
    T t;
    __host__ __device__ xyz(T &t) : t(t) {
    }

    __host__ __device__ xyz &operator=(float3 const &r) {
        t.x = r.x;
        t.y = r.y;
        t.z = r.z;
        return *this;
    }

    __host__ __device__ xyz &operator=(xyz const &r) {
        t.x = r.t.x;
        t.y = r.t.y;
        t.z = r.t.z;
        return *this;
    }
};

__global__ void applybc1(LBM lbm) {
    for (GSL(z, 0, N)) for (GSL(y, 0, N)) {
        /*xyz(lbm.vel.at(0, y, z)) = make_float3(0.1f, 0.f, 0.f);
        for (int q = 0; q < 15; q++) {
            lbm.f_old.at(q, 0, y, z) =
                lbm.f_eq(q, 0, y, z) - lbm.f_eq(q, 1, y, z)
                + lbm.f_old.at(q, 1, y, z);
        }
        xyz(lbm.vel.at(N - 1, y, z)) = xyz(lbm.vel.at(N - 2, y, z));
        for (int q = 0; q < 15; q++) {
            lbm.f_old.at(q, N - 1, y, z) =
                lbm.f_eq(q, N - 1, y, z) - lbm.f_eq(q, N - 2, y, z)
                + lbm.f_old.at(q, N - 2, y, z);
        }*/
        xyz(lbm.vel.at(4, y, z)) = make_float3(0.1f, 0.f, 0.f);
        lbm.vel.at(4, y, z).w = 1.f;
    }
}

LBM lbm;
float *pixels;

void initFunc() {
    lbm.allocate();
    checkCudaErrors(hipMallocManaged(&pixels, N * N * sizeof(float)));
    initialize<<<dim3(N / 8, N / 8, N / 8), dim3(8, 8, 8)>>>(lbm);
}

void stepFunc() {
    substep1<<<dim3(N / 8, N / 8, N / 8), dim3(8, 8, 8)>>>(lbm);
    substep2<<<dim3(N / 8, N / 8, N / 8), dim3(8, 8, 8)>>>(lbm);
    applybc1<<<dim3(1, N / 16, N / 16), dim3(1, 16, 16)>>>(lbm);
}

__global__ void render(float *pixels, LBM lbm) {
    for (GSL(y, 0, N)) for (GSL(x, 0, N)) {
        float4 v = lbm.vel.at(x, y, N / 2);
        float val = sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
        pixels[y * N + x] = 0.0f + val * 400.f;
    }
}

void renderFunc() {
    render<<<dim3(N / 16, N / 16, 1), dim3(16, 16, 1)>>>(pixels, lbm);
    checkCudaErrors(hipDeviceSynchronize());
}

void displayFunc() {
    glClear(GL_COLOR_BUFFER_BIT);
    glDrawPixels(N, N, GL_RED, GL_FLOAT, pixels);
    glFlush();
}

#define ITV 100
void timerFunc(int unused) {
    stepFunc();
    renderFunc();
    glutPostRedisplay();
    glutTimerFunc(ITV, timerFunc, 0);
}

void keyboardFunc(unsigned char key, int x, int y) {
    if (key == 27)
        exit(0);
}

int main(int argc, char **argv) {
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DEPTH | GLUT_SINGLE | GLUT_RGBA);
    glutInitWindowPosition(100, 100);
    glutInitWindowSize(N, N);
    glutCreateWindow("GLUT Window");
    glutDisplayFunc(displayFunc);
    glutKeyboardFunc(keyboardFunc);
    initFunc();
    renderFunc();
    glutTimerFunc(ITV, timerFunc, 0);
    glutMainLoop();
}
