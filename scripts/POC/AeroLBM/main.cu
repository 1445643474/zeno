#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
#include <cassert>
#include <cstdio>
#include <cmath>
#include <GL/glut.h>

#define NX 256
#define NY 64
#define NZ 64

template <class T>
struct xyz {
    T t;
    __host__ __device__ xyz(T &t) : t(t) {
    }

    __host__ __device__ xyz &operator=(float3 const &r) {
        t.x = r.x;
        t.y = r.y;
        t.z = r.z;
        return *this;
    }

    __host__ __device__ xyz &operator=(xyz const &r) {
        t.x = r.t.x;
        t.y = r.t.y;
        t.z = r.t.z;
        return *this;
    }
};

template <class T>
struct volume {
    T *grid;

    void allocate() {
        size_t size = NX * NY * NZ;
        checkCudaErrors(hipMallocManaged(&grid, size * sizeof(T)));
    }

    void free() {
        checkCudaErrors(hipFree(grid));
    }

    __host__ __device__ T &at(int i, int j, int k) const {
        return grid[i + j * NX + k * NX * NY];
    }

    __host__ __device__ auto &at(int c, int i, int j, int k) const {
        return at(i, j, k)[c];
    }
};

#define GSL(_, start, end) \
    int _ = (start) + blockDim._ * blockIdx._ + threadIdx._; \
    _ < (end); _ += blockDim._ * gridDim._


static inline __constant__ const int directions[][3] = {{0,0,0},{1,0,0},{-1,0,0},{0,1,0},{0,-1,0},{0,0,1},{0,0,-1},{1,1,1},{-1,-1,-1},{1,1,-1},{-1,-1,1},{1,-1,1},{-1,1,-1},{-1,1,1},{1,-1,-1}};
static inline __constant__ const float weights[] = {2.f/9.f, 1.f/9.f, 1.f/9.f, 1.f/9.f, 1.f/9.f, 1.f/9.f, 1.f/9.f,1.f/72.f, 1.f/72.f, 1.f/72.f, 1.f/72.f, 1.f/72.f, 1.f/72.f, 1.f/72.f, 1.f/72.f};

static_assert(sizeof(weights) / sizeof(weights[0]) == 15);

static inline const float niu = 0.005f;
static inline const float tau = 3.f * niu + 0.5f;
static inline const float inv_tau = 1.f / tau;

struct LBM {
    volume<float4> vel;
    volume<float[16]> f_new;
    volume<float[16]> f_old;

    void allocate() {
        vel.allocate();
        f_new.allocate();
        f_old.allocate();
    }

    __device__ float f_eq(int q, int x, int y, int z) {
        float4 v = vel.at(x, y, z);
        float eu = v.x * directions[q][0]
            + v.y * directions[q][1] + v.z * directions[q][2];
        float uv = v.x * v.x + v.y * v.y + v.z * v.z;
        float term = 1.f + 3.f * eu + 4.5f * eu * eu - 1.5f * uv;
        float feq = weights[q] * v.w * term;
        return feq;
    }
};

__global__ void initialize1(LBM lbm) {
    for (GSL(z, 0, NZ)) for (GSL(y, 0, NY)) for (GSL(x, 0, NX)) {
        lbm.vel.at(x, y, z) = make_float4(0.f, 0.f, 0.f, 1.f);
    }
}

__global__ void initialize2(LBM lbm) {
    for (GSL(z, 0, NZ)) for (GSL(y, 0, NY)) for (GSL(x, 0, NX)) {
        for (int q = 0; q < 15; q++) {
            float f = lbm.f_eq(q, x, y, z);
            lbm.f_new.at(q, x, y, z) = f;
            lbm.f_old.at(q, x, y, z) = f;
        }
    }
}

__global__ void substep1(LBM lbm) {
    //for (GSL(z, 1, NZ - 1)) for (GSL(y, 1, NY - 1)) for (GSL(x, 1, NX - 1)) {
    for (GSL(z, 0, NZ)) for (GSL(y, 0, NY)) for (GSL(x, 0, NX)) {
        for (int q = 0; q < 15; q++) {
            //int mdx = x - directions[q][0];
            //int mdy = y - directions[q][1];
            //int mdz = z - directions[q][2];
            int mdx = (x - directions[q][0] + NX) % NX;
            int mdy = (y - directions[q][1] + NY) % NY;
            int mdz = (z - directions[q][2] + NZ) % NZ;
            lbm.f_new.at(q, x, y, z) = lbm.f_old.at(q, mdx, mdy, mdz)
                * (1.f - inv_tau) + lbm.f_eq(q, mdx, mdy, mdz) * inv_tau;
        }
    }
}

__global__ void substep2(LBM lbm) {
    //for (GSL(z, 1, NZ - 1)) for (GSL(y, 1, NY - 1)) for (GSL(x, 1, NX - 1)) {
    for (GSL(z, 0, NZ)) for (GSL(y, 0, NY)) for (GSL(x, 0, NX)) {
        float m = 0.f;
        float vx = 0.f, vy = 0.f, vz = 0.f;
        for (int q = 0; q < 15; q++) {
            float f = lbm.f_new.at(q, x, y, z);
            lbm.f_old.at(q, x, y, z) = f;
            vx += f * directions[q][0];
            vy += f * directions[q][1];
            vz += f * directions[q][2];
            m += f;
        }
        float mscale = 1.f / fmaxf(m, 1e-6f);
        vx *= mscale; vy *= mscale; vz *= mscale;
        lbm.vel.at(x, y, z) = make_float4(vx, vy, vz, m);
    }
}

__global__ void applybc1(LBM lbm) {
    for (GSL(z, 1, NZ - 1)) for (GSL(y, 1, NY - 1)) {
    //for (GSL(z, 0, NZ)) for (GSL(y, 0, NY)) {
        lbm.vel.at(0, y, z) = lbm.vel.at(1, y, z);
        lbm.vel.at(0, y, z).x = 0.15f;
        lbm.vel.at(0, y, z).y = 0.f;
        lbm.vel.at(0, y, z).z = 0.f;
        for (int q = 0; q < 15; q++) {
            lbm.f_old.at(q, 0, y, z) =
                lbm.f_eq(q, 0, y, z) - lbm.f_eq(q, 1, y, z)
                + lbm.f_old.at(q, 1, y, z);
        }
        lbm.vel.at(NX - 1, y, z) = lbm.vel.at(NX - 2, y, z);
        for (int q = 0; q < 15; q++) {
            lbm.f_old.at(q, NX - 1, y, z) =
                lbm.f_eq(q, NX - 1, y, z) - lbm.f_eq(q, NX - 2, y, z)
                + lbm.f_old.at(q, NX - 2, y, z);
        }
    }
}

__global__ void applybc2(LBM lbm) {
    for (GSL(z, 0, NZ)) for (GSL(x, 0, NX)) {
        lbm.vel.at(x, 0, z) = lbm.vel.at(x, 1, z);
        lbm.vel.at(x, 0, z).x = 0.f;
        lbm.vel.at(x, 0, z).y = 0.f;
        lbm.vel.at(x, 0, z).z = 0.f;
        for (int q = 0; q < 15; q++) {
            lbm.f_old.at(q, x, 0, z) =
                lbm.f_eq(q, x, 0, z) - lbm.f_eq(q, x, 1, z)
                + lbm.f_old.at(q, x, 1, z);
        }
        lbm.vel.at(x, NY - 1, z) = lbm.vel.at(x, NY - 2, z);
        lbm.vel.at(x, NY - 1, z).x = 0.f;
        lbm.vel.at(x, NY - 1, z).y = 0.f;
        lbm.vel.at(x, NY - 1, z).z = 0.f;
        for (int q = 0; q < 15; q++) {
            lbm.f_old.at(q, x, NY - 1, z) =
                lbm.f_eq(q, x, NY - 1, z) - lbm.f_eq(q, x, NY - 2, z)
                + lbm.f_old.at(q, x, NY - 2, z);
        }
    }
}

__global__ void applybc3(LBM lbm) {
    for (GSL(y, 0, NY)) for (GSL(x, 0, NX)) {
        lbm.vel.at(x, y, 0) = lbm.vel.at(x, y, 1);
        lbm.vel.at(x, y, 0).x = 0.f;
        lbm.vel.at(x, y, 0).y = 0.f;
        lbm.vel.at(x, y, 0).z = 0.f;
        for (int q = 0; q < 15; q++) {
            lbm.f_old.at(q, x, y, 0) =
                lbm.f_eq(q, x, y, 0) - lbm.f_eq(q, x, y, 1)
                + lbm.f_old.at(q, x, y, 1);
        }
        lbm.vel.at(x, y, NZ - 1) = lbm.vel.at(x, y, NZ - 2);
        lbm.vel.at(x, y, NZ - 1).x = 0.f;
        lbm.vel.at(x, y, NZ - 1).y = 0.f;
        lbm.vel.at(x, y, NZ - 1).z = 0.f;
        for (int q = 0; q < 15; q++) {
            lbm.f_old.at(q, x, y, NZ - 1) =
                lbm.f_eq(q, x, y, NZ - 1) - lbm.f_eq(q, x, y, NZ - 2)
                + lbm.f_old.at(q, x, y, NZ - 2);
        }
    }
}

__global__ void applybc4(LBM lbm) {
    for (GSL(z, 0, NZ)) for (GSL(y, 0, NY)) for (GSL(x, 0, NX)) {
        float fx = x * 8.f / NX - 1.f;
        float fy = y * 2.f / NY - 1.f;
        float fz = z * 2.f / NZ - 1.f;
        if (fx * fx + fy * fy + fz * fz >= .08f) {
            continue;
        }
        lbm.vel.at(x, y, z).x = 0.f;
        lbm.vel.at(x, y, z).y = 0.f;
        lbm.vel.at(x, y, z).z = 0.f;
    }
}

LBM lbm;
float *pixels;

void initFunc() {
    lbm.allocate();
    checkCudaErrors(hipMallocManaged(&pixels, NX * NY * sizeof(float)));
    initialize1<<<dim3(NX / 8, NY / 8, NZ / 8), dim3(8, 8, 8)>>>(lbm);
    initialize2<<<dim3(NX / 8, NY / 8, NZ / 8), dim3(8, 8, 8)>>>(lbm);
}

void stepFunc() {
    substep1<<<dim3(NX / 8, NY / 8, NZ / 8), dim3(8, 8, 8)>>>(lbm);
    substep2<<<dim3(NX / 8, NY / 8, NZ / 8), dim3(8, 8, 8)>>>(lbm);
    applybc1<<<dim3(1, NY / 16, NZ / 16), dim3(1, 16, 16)>>>(lbm);
    applybc2<<<dim3(NX / 16, 1, NZ / 16), dim3(16, 1, 16)>>>(lbm);
    applybc3<<<dim3(NX / 16, NY / 16, 1), dim3(16, 16, 1)>>>(lbm);
    applybc4<<<dim3(NX / 16, NY / 16, NZ / 16), dim3(8, 8, 8)>>>(lbm);
}

__global__ void render(float *pixels, LBM lbm) {
    for (GSL(y, 0, NY)) for (GSL(x, 0, NX)) {
        float4 v = lbm.vel.at(x, y, NZ / 2);
        //float val = sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
        float val = 4.f * sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
        //float val = v.x * 4.f;
        //float val = v.w * 0.3f;
        pixels[y * NX + x] = val;
    }
}

void renderFunc() {
    render<<<dim3(NX / 16, NY / 16, 1), dim3(16, 16, 1)>>>(pixels, lbm);
    checkCudaErrors(hipDeviceSynchronize());
    /*printf("03:%f\n", pixels[0 * N + 3]);
    printf("30:%f\n", pixels[3 * NX + 0]);
    printf("33:%f\n", pixels[3 * NX + 3]);*/
}

void displayFunc() {
    glClear(GL_COLOR_BUFFER_BIT);
    glDrawPixels(NX, NY, GL_RED, GL_FLOAT, pixels);
    glFlush();
}

#define ITV 0
void timerFunc(int unused) {
    stepFunc();
    renderFunc();
    glutPostRedisplay();
    glutTimerFunc(ITV, timerFunc, 0);
}

void keyboardFunc(unsigned char key, int x, int y) {
    if (key == 27)
        exit(0);
}

int main(int argc, char **argv) {
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DEPTH | GLUT_SINGLE | GLUT_RGBA);
    glutInitWindowPosition(100, 100);
    glutInitWindowSize(NX, NY);
    glutCreateWindow("GLUT Window");
    glutDisplayFunc(displayFunc);
    glutKeyboardFunc(keyboardFunc);
    initFunc();
    renderFunc();
    glutTimerFunc(ITV, timerFunc, 0);
    glutMainLoop();
}
