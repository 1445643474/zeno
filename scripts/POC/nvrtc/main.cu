#include "hip/hip_runtime.h"
#if 0
#include <stdio.h>
__global__ void test() { printf("FuCK U NVIDIA!\n"); } int main(void) { test<<<1, 1>>>(); hipDeviceSynchronize(); }
#else

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_vector_types.h"
#include "nvrtc_helper.h"
#include <cassert>
#include <cstdio>
#include <cmath>


int main(int argc, char **argv)
{
    char *cubin;
    size_t cubinSize;
    char *kernel_file = sdkFindFilePath("kernel.cu", argv[0]);
    compileFileToCUBIN(kernel_file, argc, argv, &cubin, &cubinSize, 0);

    hipModule_t module = loadCUBIN(cubin, argc, argv);
    hipFunction_t kernel_addr;
    checkCudaErrors(hipModuleGetFunction(&kernel_addr, module, "kernel_func"));

    void *args[] = {};
    checkCudaErrors(hipModuleLaunchKernel(kernel_addr, 1, 1, 1, 1, 1, 1,
        0, 0, args, 0));

    checkCudaErrors(hipCtxSynchronize());

    return 0;
}
#endif
