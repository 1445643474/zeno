#include "hip/hip_runtime.h"
#include "PBD.cuh"

namespace zeno {

void PBDSystem::PrimitiveHandle::initGeo() {
    // init rest volumes & edge lengths
    auto cudaPol = zs::cuda_exec();
    using namespace zs;
    constexpr auto space = zs::execspace_e::cuda;
    elesPtr->append_channels(cudaPol, {{"rv", 1}});
    cudaPol(zs::Collapse{elesPtr->size()},
            [eles = proxy<space>({}, *elesPtr), verts = proxy<space>({}, *vertsPtr)] ZS_LAMBDA(int ei) mutable {
                auto quad = eles.template pack<4>("inds", ei).template reinterpret_bits<int>();
                vec3 xs[4];
                for (int d = 0; d != 4; ++d)
                    xs[d] = verts.template pack<3>("x", quad[d]);
                vec3 ds[3] = {xs[1] - xs[0], xs[2] - xs[0], xs[3] - xs[0]};
                mat3 D{};
                for (int d = 0; d != 3; ++d)
                    for (int i = 0; i != 3; ++i)
                        D(d, i) = ds[i][d];

                eles("rv", ei) = zs::abs(zs::determinant(D)) / 6;
            });
    surfEdgesPtr->append_channels(cudaPol, {{"rl", 1}});
    cudaPol(zs::Collapse{surfEdgesPtr->size()},
            [ses = proxy<space>({}, *surfEdgesPtr), verts = proxy<space>({}, *vertsPtr)] ZS_LAMBDA(int sei) mutable {
                auto line = ses.template pack<2>("inds", sei).template reinterpret_bits<int>();
                vec3 xs[2];
                for (int d = 0; d != 2; ++d)
                    xs[d] = verts.template pack<3>("x", line[d]);
                ses("rl", sei) = (xs[1] - xs[0]).length();
            });
}

PBDSystem::PrimitiveHandle::PrimitiveHandle(ZenoParticles &zsprim, std::size_t &vOffset, std::size_t &sfOffset,
                                            std::size_t &seOffset, std::size_t &svOffset, zs::wrapv<4>)
    : zsprimPtr{&zsprim, [](void *) {}}, models{zsprim.getModel()}, vertsPtr{&zsprim.getParticles<true>(),
                                                                             [](void *) {}},
      elesPtr{&zsprim.getQuadraturePoints(), [](void *) {}}, surfTrisPtr{&zsprim[ZenoParticles::s_surfTriTag],
                                                                         [](void *) {}},
      surfEdgesPtr{&zsprim[ZenoParticles::s_surfEdgeTag], [](void *) {}},
      surfVertsPtr{&zsprim[ZenoParticles::s_surfVertTag], [](void *) {}}, vOffset{vOffset}, sfOffset{sfOffset},
      seOffset{seOffset}, svOffset{svOffset}, category{zsprim.category} {
    if (category != ZenoParticles::tet)
        throw std::runtime_error("dimension of 4 but is not tetrahedra");
    initGeo();
    vOffset += getVerts().size();
    sfOffset += getSurfTris().size();
    seOffset += getSurfEdges().size();
    svOffset += getSurfVerts().size();
}

void PBDSystem::initialize(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    stInds = tiles_t{vtemp.get_allocator(), {{"inds", 3}}, sfOffset};
    seInds = tiles_t{vtemp.get_allocator(), {{"inds", 2}}, seOffset};
    svInds = tiles_t{vtemp.get_allocator(), {{"inds", 1}}, svOffset};
    for (auto &primHandle : prims) {
        auto &verts = primHandle.getVerts();
        // record surface (tri) indices
        if (primHandle.category != ZenoParticles::category_e::curve) {
            auto &tris = primHandle.getSurfTris();
            pol(Collapse(tris.size()),
                [stInds = proxy<space>({}, stInds), tris = proxy<space>({}, tris), voffset = primHandle.vOffset,
                 sfoffset = primHandle.sfOffset] __device__(int i) mutable {
                    stInds.template tuple<3>("inds", sfoffset + i) =
                        (tris.template pack<3>("inds", i).template reinterpret_bits<int>() + (int)voffset)
                            .template reinterpret_bits<float>();
                });
        }
        auto &edges = primHandle.getSurfEdges();
        pol(Collapse(edges.size()),
            [seInds = proxy<space>({}, seInds), edges = proxy<space>({}, edges), voffset = primHandle.vOffset,
             seoffset = primHandle.seOffset] __device__(int i) mutable {
                seInds.template tuple<2>("inds", seoffset + i) =
                    (edges.template pack<2>("inds", i).template reinterpret_bits<int>() + (int)voffset)
                        .template reinterpret_bits<float>();
            });
        auto &points = primHandle.getSurfVerts();
        pol(Collapse(points.size()),
            [svInds = proxy<space>({}, svInds), points = proxy<space>({}, points), voffset = primHandle.vOffset,
             svoffset = primHandle.svOffset] __device__(int i) mutable {
                svInds("inds", svoffset + i) =
                    reinterpret_bits<float>(reinterpret_bits<int>(points("inds", i)) + (int)voffset);
            });
    }
    reinitialize(pol, dt);
}

void PBDSystem::reinitialize(zs::CudaExecutionPolicy &pol, T framedt) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    this->dt = framedt;
    for (auto &primHandle : prims) {
        auto &verts = primHandle.getVerts();
        // initialize BC info
        // predict pos, initialize augmented lagrangian, constrain weights
        pol(Collapse(verts.size()), [vtemp = proxy<space>({}, vtemp), verts = proxy<space>({}, verts),
                                     voffset = primHandle.vOffset, dt = dt] __device__(int i) mutable {
            auto x = verts.pack<3>("x", i);
            auto v = verts.pack<3>("v", i);
            vtemp.tuple<3>("x", voffset + i) = x;
            vtemp.tuple<3>("xpre", voffset + i) = x;
            vtemp.tuple<3>("v", voffset + i) = v;
        });
    }
}

PBDSystem::PBDSystem(std::vector<ZenoParticles *> zsprims, vec3 extForce, T dt, int numSolveIters)
    : extForce{extForce}, solveIterCap{numSolveIters}, prims{}, coOffset{0}, numDofs{0}, sfOffset{0}, seOffset{0},
      svOffset{0}, vtemp{}, temp{}, stInds{}, seInds{}, svInds{}, dt{dt} {
    for (auto primPtr : zsprims) {
        if (primPtr->category == ZenoParticles::category_e::tet) {
            prims.emplace_back(*primPtr, coOffset, sfOffset, seOffset, svOffset, zs::wrapv<4>{});
        }
    }
    zeno::log_info("num total obj <verts, surfV, surfE, surfT>: {}, {}, {}, {}\n", coOffset, svOffset, seOffset,
                   sfOffset);
    numDofs = coOffset; // if there are boundaries, then updated
    vtemp = dtiles_t{zsprims[0]->getParticles().get_allocator(), {{"x", 3}, {"xpre", 3}, {"v", 3}}, numDofs};

    auto cudaPol = zs::cuda_exec();
    initialize(cudaPol);
}

} // namespace zeno