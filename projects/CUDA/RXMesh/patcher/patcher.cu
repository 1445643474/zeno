#include "hip/hip_runtime.h"
#include <zeno/utils/log.h>
#include <algorithm>
#include <numeric>
#include <random>
#include <cstring>
#include <assert.h>
#include <stdint.h>
#include <functional>
#include <iomanip>
#include <queue>
#include <unordered_map>
#include <cub/device/device_reduce.cuh>
#include <cub/device/device_scan.cuh>
#include "../utils/util.cuh"
#include "patcher.h"
#include "patcher_kernel.cuh"


namespace zeno::rxmesh {
        
Patcher::Patcher(uint32_t                                        patch_size,
                 const std::vector<uint32_t>&                    ff_offset,
                 const std::vector<uint32_t>&                    ff_values,
                 const std::vector<std::vector<uint32_t>>&       fv,
                 const std::unordered_map<std::pair<uint32_t, uint32_t>,
                                          uint32_t,
                                          zeno::rxmesh::detail::edge_key_hash> edges_map,
                 const uint32_t                                  num_vertices,
                 const uint32_t                                  num_edges)
    : m_patch_size(patch_size),
      m_num_patches(0),
      m_num_vertices(num_vertices),
      m_num_edges(num_edges),
      m_num_faces(fv.size()),
      m_num_seeds(0),
      m_max_num_patches(0),
      m_num_components(0),
      m_num_lloyd_run(0),
      m_d_face_patch(nullptr),
      m_d_vertex_patch(nullptr),
      m_d_edge_patch(nullptr),
      m_d_patches_offset(nullptr),
      m_d_patches_size(nullptr),
      m_d_patches_val(nullptr),
      m_patching_time_ms(0.0),
      m_d_seeds(nullptr),
      m_d_ff_values(nullptr),
      m_d_ff_offset(nullptr),
      m_d_queue(nullptr),
      m_d_queue_ptr(nullptr),
      m_d_new_num_patches(nullptr),
      m_d_max_patch_size(nullptr),
      m_d_cub_temp_storage_scan(nullptr),
      m_d_cub_temp_storage_max(nullptr),
      m_cub_scan_bytes(0),
      m_cub_max_bytes(0) {

    m_num_patches = (m_num_faces + m_patch_size - 1) / m_patch_size;

    m_max_num_patches = 5 * m_num_patches;

    m_num_seeds = m_num_patches;

    allocate_memory();

    // degenerate cases
    if (m_num_patches <= 1) {
        m_patches_offset[0] = m_num_faces;
        m_num_seeds         = 1;
        m_num_components    = 1;
        m_num_lloyd_run     = 0;
        for (uint32_t i = 0; i < m_num_faces; ++i) {
            m_face_patch[i]  = 0;
            m_patches_val[i] = i;
        }        
        allocate_device_memory(ff_offset, ff_values);
        assign_patch(fv, edges_map);
    } else {
        initialize_random_seeds(ff_offset, ff_values);
        allocate_device_memory(ff_offset, ff_values);
        run_lloyd();
        postprocess(fv, ff_offset, ff_values);
        assign_patch(fv, edges_map);
    }
}

Patcher::~Patcher() {
    GPU_FREE(m_d_face_patch);
    GPU_FREE(m_d_vertex_patch);
    GPU_FREE(m_d_edge_patch);
}

void Patcher::allocate_memory() {
    m_seeds.reserve(m_num_seeds);

    // patches assigned to each face, vertex, and edge
    m_face_patch.resize(m_num_faces);
    std::fill(m_face_patch.begin(), m_face_patch.end(), INVALID32);

    m_vertex_patch.resize(m_num_vertices);
    std::fill(m_vertex_patch.begin(), m_vertex_patch.end(), INVALID32);

    m_edge_patch.resize(m_num_edges);
    std::fill(m_edge_patch.begin(), m_edge_patch.end(), INVALID32);

    // explicit patches in compressed format
    m_patches_val.resize(m_num_faces);

    // we allow up to double the number of faces due to patch bisecting
    m_patches_offset.resize(m_max_num_patches);

    // external ribbon. it assumes first that all faces will be in there and
    // then shrink to fit after the construction is done
    m_ribbon_ext_offset.resize(m_max_num_patches, 0);

    m_ribbon_ext_val.resize(m_num_faces);
}

void Patcher::allocate_device_memory(const std::vector<uint32_t>& ff_offset,
                                     const std::vector<uint32_t>& ff_values) {
    // ff
    CUDA_ERROR(hipMalloc((void**)&m_d_ff_values,
                          ff_values.size() * sizeof(uint32_t)));
    CUDA_ERROR(hipMalloc((void**)&m_d_ff_offset,
                          ff_offset.size() * sizeof(uint32_t)));

    CUDA_ERROR(hipMemcpy((void**)m_d_ff_values,
                          ff_values.data(),
                          ff_values.size() * sizeof(uint32_t),
                          hipMemcpyHostToDevice));

    CUDA_ERROR(hipMemcpy((void**)m_d_ff_offset,
                          ff_offset.data(),
                          ff_offset.size() * sizeof(uint32_t),
                          hipMemcpyHostToDevice));
    // face/vertex/edge patch
    CUDA_ERROR(hipMalloc((void**)&m_d_face_patch, m_num_faces * sizeof(uint32_t)));
    CUDA_ERROR(hipMalloc((void**)&m_d_vertex_patch, m_num_vertices * sizeof(uint32_t)));
    CUDA_ERROR(hipMalloc((void**)&m_d_edge_patch, m_num_edges * sizeof(uint32_t)));

    // seeds
    CUDA_ERROR(hipMalloc((void**)&m_d_seeds, m_max_num_patches * sizeof(uint32_t)));

    CUDA_ERROR(hipMemcpy((void**)m_d_seeds,
                          m_seeds.data(),
                          m_num_patches * sizeof(uint32_t),
                          hipMemcpyHostToDevice));

    // utility
    // 0 -> queue start
    // 1-> queue end
    // 2-> next queue end
    std::vector<uint32_t> h_queue_ptr{0, m_num_patches, m_num_patches};
    CUDA_ERROR(hipMalloc((void**)&m_d_queue, m_num_faces * sizeof(uint32_t)));
    CUDA_ERROR(hipMalloc((void**)&m_d_queue_ptr, 3 * sizeof(uint32_t)));
    CUDA_ERROR(hipMemcpy(m_d_queue_ptr,
                          h_queue_ptr.data(),
                          3 * sizeof(uint32_t),
                          hipMemcpyHostToDevice));

    // patch offset/size/value and max patch size
    CUDA_ERROR(hipMalloc((void**)&m_d_patches_offset,
                          m_max_num_patches * sizeof(uint32_t)));
    CUDA_ERROR(hipMalloc((void**)&m_d_patches_size,
                          m_max_num_patches * sizeof(uint32_t)));
    CUDA_ERROR(hipMalloc((void**)&m_d_patches_val, m_num_faces * sizeof(uint32_t)));
    CUDA_ERROR(hipMalloc((void**)&m_d_max_patch_size, sizeof(uint32_t)));

    CUDA_ERROR(hipMalloc((void**)&m_d_new_num_patches, sizeof(uint32_t)));

    CUDA_ERROR(hipMemcpy((void**)m_d_new_num_patches,
                          &m_num_patches,
                          sizeof(uint32_t),
                          hipMemcpyHostToDevice));

    // CUB temp memory
    m_d_cub_temp_storage_scan = nullptr;
    m_d_cub_temp_storage_max  = nullptr;
    m_cub_scan_bytes          = 0;
    m_cub_max_bytes           = 0;
    hipcub::DeviceScan::InclusiveSum(m_d_cub_temp_storage_scan,
                                    m_cub_scan_bytes,
                                    m_d_patches_size,
                                    m_d_patches_offset,
                                    m_max_num_patches);
    hipcub::DeviceReduce::Max(m_d_cub_temp_storage_max,
                             m_cub_max_bytes,
                             m_d_patches_size,
                             m_d_max_patch_size,
                             m_max_num_patches);
    CUDA_ERROR(hipMalloc((void**)&m_d_cub_temp_storage_scan, m_cub_scan_bytes));
    CUDA_ERROR(hipMalloc((void**)&m_d_cub_temp_storage_max, m_cub_max_bytes));
}

void Patcher::initialize_random_seeds(const std::vector<uint32_t>& ff_offset,
                                      const std::vector<uint32_t>& ff_values) {

    // 1) Identify the components i.e., for each component list the faces
    // that belong to that it
    // 2) Generate number of (random) seeds in each component
    // proportional to the number of faces it contain

    std::vector<std::vector<uint32_t>> components;
    get_multi_components(components, ff_offset, ff_values);

    m_num_components = components.size();
    if (m_num_components == 1) {
        initialize_random_seeds_single_component();
    } else {
        if (m_num_seeds <= m_num_components) {
            // too many components
            m_num_seeds = m_num_components;
            for (auto& comp : components) {
                generate_random_seed_from_component(comp, 1);
            }
        } else {
            // more seeds than components
            uint32_t num_remaining_seeds      = m_num_seeds - m_num_components;
            uint32_t num_extra_seeds_inserted = 0;

            std::vector<size_t> component_order(components.size());
            size_t* start = component_order.data();
            std::iota(start, start + component_order.size(), 0);
            std::sort(component_order.begin(),
                      component_order.end(),
                      [&components](const size_t& a, const size_t& b) {
                          return components[a].size() > components[b].size();
                      });

            for (size_t c = 0; c < component_order.size(); ++c) {

                std::vector<uint32_t>& comp = components[component_order[c]];

                uint32_t size = comp.size();
                float weight =
                    static_cast<float>(size) / static_cast<float>(m_num_faces);
                uint32_t component_num_seeds = static_cast<uint32_t>(std::ceil(
                    weight * static_cast<float>(num_remaining_seeds)));

                num_extra_seeds_inserted += component_num_seeds;
                if (num_extra_seeds_inserted > num_remaining_seeds) {
                    if (num_extra_seeds_inserted - num_remaining_seeds >
                        component_num_seeds) {
                        component_num_seeds = 0;
                    } else {
                        component_num_seeds -=
                            (num_extra_seeds_inserted - num_remaining_seeds);
                    }
                }

                component_num_seeds += 1;
                generate_random_seed_from_component(comp, component_num_seeds);
            }
        }
    }

    assert(m_num_patches == m_seeds.size());
}

void Patcher::initialize_random_seeds_single_component() {
    std::vector<uint32_t> rand_num(m_num_faces);
    uint32_t* start = rand_num.data();
    size_t size = rand_num.size();
    std::iota(start, start + size, 0);
    std::random_device rd;
    std::mt19937       g(rd());
    std::shuffle(start, start + size, g);
    m_seeds.resize(m_num_seeds);
    std::memcpy(
        m_seeds.data(), start, m_num_seeds * sizeof(uint32_t));
}

void Patcher::generate_random_seed_from_component(
    std::vector<uint32_t>& component,
    const uint32_t         num_seeds) {
    uint32_t num_seeds_before = m_seeds.size();
    if (num_seeds < 1) {
        zeno::log_error(
            "Patcher::generate_random_seed_in_component() num_seeds should be "
            "no smaller than 1");
    }

    uint32_t* start = component.data();
    std::random_device rd;
    std::mt19937       g(rd());
    std::shuffle(start, start + component.size(), g);
    m_seeds.resize(num_seeds_before + num_seeds);
    std::memcpy(m_seeds.data() + num_seeds_before,
                start,
                num_seeds * sizeof(uint32_t));
}


void Patcher::get_multi_components(
    std::vector<std::vector<uint32_t>>& components,
    const std::vector<uint32_t>&        ff_offset,
    const std::vector<uint32_t>&        ff_values) {
    std::vector<bool> visited(m_num_faces, false);
    for (uint32_t f = 0; f < m_num_faces; ++f) {
        if (!visited[f]) {
            std::vector<uint32_t> current_component;
            // just a guess
            current_component.reserve(
                static_cast<uint32_t>(static_cast<double>(m_num_faces) / 10.0));

            std::queue<uint32_t> face_queue;
            // bfs faces
            face_queue.push(f);
            while (!face_queue.empty()) {
                uint32_t face = face_queue.front();
                face_queue.pop();
                uint32_t start = (face == 0) ? 0 : ff_offset[face - 1];
                uint32_t end   = ff_offset[face];
                for (uint32_t f = start; f < end; ++f) {
                    uint32_t n_face = ff_values[f];
                    if (!visited[n_face]) {
                        current_component.push_back(n_face);
                        face_queue.push(n_face);
                        visited[n_face] = true;
                    }
                }
            }

            components.push_back(current_component);
        }
    }
}

void Patcher::postprocess(const std::vector<std::vector<uint32_t>>& fv,
                          const std::vector<uint32_t>&              ff_offset,
                          const std::vector<uint32_t>&              ff_values) {
    // Post process the patches by extracting the ribbons 

    std::vector<uint32_t> frontier;
    frontier.reserve(m_num_faces);

    std::vector<uint32_t> bd_vertices;
    bd_vertices.reserve(m_patch_size);

    // build vertex incident faces
    std::vector<std::vector<uint32_t>> vertex_incident_faces(
        m_num_vertices, std::vector<uint32_t>(10));
    for (uint32_t i = 0; i < vertex_incident_faces.size(); ++i) {
        vertex_incident_faces[i].clear();
    }
    for (uint32_t face = 0; face < m_num_faces; ++face) {
        for (uint32_t v = 0; v < fv[face].size(); ++v) {
            vertex_incident_faces[fv[face][v]].push_back(face);
        }
    }

    for (uint32_t cur_p = 0; cur_p < m_num_patches; ++cur_p) {

        uint32_t p_start = (cur_p == 0) ? 0 : m_patches_offset[cur_p - 1];
        uint32_t p_end   = m_patches_offset[cur_p];

        bd_vertices.clear();
        frontier.clear();

        //***** Pass One
        // 1) build a frontier of the boundary faces by loop over all faces and
        // add those that has an edge on the patch boundary
        for (uint32_t fb = p_start; fb < p_end; ++fb) {
            uint32_t face = m_patches_val[fb];

            bool     added = false;
            uint32_t start = (face == 0) ? 0 : ff_offset[face - 1];
            uint32_t end   = ff_offset[face];

            for (uint32_t g = start; g < end; ++g) {
                uint32_t n       = ff_values[g];
                uint32_t n_patch = get_face_patch_id(n);

                if (n_patch != cur_p) {
                    if (!added) {
                        frontier.push_back(face);
                        added = true;
                    }

                    for (uint32_t i = 0; i < fv[face].size(); ++i) {
                        auto it_vf =
                            std::find(fv[n].begin(), fv[n].end(), fv[face][i]);
                        if (it_vf != fv[n].end()) {
                            bd_vertices.push_back(fv[face][i]);
                        }
                    }
                }
            }
        }

        std::sort(bd_vertices.begin(), bd_vertices.end());
        uint32_t next_unique_id = 1;
        uint32_t prev_value = bd_vertices.front();
        for (uint32_t i = 1; i < bd_vertices.size(); ++i) {
            uint32_t curr_val = bd_vertices[i];
            if (curr_val != prev_value) {
                bd_vertices[next_unique_id++] = curr_val;
                prev_value = curr_val;
            }
        }

        bd_vertices.resize(next_unique_id);


        //***** Pass Two
        // 3) for every vertex on the patch boundary, we add all the faces
        // that are incident to it and not in the current patch

        m_ribbon_ext_offset[cur_p] =
            (cur_p == 0) ? 0 : m_ribbon_ext_offset[cur_p - 1];
        uint32_t r_start = m_ribbon_ext_offset[cur_p];

        for (uint32_t v = 0; v < bd_vertices.size(); ++v) {
            uint32_t vert = bd_vertices[v];

            for (uint32_t f = 0; f < vertex_incident_faces[vert].size(); ++f) {
                uint32_t face = vertex_incident_faces[vert][f];
                if (get_face_patch_id(face) != cur_p) {
                    bool     added = false;
                    uint32_t r_end = m_ribbon_ext_offset[cur_p];
                    for (uint32_t r = r_start; r < r_end; ++r) {
                        if (m_ribbon_ext_val[r] == face) {
                            added = true;
                            break;
                        }
                    }
                    if (!added) {

                        m_ribbon_ext_val[m_ribbon_ext_offset[cur_p]] = face;
                        m_ribbon_ext_offset[cur_p]++;
                        if (m_ribbon_ext_offset[cur_p] == m_num_faces) {
                            uint32_t new_size = m_ribbon_ext_val.size() * 2;
                            m_ribbon_ext_val.resize(new_size);
                        }
                        assert(m_ribbon_ext_offset[cur_p] <=
                               m_ribbon_ext_val.size());
                    }
                }
            }
        }
    }

    m_ribbon_ext_val.resize(m_ribbon_ext_offset[m_num_patches - 1]);
}

void Patcher::assign_patch(
    const std::vector<std::vector<uint32_t>>&                 fv,
    const std::unordered_map<std::pair<uint32_t, uint32_t>,
                             uint32_t,
                             ::zeno::rxmesh::detail::edge_key_hash> edges_map) {
    // For every patch p, for every face in the patch, find the three edges
    // that bound that face, and assign them to the patch. For boundary vertices
    // and edges assign them to one patch the first patch.

    for (uint32_t cur_p = 0; cur_p < m_num_patches; ++cur_p) {

        uint32_t p_start = (cur_p == 0) ? 0 : m_patches_offset[cur_p - 1];
        uint32_t p_end   = m_patches_offset[cur_p];

        for (uint32_t f = p_start; f < p_end; ++f) {

            uint32_t face = m_patches_val[f];

            uint32_t v1 = fv[face].back();
            for (uint32_t v = 0; v < fv[face].size(); ++v) {
                uint32_t v0 = fv[face][v];

                std::pair<uint32_t, uint32_t> key =
                    ::zeno::rxmesh::detail::edge_key(v0, v1);
                uint32_t edge_id = edges_map.at(key);

                if (m_vertex_patch[v0] == INVALID32) {
                    m_vertex_patch[v0] = cur_p;
                }

                if (m_edge_patch[edge_id] == INVALID32) {
                    m_edge_patch[edge_id] = cur_p;
                }

                v1 = v0;
            }
        }
    }


    CUDA_ERROR(hipMemcpy(m_d_edge_patch,
                          m_edge_patch.data(),
                          sizeof(uint32_t) * (m_num_edges),
                          hipMemcpyHostToDevice));
    CUDA_ERROR(hipMemcpy(m_d_vertex_patch,
                          m_vertex_patch.data(),
                          sizeof(uint32_t) * (m_num_vertices),
                          hipMemcpyHostToDevice));
}

void Patcher::run_lloyd() {
    std::vector<uint32_t> h_queue_ptr{0, m_num_patches, m_num_patches};

    m_num_lloyd_run = 0;
    while (true) {
        ++m_num_lloyd_run;

        const uint32_t threads_s = 256;
        const uint32_t blocks_s  = (m_num_patches + threads_s - 1) / threads_s;
        const uint32_t threads_f = 256;
        const uint32_t blocks_f  = (m_num_faces + threads_f - 1) / threads_f;

        // add more seeds if needed
        if (m_num_lloyd_run % 5 == 0 && m_num_lloyd_run > 0) {
            uint32_t threshold = m_patch_size;

            CUDA_ERROR(hipMemcpy(m_d_new_num_patches,
                                  &m_num_patches,
                                  sizeof(uint32_t),
                                  hipMemcpyHostToDevice));
            add_more_seeds<<<m_num_patches, 1>>>(m_num_patches,
                                                 m_d_new_num_patches,
                                                 m_d_seeds,
                                                 m_d_patches_offset,
                                                 m_d_patches_val,
                                                 threshold);

            CUDA_ERROR(hipMemcpy(&m_num_patches,
                                  m_d_new_num_patches,
                                  sizeof(uint32_t),
                                  hipMemcpyDeviceToHost));

            if (m_num_patches >= m_max_num_patches) {
                zeno::log_error(
                    "Patcher::run_lloyd() m_num_patches exceeds "
                    "m_max_num_patches");
            }
        }
        h_queue_ptr[0] = 0;
        h_queue_ptr[1] = m_num_patches;
        h_queue_ptr[2] = m_num_patches;
        CUDA_ERROR(hipMemcpy(m_d_queue_ptr,
                              h_queue_ptr.data(),
                              3 * sizeof(uint32_t),
                              hipMemcpyHostToDevice));

        memset<<<blocks_f, threads_f>>>(
            m_d_face_patch, INVALID32, m_num_faces);

        memcpy<<<blocks_s, threads_s>>>(
            m_d_queue, m_d_seeds, m_num_patches);

        memset<<<blocks_s, threads_s>>>(
            m_d_patches_size, 0u, m_num_patches);

        write_initial_face_patch<<<blocks_s, threads_s>>>(
            m_num_patches, m_d_face_patch, m_d_seeds, m_d_patches_size);

        // Cluster seed propagation
        while (true) {
            cluster_seed_propagation<<<blocks_f, threads_f>>>(m_num_faces,
                                                              m_num_patches,
                                                              m_d_queue_ptr,
                                                              m_d_queue,
                                                              m_d_face_patch,
                                                              m_d_patches_size,
                                                              m_d_ff_offset,
                                                              m_d_ff_values);

            reset_queue_ptr<<<1, 1>>>(m_d_queue_ptr);

            CUDA_ERROR(hipMemcpy(h_queue_ptr.data(),
                                  m_d_queue_ptr,
                                  sizeof(uint32_t),
                                  hipMemcpyDeviceToHost));

            if (h_queue_ptr[0] >= m_num_faces) {
                break;
            }
        }

        uint32_t max_patch_size = construct_patches_compressed_format();

        uint32_t threads_i   = 512;
        uint32_t shmem_bytes = max_patch_size * (sizeof(uint32_t));
        memset<<<blocks_f, threads_f>>>(
            m_d_queue, INVALID32, m_num_faces);
        interior<<<m_num_patches, threads_i, shmem_bytes>>>(m_num_patches,
                                                            m_d_patches_offset,
                                                            m_d_patches_val,
                                                            m_d_face_patch,
                                                            m_d_seeds,
                                                            m_d_ff_offset,
                                                            m_d_ff_values,
                                                            m_d_queue);
        // if current max_patch_size is already smaller than m_patch_size,
        // the lloyd algorithm stops.
        if (max_patch_size < m_patch_size) {
            shift<<<blocks_f, threads_f>>>(
                m_num_faces, m_d_face_patch, m_d_patches_val);

            break;
        }
    }

    CUDA_ERROR(hipDeviceSynchronize());
    CUDA_ERROR(hipGetLastError());

    m_num_seeds = m_num_patches;
    m_seeds.resize(m_num_seeds);
    CUDA_ERROR(hipMemcpy(m_seeds.data(),
                          m_d_seeds,
                          m_num_seeds * sizeof(uint32_t),
                          hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(m_face_patch.data(),
                          m_d_face_patch,
                          sizeof(uint32_t) * m_num_faces,
                          hipMemcpyDeviceToHost));
    m_patches_offset.resize(m_num_patches);
    CUDA_ERROR(hipMemcpy(m_patches_offset.data(),
                          m_d_patches_offset,
                          sizeof(uint32_t) * m_num_patches,
                          hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(m_patches_val.data(),
                          m_d_patches_val,
                          sizeof(uint32_t) * m_num_faces,
                          hipMemcpyDeviceToHost));

    GPU_FREE(m_d_ff_values);
    GPU_FREE(m_d_ff_offset);

    GPU_FREE(m_d_new_num_patches);
    GPU_FREE(m_d_max_patch_size);

    GPU_FREE(m_d_cub_temp_storage_scan);
    GPU_FREE(m_d_cub_temp_storage_max);
    m_cub_max_bytes  = 0;
    m_cub_scan_bytes = 0;

    GPU_FREE(m_d_seeds);
    GPU_FREE(m_d_queue);
    GPU_FREE(m_d_queue_ptr);

    GPU_FREE(m_d_patches_offset);
    GPU_FREE(m_d_patches_size);
    GPU_FREE(m_d_patches_val);
}

uint32_t Patcher::construct_patches_compressed_format() {
    uint32_t       max_patch_size = 0;
    const uint32_t threads_s      = 256;
    const uint32_t blocks_s       = (m_num_patches + threads_s - 1) / threads_s;
    const uint32_t threads_f      = 256;
    const uint32_t blocks_f       = (m_num_faces + threads_f - 1) / threads_f;

    // Compute max patch size
    max_patch_size = 0;
    hipcub::DeviceReduce::Max(m_d_cub_temp_storage_max,
                             m_cub_max_bytes,
                             m_d_patches_size,
                             m_d_max_patch_size,
                             m_num_patches);
    CUDA_ERROR(hipMemcpy(&max_patch_size,
                          m_d_max_patch_size,
                          sizeof(uint32_t),
                          hipMemcpyDeviceToHost));

    hipcub::DeviceScan::InclusiveSum(m_d_cub_temp_storage_scan,
                                    m_cub_scan_bytes,
                                    m_d_patches_size,
                                    m_d_patches_offset,
                                    m_num_patches);
    memset<<<blocks_s, threads_s>>>(
        m_d_patches_size, 0u, m_num_patches);

    construct_patches_compressed<<<blocks_f, threads_f>>>(m_num_faces,
                                                          m_d_face_patch,
                                                          m_num_patches,
                                                          m_d_patches_offset,
                                                          m_d_patches_size,
                                                          m_d_patches_val);

    return max_patch_size;
}
}  // namespace rxmesh