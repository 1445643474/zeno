#include "hip/hip_runtime.h"
#include "FastCloth.cuh"
#include "TopoUtils.hpp"
#include "collision_energy/vertex_face_sqrt_collision.hpp"
#include "zensim/Logger.hpp"
#include "zensim/geometry/Distance.hpp"
#include "zensim/geometry/SpatialQuery.hpp"
#include <zeno/core/INode.h>
#include <zeno/types/ListObject.h>
#include <zeno/utils/log.h>
#include <zeno/zeno.h>

namespace zeno {

void FastClothSystem::initialStepping(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    /// @brief Xinit
    pol(zs::range(numDofs), [vtemp = proxy<space>({}, vtemp), D = D] ZS_LAMBDA(int i) mutable {
        auto xk = vtemp.pack(dim_c<3>, "xn", i);
        auto ykp1 = vtemp.pack(dim_c<3>, "yn", i);
        auto diff = ykp1 - xk;
        T coeff = 1;
        if (auto len2 = diff.l2NormSqr(); len2 > limits<T>::epsilon() * 10)
            coeff = zs::min(D / zs::sqrt(len2), (T)1);
        vtemp.tuple(dim_c<3>, "xinit", i) = xk + coeff * diff;
    });
}

void FastClothSystem::findConstraints(zs::CudaExecutionPolicy &pol, T dHat, const zs::SmallString &tag) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    // zs::CppTimer timer;
    if (enableContact) {
        nPP.setVal(0);
        if (enableContactSelf) {
            auto pBvs = retrieve_bounding_volumes(pol, vtemp, tag, svInds, zs::wrapv<1>{}, 0);

            /// bvh
            if constexpr (s_enableProfile)
                timer.tick();

            svBvh.refit(pol, pBvs);

            if constexpr (s_enableProfile) {
                timer.tock();
                auxTime[0] += timer.elapsed();
            }

            /// sh
            if constexpr (s_enableProfile)
                timer.tick();

            // svSh.build(pol, LRef, pBvs);

            if constexpr (s_enableProfile) {
                timer.tock();
                auxTime[2] += timer.elapsed();
            }

            /// @note all cloth edge lower-bound constraints inheritly included
            findCollisionConstraints(pol, dHat, false);
        }
        if (hasBoundary()) {
            auto pBvs = retrieve_bounding_volumes(pol, vtemp, tag, *coPoints, zs::wrapv<1>{}, coOffset);

            /// bvh
            if constexpr (s_enableProfile)
                timer.tick();

            bouSvBvh.refit(pol, pBvs);

            if constexpr (s_enableProfile) {
                timer.tock();
                auxTime[0] += timer.elapsed();
            }

            /// sh
            if constexpr (s_enableProfile)
                timer.tick();

            // bouSvSh.build(pol, LRef, pBvs);

            if constexpr (s_enableProfile) {
                timer.tock();
                auxTime[2] += timer.elapsed();
            }
            findCollisionConstraints(pol, dHat, true);
        }
    }
    /// @note check upper-bound constraints for cloth edges
    nE.setVal(0);
    for (auto &primHandle : prims) {
        if (primHandle.isBoundary())
            continue;
        auto &ses = primHandle.getSurfEdges();
        pol(Collapse{ses.size()},
            [ses = proxy<space>({}, ses), vtemp = proxy<space>({}, vtemp), E = proxy<space>(E), nE = proxy<space>(nE),
             threshold = L * L - epsSlack, vOffset = primHandle.vOffset, tag] __device__(int sei) mutable {
                const auto vij = ses.pack(dim_c<2>, "inds", sei).reinterpret_bits(int_c) + vOffset;
                const auto &vi = vij[0];
                const auto &vj = vij[1];
                auto pi = vtemp.pack(dim_c<3>, tag, vi);
                auto pj = vtemp.pack(dim_c<3>, tag, vj);
                if (auto d2 = dist2_pp(pi, pj); d2 >= threshold) {
                    auto no = atomic_add(exec_cuda, &nE[0], 1);
                    E[no] = vij;
                }
            });
    }
    std::tie(npp, ne) = getConstraintCnt();
}

#define PROFILE_CD 0

void FastClothSystem::findCollisionConstraints(zs::CudaExecutionPolicy &pol, T dHat, bool withBoundary) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    pol.profile(PROFILE_CD);
    /// pt
    if constexpr (s_enableProfile)
        timer.tick();

    const auto &svbvh = withBoundary ? bouSvBvh : svBvh;
    pol(Collapse{svInds.size()},
        [svInds = proxy<space>({}, svInds), eles = proxy<space>({}, withBoundary ? *coPoints : svInds),
         eTab = proxy<space>(eTab), vtemp = proxy<space>({}, vtemp), bvh = proxy<space>(svbvh), PP = proxy<space>(PP),
         nPP = proxy<space>(nPP), dHat2 = dHat * dHat, thickness = dHat, voffset = withBoundary ? coOffset : 0,
         withBoundary] __device__(int i) mutable {
            auto vi = reinterpret_bits<int>(svInds("inds", i));
            auto pi = vtemp.pack(dim_c<3>, "xn", vi);
            auto bv = bv_t{get_bounding_box(pi - thickness, pi + thickness)};
#if 0
            auto f = [&](int svI) {
                auto vj = reinterpret_bits<int>(eles("inds", svI)) + voffset;
                if ((!withBoundary) && (vi >= vj))
                    return;
                auto pj = vtemp.pack(dim_c<3>, "xn", vj);                  
                // skip edges for point-point lower-bound constraints 
                if (!withBoundary && (eTab.single_query(ivec2 {vi, vj}) >= 0 || eTab.single_query(ivec2 {vj, vi}) >= 0))
                    return; 
                if (auto d2 = dist2_pp(pi, pj); d2 <= dHat2) {
                    auto no = atomic_add(exec_cuda, &nPP[0], 1);
                    PP[no] = pair_t{vi, vj};
                }
            };
            bvh.iter_neighbors(bv, f);
#else
            {
                const auto &lbvh = bvh;
                using bvh_t = RM_CVREF_T(lbvh);
                using index_t = typename bvh_t::index_t;
                index_t node = 0;
                while (node != -1 && node != lbvh._numNodes) {
                    index_t level = lbvh._levels[node];
                    // level and node are always in sync
                    for (; level; --level, ++node)
                        if (!overlaps(lbvh.getNodeBV(node), bv))
                            break;
                    // leaf node check
                    if (level == 0) {
                        if (overlaps(lbvh.getNodeBV(node), bv)) {
                            int svI = lbvh._auxIndices[node];
                            {
                                auto vj = reinterpret_bits<int>(eles("inds", svI)) + voffset;
                                if ((!withBoundary) && (vi >= vj))
                                    goto NEXT;
                                auto pj = vtemp.pack(dim_c<3>, "xn", vj);
                                // skip edges for point-point lower-bound constraints
                                if (!withBoundary && (eTab.query(ivec2{vi, vj}) >= 0 || eTab.query(ivec2{vj, vi}) >= 0))
                                    goto NEXT;
                                if (auto d2 = dist2_pp(pi, pj); d2 <= dHat2) {
                                    auto no = atomic_add(exec_cuda, &nPP[0], 1);
                                    PP[no] = pair_t{vi, vj};
                                }
                            }
                        }
                        NEXT:
                        node++;
                    } else // separate at internal nodes
                        node = lbvh._auxIndices[node];
                }
            }
#endif
        });

    if constexpr (s_enableProfile) {
        timer.tock();
        auxTime[1] += timer.elapsed();
    }

    /// sh
    if constexpr (s_enableProfile)
        timer.tick();

    const auto &sh = withBoundary ? bouSvSh : svSh;
    pol(Collapse{svInds.size()},
        [svInds = proxy<space>({}, svInds), eles = proxy<space>({}, withBoundary ? *coPoints : svInds),
         eTab = proxy<space>(eTab), vtemp = proxy<space>({}, vtemp), sh = proxy<space>(sh), PP = proxy<space>(PP),
         nPP = proxy<space>(nPP), dHat2 = dHat * dHat, thickness = dHat, voffset = withBoundary ? coOffset : 0,
         withBoundary] __device__(int i) mutable {
            auto vi = reinterpret_bits<int>(svInds("inds", i));
            auto pi = vtemp.pack(dim_c<3>, "xn", vi);
            auto bv = bv_t{get_bounding_box(pi - thickness, pi + thickness)};
            auto f = [&](int svI) {
                auto vj = reinterpret_bits<int>(eles("inds", svI)) + voffset;
                if ((!withBoundary) && (vi >= vj))
                    return;
                auto pj = vtemp.pack(dim_c<3>, "xn", vj);
                // skip edges for point-point lower-bound constraints
                if (!withBoundary && (eTab.single_query(ivec2{vi, vj}) >= 0 || eTab.single_query(ivec2{vj, vi}) >= 0))
                    return;
                if (auto d2 = dist2_pp(pi, pj); d2 <= dHat2) {
                    // auto no = atomic_add(exec_cuda, &nPP[0], 1);
                    // PP[no] = pair_t{vi, vj};
                }
            };
            sh.iter_neighbors(bv, f);
        });

    if constexpr (s_enableProfile) {
        timer.tock();
        auxTime[3] += timer.elapsed();
    }
    pol.profile(false);
}

bool FastClothSystem::collisionStep(zs::CudaExecutionPolicy &pol, bool enableHardPhase) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    std::tie(npp, ne) = getConstraintCnt();
    fmt::print("collision stepping [pp, edge constraints]: {}, {}\n", npp, ne);

    ///
    /// @brief soft phase for constraints
    ///
    pol(range(numDofs), [vtemp = proxy<space>({}, vtemp)] __device__(int i) mutable {
        auto xinit = vtemp.pack(dim_c<3>, "xinit", i);
#pragma unroll 3
        for (int d = 0; d < 3; ++d) {
            vtemp("xn", d, i) = xinit(d); // soft phase optimization starts from xinit
        }
    });
    for (int l = 0; l != ISoft; ++l) {
        softPhase(pol);
    }

    ///
    /// @brief check whether constraints satisfied
    ///
    if (constraintSatisfied(pol))
    {
        fmt::print(fg(fmt::color::yellow),"\tsoft phase finished successfully!\n"); 
        return true;
    }
    fmt::print(fg(fmt::color::red),"\tsoft phase failed!\n"); 
    if (!enableHardPhase)
        return false;

    ///
    /// @brief hard phase for constraints
    ///
    fmt::print(fg(fmt::color::light_golden_rod_yellow), "entering hard phase.\n");
    /// @note start from collision-free state x^k
    pol(zs::range(numDofs), [vtemp = proxy<space>({}, vtemp)] ZS_LAMBDA(int i) mutable {
        vtemp.tuple(dim_c<3>, "xn", i) = vtemp.pack(dim_c<3>, "xk", i);
    });
    for (int l = 0; l != IHard; ++l) {
        /// @note "xk" will be used for backtracking in hardphase
        hardPhase(pol);
    }

    return constraintSatisfied(pol, false);
}
void FastClothSystem::softPhase(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    T descentStepsize = 0.1f; 
    /// @note shape matching
    pol(range(coOffset), [vtemp = proxy<space>({}, vtemp)] __device__(int i) mutable {
        auto xinit = vtemp.pack(dim_c<3>, "xinit", i);
        auto xn = vtemp.pack(dim_c<3>, "xn", i);
#pragma unroll 3
        for (int d = 0; d < 3; ++d) {
            vtemp("dir", d, i) = 2.0f * (xinit(d) - xn(d)); // minus grad of ||x-xinit||^2
        }
    });
    /// @note constraints
    pol(range(npp), [vtemp = proxy<space>({}, vtemp), PP = proxy<space>(PP), rho = rho, dHat2 = dHat * dHat] __device__(int i) mutable {
        auto pp = PP[i];
        auto x0 = vtemp.pack(dim_c<3>, "xn", pp[0]); 
        auto x1 = vtemp.pack(dim_c<3>, "xn", pp[1]); 
        // ||v0 - v1||^2 >= (B + Bt)^2 + epsSlack 
        // c(x) = ||v0 - v1||^2 - (B + Bt)^2
        if ((x0 - x1).l2NormSqr() >= dHat2)
            return; 
        auto grad0 = - rho * (T)2.0 * (x0 - x1);
#pragma unroll 3
        for (int d = 0; d < 3; d++) {
            atomic_add(exec_cuda, &vtemp("dir", d, pp[0]), -grad0(d)); 
            atomic_add(exec_cuda, &vtemp("dir", d, pp[1]), grad0(d)); 
        } 
    }); 

    pol(range(ne), [vtemp = proxy<space>({}, vtemp), E = proxy<space>(E), rho = rho, 
        maxLen2 = L * L - epsSlack] __device__(int i) mutable {
        auto e = E[i];
        auto x0 = vtemp.pack(dim_c<3>, "xn", e[0]); 
        auto x1 = vtemp.pack(dim_c<3>, "xn", e[1]); 
        // ||v0 - v1||^2 <= L^2 - epsSlack 
        // i.e. L^2 - ||v0 - v1||^2 >= epsSlack
        // c(x) = L^2 - ||v0 - v1||^2
        if ((x0 - x1).l2NormSqr() <= maxLen2)
            return; 
        auto grad0 = rho * (T)2.0 * (x0 - x1);
#pragma unroll 3
        for (int d = 0; d < 3; d++) {
            atomic_add(exec_cuda, &vtemp("dir", d, e[0]), -grad0(d)); 
            atomic_add(exec_cuda, &vtemp("dir", d, e[1]), grad0(d)); 
        }
    });
    pol(range(coOffset), [vtemp = proxy<space>({}, vtemp), 
            descentStepsize] __device__(int i) mutable {
        auto dir = vtemp.pack(dim_c<3>, "dir", i);
        auto xn = vtemp.pack(dim_c<3>, "xn", i); 
#pragma unroll 3
        for (int d = 0; d < 3; ++d) {
            atomic_add(exec_cuda, &vtemp("xn", d, i), descentStepsize * dir(d));
        }
    });
}
void FastClothSystem::hardPhase(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    /// @note shape matching (reset included)
    pol(range(numDofs), [vtemp = proxy<space>({}, vtemp), coOffset = coOffset] __device__(int i) mutable {
        auto xinit = vtemp.pack(dim_c<3>, "xinit", i);
        auto xn = vtemp.pack(dim_c<3>, "xn", i);
#pragma unroll 3
        for (int d = 0; d < 3; ++d)
            vtemp("dir", d, i) = 2.0f * (xinit(d) - xn(d));
    });
    /// @note constraints
    pol(range(npp), [vtemp = proxy<space>({}, vtemp), PP = proxy<space>(PP), mu = mu,
                     Btot2 = (B + Btight) * (B + Btight), eps = epsSlack, dHat2 = dHat * dHat, 
                     a2 = a2, a3 = a3, coOffset = coOffset] __device__(int i) mutable {
        auto pp = PP[i];
        auto x0 = vtemp.pack(dim_c<3>, "xn", pp[0]); 
        auto x1 = vtemp.pack(dim_c<3>, "xn", pp[1]); 
        if ((x0 - x1).l2NormSqr() >= dHat2)
            return; 
#if 0
        zs::vec<T, 3> vs[2] = {x0, x1};
        const auto &a = vs[0];
        const auto &b = vs[1];
        const auto t2 = a[0] * 2;
        const auto t3 = a[1] * 2;
        const auto t4 = a[2] * 2;
        const auto t5 = b[0] * 2;
        const auto t6 = b[1] * 2;
        const auto t7 = b[2] * 2;

        auto t8 = -Btot2;
        auto t9 = -b[0];
        auto t11 = -b[1];
        auto t13 = -b[2];
        auto t15 = 1 / eps;
        auto t10 = -t5;
        auto t12 = -t6;
        auto t14 = -t7;
        auto t16 = t15 * t15;
        auto t17 = a[0] + t9;
        auto t18 = a[1] + t11;
        auto t19 = a[2] + t13;
        auto t20 = t2 + t10;
        auto t21 = t3 + t12;
        auto t22 = t4 + t14;
        auto t23 = t17 * t17;
        auto t24 = t18 * t18;
        auto t25 = t19 * t10;
        auto t26 = t8 + t23 + t24 + t25;
        auto t27 = t26 * t26;
        auto t28 = t26 * t26 * t26;
        auto t32 = t15 * t20 * t26 * 2;
        auto t33 = t15 * t21 * t26 * 2;
        auto t34 = t15 * t22 * t26 * 2;
        auto t29 = t15 * t27;
        auto t30 = t16 * t28;
        auto t35 = t16 * t20 * t27 * 3;
        auto t36 = t16 * t21 * t27 * 3;
        auto t37 = t16 * t22 * t27 * 3;
        auto t31 = -t30;
        auto t38 = -t35;
        auto t39 = -t36;
        auto t40 = -t37;
        auto t41 = t20 + t32 + t38;
        auto t42 = t21 + t33 + t39;
        auto t43 = t22 + t34 + t40;
        auto t44 = t26 + t29 + t31;
        auto t45 = 1 / t44;
        auto t46 = mu * t41 * t45;
        auto t47 = mu * t42 * t45;
        auto t48 = mu * t43 * t45;
        auto grad = zs::vec<T, 6>{-t46, -t47, -t48, t46, t47, t48};
#pragma unroll 3
        for (int d = 0; d < 3; ++d) {
            atomic_add(exec_cuda, &vtemp("dir", d, pp[0]), grad(d));
            atomic_add(exec_cuda, &vtemp("dir", d, pp[1]), grad(3 + d));
        }
#else 
        auto cons = (x0 - x1).l2NormSqr() - Btot2; 
        if (cons >= eps)
            return; 
        auto f = cons * (cons * (cons * a3 + a2) + 1.0f); 
        auto grad = mu * (cons * (cons * 3.0f * a3 + 2.0f * a2) + 1.0f) / f * 2.0f * (x0 - x1); // negative grad for x0
#pragma unroll 3
        for (int d = 0; d < 3; ++d) {
            if (pp[0] < coOffset)
                atomic_add(exec_cuda, &vtemp("dir", d, pp[0]), grad(d));
            if (pp[1] < coOffset)
                atomic_add(exec_cuda, &vtemp("dir", d, pp[1]), -grad(d));
        }
#endif 
    });
    pol(range(ne), [vtemp = proxy<space>({}, vtemp), E = proxy<space>(E), mu = mu, L2 = L * L,
                    eps = epsSlack, maxLen2 = L * L - epsSlack, a2 = a2, a3 = a3, coOffset = coOffset] __device__(int i) mutable {
        auto e = E[i];
        auto x0 = vtemp.pack(dim_c<3>, "xn", e[0]); 
        auto x1 = vtemp.pack(dim_c<3>, "xn", e[1]); 
        if ((x0 - x1).l2NormSqr() <= maxLen2)
            return; 
#if 0
        zs::vec<T, 3> vs[2] = {x0, x1};
        const auto &a = vs[0];
        const auto &b = vs[1];
        const auto t2 = a[0] * 2;
        const auto t3 = a[1] * 2;
        const auto t4 = a[2] * 2;
        const auto t5 = b[0] * 2;
        const auto t6 = b[1] * 2;
        const auto t7 = b[2] * 2;
        auto t8 = -L2;
        auto t12 = -b[0];
        auto t14 = -b[1];
        auto t16 = -b[2];
        auto t18 = 1 / eps;
        auto t9 = -t2;
        auto t10 = -t3;
        auto t11 = -t4;
        auto t13 = -t5;
        auto t15 = -t6;
        auto t17 = -t7;
        auto t19 = t18 * t18;
        auto t20 = a[0] + t12;
        auto t21 = a[1] + t14;
        auto t22 = a[2] + t16;
        auto t23 = t2 + t13;
        auto t24 = t3 + t15;
        auto t25 = t4 + t17;
        auto t26 = t20 * t20;
        auto t27 = t21 * t21;
        auto t28 = t22 * t22;
        auto t29 = -t26;
        auto t30 = -t27;
        auto t31 = -t28;
        auto t32 = t8 + t26 + t27 + t28;
        auto t33 = t32 * t32;
        auto t34 = t32 * t32 * t32;
        auto t37 = t18 * t23 * t32 * 2;
        auto t38 = t18 * t24 * t32 * 2;
        auto t39 = t18 * t25 * t32 * 2;
        auto t35 = t18 * t33;
        auto t36 = t19 * t34;
        auto t40 = t19 * t23 * t33 * 3;
        auto t41 = t19 * t24 * t33 * 3;
        auto t42 = t19 * t25 * t33 * 3;
        auto t43 = t5 + t9 + t37 + t40;
        auto t44 = t6 + t10 + t38 + t41;
        auto t45 = t7 + t11 + t39 + t42;
        auto t46 = L2 + t29 + t30 + t31 + t35 + t36;
        auto t47 = 1 / t46;
        auto t48 = mu * t43 * t47;
        auto t49 = mu * t44 * t47;
        auto t50 = mu * t45 * t47;
        auto grad = zs::vec<T, 6>{-t48, -t49, -t50, t48, t49, t50};
#pragma unroll 3
        for (int d = 0; d < 3; ++d) {
            atomic_add(exec_cuda, &vtemp("dir", d, e[0]), -grad(d));
            atomic_add(exec_cuda, &vtemp("dir", d, e[1]), -grad(3 + d));
        }
#else 
        auto cons = L2 - (x0 - x1).l2NormSqr(); 
        if (cons >= eps)
            return; 
        auto f = cons * (cons * (cons * a3 + a2) + 1.0f); 
        auto grad = - mu * (cons * (cons * 3.0f * a3 + 2.0f * a2) + 1.0f) / f * 2.0f * (x0 - x1); // negative grad for x0
#pragma unroll 3
        for (int d = 0; d < 3; ++d) {
            if (e[0] < coOffset)
                atomic_add(exec_cuda, &vtemp("dir", d, e[0]), grad(d));
            if (e[1] < coOffset)
                atomic_add(exec_cuda, &vtemp("dir", d, e[1]), -grad(d));
        }
#endif 
    });
    /// @brief compute appropriate step size that does not violates constraints
    auto alpha = (T)0.1;
    /// @note vertex displacement constraint. ref 4.2.2, item 3
    auto displacement = infNorm(pol); // "dir"
    // if (auto v = std::sqrt((B + Btight) * (B + Btight) - B * B) / displacement; v < alpha)
    //     alpha = v;

    pol(zs::range(numDofs), [vtemp = proxy<space>({}, vtemp)] ZS_LAMBDA(int i) mutable {
        vtemp.tuple(dim_c<3>, "xn0", i) = vtemp.pack(dim_c<3>, "xn", i);
    });
    auto E0 = constraintEnergy(pol); // "xn"
    auto c1m = armijoParam * dot(pol, "dir", "dir");
    fmt::print(fg(fmt::color::white), "c1m : {}\n", c1m);
    do {
        pol(zs::range(numDofs), [vtemp = proxy<space>({}, vtemp), alpha] ZS_LAMBDA(int i) mutable {
            vtemp.tuple(dim_c<3>, "xn", i) = vtemp.pack(dim_c<3>, "xn0", i) + alpha * vtemp.pack(dim_c<3>, "dir", i);
        });

        ///
        /// @note check c_ij(x^{l+1}). ref 4.2.2, item 1
        ///
        temp.setVal(0);
        auto B2 = B * B; 
        pol(range(npp), [vtemp = proxy<space>({}, vtemp), PP = proxy<space>(PP), mark = proxy<space>(temp),
                         threshold = (B + Btight) * (B + Btight) + epsCond, B2 = B2 + epsCond] __device__(int i) mutable { // no constraints margin here according to paper 4.2.2
            auto pp = PP[i];
            auto x0 = vtemp.pack(dim_c<3>, "xn", pp[0]);
            auto x1 = vtemp.pack(dim_c<3>, "xn", pp[1]);
            auto x0k = vtemp.pack(dim_c<3>, "xn0", pp[0]); 
            auto x1k = vtemp.pack(dim_c<3>, "xn0", pp[1]); 
            auto ek = x1k - x0k, ek1 = x1 - x0; 
            auto dir = ek1 - ek; 
            auto de2 = dir.l2NormSqr(); 
            if (de2 > 10 * limits<T>::epsilon()) // check continuous constraints 4.2.1 & 4.1
            {
                auto numerator = -ek.dot(dir); 
                auto t = numerator / de2; 
                if (t > 0 && t < 1)
                {
                    auto et = t * dir + ek;
                    if (et.l2NormSqr() < B2)
                    {
                        printf("linesearch t: %f, et.l2NormSqr: %f, threshold: %f, pp: %d, %d, last: %f, de2: %f\n", 
                            (float)t, (float)(et.l2NormSqr()), (float)threshold, pp[0], pp[1], (float)ek.l2NormSqr(), (float)de2); 
                        mark[0] = 1; 
                        return; 
                    }
                }
            } 
            if (auto d2 = dist2_pp(x0, x1); d2 < threshold)
            {
                auto dir0 = vtemp.pack(dim_c<3>, "dir", pp[0]).l2NormSqr();
                auto dir1 = vtemp.pack(dim_c<3>, "dir", pp[1]).l2NormSqr();
                printf("linesearch discrete pp, d2: %f, pp: %d, %d, dir: %f, %f \n", (float)d2, pp[0], pp[1], (float)dir0, (float)dir1); 
                mark[0] = 1;
            }
        });
#if 1
        if (temp.getVal() == 0) {
            pol(range(ne), [vtemp = proxy<space>({}, vtemp), E = proxy<space>(E), mark = proxy<space>(temp),
                            threshold = L * L - epsCond] __device__(int i) mutable { // no constraints margin here according to paper 4.2.2
                auto e = E[i];
                auto x0 = vtemp.pack(dim_c<3>, "xn", e[0]);
                auto x1 = vtemp.pack(dim_c<3>, "xn", e[1]);
                auto xk0 = vtemp.pack(dim_c<3>, "xn0", e[0]);
                auto xk1 = vtemp.pack(dim_c<3>, "xn0", e[1]);
                if (auto d2 = dist2_pp(x0, x1); d2 > threshold)
                {
                    auto dir0 = vtemp.pack(dim_c<3>, "dir", e[0]).l2NormSqr(); 
                    auto dir1 = vtemp.pack(dim_c<3>, "dir", e[1]).l2NormSqr(); 
                    auto dk2 = dist2_pp(xk0, xk1); 
#if 0
                    printf("linesearch ee, ee: %d, %d, d2: %f, dir: %f, %f, dk2: %f, threshold: %f\n", 
                        e[0], e[1], (float)d2, (float)dir0, (float)dir1, (float)dk2, (float)threshold); 
#endif 
                    mark[0] = 1;
                }
            });
        }
#endif 

        /// @brief backtracking if discrete constraints violated
        if (temp.getVal() == 1) {
            if (alpha < 1e-15)
            {
                throw std::runtime_error("stepsize too tiny in hard phase collision solve"); 
            }
            alpha /= 2.0f;
            fmt::print("\t[back-tracing] alpha: {} constraint not satisfied\n", alpha); 
            continue;
        }
        fmt::print("[back-tracing] acceptable alpha: {}\n", alpha); 

        ///
        /// @note objective decreases adequately. ref 4.2.2, item 2
        ///
        auto E = constraintEnergy(pol);
        break;  // debug: remove energy linesearch
        if (E <= E0 + alpha * c1m)
        {
            fmt::print("\t[back-tracing] alpha: {} line search finished!\n", alpha);
            break;
        }
        alpha /= 2;
    } while (true);
    fmt::print(fg(fmt::color::antique_white), "alpha_l^hard: {}\n", alpha);
}

bool FastClothSystem::constraintSatisfied(zs::CudaExecutionPolicy &pol, bool hasEps) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    temp.setVal(0);
    auto threshold = (B + Btight) * (B + Btight); 
    auto B2 = B * B; 
    if (hasEps)
    {
        B2 += epsCond; 
        threshold += epsCond; 
    }
    pol(range(npp), [vtemp = proxy<space>({}, vtemp), PP = proxy<space>(PP), mark = proxy<space>(temp),
                     threshold, B2, hasEps] __device__(int i) mutable { // epsCond: paper 4.2.2
        auto pp = PP[i];
        auto x0 = vtemp.pack(dim_c<3>, "xn", pp[0]);
        auto x1 = vtemp.pack(dim_c<3>, "xn", pp[1]);
        auto x0k = vtemp.pack(dim_c<3>, "xk", pp[0]); 
        auto x1k = vtemp.pack(dim_c<3>, "xk", pp[1]); 
        auto ek = x1k - x0k, ek1 = x1 - x0; 
        auto dir = ek1 - ek; 
        auto de2 = dir.l2NormSqr(); 
        if (hasEps && de2 > limits<T>::epsilon()) // check continuous constraints 4.2.1 & 4.1
        {
            auto numerator = -ek.dot(dir); 
            auto t = numerator / de2; 
            if (t > 0 && t < 1)
            {
                auto et = t * dir + ek;
                if (et.l2NormSqr() < B2)
                {
#if 0
                    printf("t: %f, et.l2NormSqr: %f, threshold: %f\n", 
                        (float)t, (float)(et.l2NormSqr()), (float)threshold); 
#endif 
                    mark[0] = 1; 
                    return; 
                }
            }
        } 
        if (auto d2 = dist2_pp(x0, x1); d2 < threshold)
            mark[0] = 1;
    });
    threshold = L * L; 
    if (hasEps)
        threshold -= epsCond; 
    if (temp.getVal() == 0) {
        pol(range(ne), [vtemp = proxy<space>({}, vtemp), E = proxy<space>(E), mark = proxy<space>(temp),
                        threshold] __device__(int i) mutable { // epsCond: paper 4.2.2
            auto e = E[i];
            auto x0 = vtemp.pack(dim_c<3>, "xn", e[0]);
            auto x1 = vtemp.pack(dim_c<3>, "xn", e[1]);
            if (auto d2 = dist2_pp(x0, x1); d2 > threshold)
                mark[0] = 1;
        });
    }
    // all constraints satisfied if temp.getVal() == 0
    return temp.getVal() == 0;
}

typename FastClothSystem::T FastClothSystem::constraintEnergy(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    temp.setVal(0);
    pol(range(numDofs),
        [vtemp = proxy<space>({}, vtemp), energy = proxy<space>(temp), n = numDofs] __device__(int i) mutable {
            auto xinit = vtemp.pack(dim_c<3>, "xinit", i);
            auto xn = vtemp.pack(dim_c<3>, "xn", i);
            reduce_to(i, n, (xinit - xn).l2NormSqr(), energy[0]);
        });
    pol(range(npp),
        [vtemp = proxy<space>({}, vtemp), PP = proxy<space>(PP), energy = proxy<space>(temp), n = npp, mu = mu,
         Btot2 = (B + Btight) * (B + Btight), eps = epsSlack, a3 = a3, a2 = a2] __device__(int i) mutable {
            auto pp = PP[i];
            zs::vec<T, 3> vs[2] = {vtemp.pack(dim_c<3>, "xn", pp[0]), vtemp.pack(dim_c<3>, "xn", pp[1])};
            T cij = (vs[1] - vs[0]).l2NormSqr() - Btot2;
            T f = eps;
            if (cij <= 0)
                printf("\n\n\nthis should not happen! pp constraint <%d, %d> cij: %f\n", (int)pp[0], (int)pp[1], cij);
            if (cij <= eps) {
                auto x2 = cij * cij;
                f = a3 * x2 * cij + a2 * x2 + cij;
            }
            T E = -mu * zs::log(f);
            reduce_to(i, n, E, energy[0]);
        });
    pol(range(ne), [vtemp = proxy<space>({}, vtemp), E = proxy<space>(E), energy = proxy<space>(temp), n = ne, mu = mu,
                    L2 = L * L, eps = epsSlack, a3 = a3, a2 = a2] __device__(int i) mutable {
        auto e = E[i];
        zs::vec<T, 3> vs[2] = {vtemp.pack(dim_c<3>, "xn", e[0]), vtemp.pack(dim_c<3>, "xn", e[1])};
        T cij = L2 - (vs[1] - vs[0]).l2NormSqr();
        T f = eps;
        if (cij <= 0)
            printf("\n\n\nthis should not happen! edge constraint <%d, %d> cij: %f\n", (int)e[0], (int)e[1], cij);
        if (cij <= eps) {
            auto x2 = cij * cij;
            f = a3 * x2 * cij + a2 * x2 + cij;
        }
        T E = -mu * zs::log(f);
        reduce_to(i, n, E, energy[0]);
    });
    return temp.getVal();
}

#if 0
void FastClothSystem::computeConstraintGradients(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    auto [npp, ne] = getConstraintCnt();
    fmt::print("dcd broad phase [pp, edge constraints]: {}, {}", npp, ne);
    pol(range(npp),
        [vtemp = proxy<space>({}, vtemp), tempPP = proxy<space>({}, tempPP), PP = proxy<space>(PP), rho = rho, mu = mu,
         Btot2 = (B + Btight) * (B + Btight), eps = epsSlack] __device__(int i) mutable {
            auto pp = PP[i];
            zs::vec<T, 3> vs[2] = {vtemp.pack(dim_c<3>, "xn", pp[0]), vtemp.pack(dim_c<3>, "xn", pp[1])};
            const auto &a = vs[0];
            const auto &b = vs[1];
            const auto t2 = a[0] * 2;
            const auto t3 = a[1] * 2;
            const auto t4 = a[2] * 2;
            const auto t5 = b[0] * 2;
            const auto t6 = b[1] * 2;
            const auto t7 = b[2] * 2;
            {

                const auto t8 = -t5;
                const auto t9 = -t6;
                const auto t10 = -t7;
                const auto t11 = t2 + t8;
                const auto t12 = t3 + t9;
                const auto t13 = t4 + t10;
                const auto t14 = rho * t11;
                const auto t15 = rho * t12;
                const auto t16 = rho * t13;
                auto grad = zs::vec<T, 6>{-t14, -t15, -t16, t14, t15, t16};
                tempPP.tuple(dim_c<6>, "softG", i) = grad;
            }
            {
                auto t8 = -Btot2;
                auto t9 = -b[0];
                auto t11 = -b[1];
                auto t13 = -b[2];
                auto t15 = 1 / eps;
                auto t10 = -t5;
                auto t12 = -t6;
                auto t14 = -t7;
                auto t16 = t15 * t15;
                auto t17 = a[0] + t9;
                auto t18 = a[1] + t11;
                auto t19 = a[2] + t13;
                auto t20 = t2 + t10;
                auto t21 = t3 + t12;
                auto t22 = t4 + t14;
                auto t23 = t17 * t17;
                auto t24 = t18 * t18;
                auto t25 = t19 * t10;
                auto t26 = t8 + t23 + t24 + t25;
                auto t27 = t26 * t26;
                auto t28 = t26 * t26 * t26;
                auto t32 = t15 * t20 * t26 * 2;
                auto t33 = t15 * t21 * t26 * 2;
                auto t34 = t15 * t22 * t26 * 2;
                auto t29 = t15 * t27;
                auto t30 = t16 * t28;
                auto t35 = t16 * t20 * t27 * 3;
                auto t36 = t16 * t21 * t27 * 3;
                auto t37 = t16 * t22 * t27 * 3;
                auto t31 = -t30;
                auto t38 = -t35;
                auto t39 = -t36;
                auto t40 = -t37;
                auto t41 = t20 + t32 + t38;
                auto t42 = t21 + t33 + t39;
                auto t43 = t22 + t34 + t40;
                auto t44 = t26 + t29 + t31;
                auto t45 = 1 / t44;
                auto t46 = mu * t41 * t45;
                auto t47 = mu * t42 * t45;
                auto t48 = mu * t43 * t45;
                auto grad = zs::vec<T, 6>{-t46, -t47, -t48, t46, t47, t48};
                tempPP.tuple(dim_c<6>, "hardG", i) = grad;
            }
        });

    pol(range(ne), [vtemp = proxy<space>({}, vtemp), tempE = proxy<space>({}, tempE), E = proxy<space>(E), rho = rho,
                    mu = mu, L2 = L * L, eps = epsSlack] __device__(int i) mutable {
        auto e = E[i];
        zs::vec<T, 3> vs[2] = {vtemp.pack(dim_c<3>, "xn", e[0]), vtemp.pack(dim_c<3>, "xn", e[1])};
        const auto &a = vs[0];
        const auto &b = vs[1];
        const auto t2 = a[0] * 2;
        const auto t3 = a[1] * 2;
        const auto t4 = a[2] * 2;
        const auto t5 = b[0] * 2;
        const auto t6 = b[1] * 2;
        const auto t7 = b[2] * 2;
        {
            const auto t8 = -t5;
            const auto t9 = -t6;
            const auto t10 = -t7;
            const auto t11 = t2 + t8;
            const auto t12 = t3 + t9;
            const auto t13 = t4 + t10;
            const auto t14 = rho * t11;
            const auto t15 = rho * t12;
            const auto t16 = rho * t13;
            auto grad = zs::vec<T, 6>{t14, t15, t16, -t14, -t15, -t16};
            tempE.tuple(dim_c<6>, "softG", i) = grad;
        }
        {
            auto t8 = -L2;
            auto t12 = -b[0];
            auto t14 = -b[1];
            auto t16 = -b[2];
            auto t18 = 1 / eps;
            auto t9 = -t2;
            auto t10 = -t3;
            auto t11 = -t4;
            auto t13 = -t5;
            auto t15 = -t6;
            auto t17 = -t7;
            auto t19 = t18 * t18;
            auto t20 = a[0] + t12;
            auto t21 = a[1] + t14;
            auto t22 = a[2] + t16;
            auto t23 = t2 + t13;
            auto t24 = t3 + t15;
            auto t25 = t4 + t17;
            auto t26 = t20 * t20;
            auto t27 = t21 * t21;
            auto t28 = t22 * t22;
            auto t29 = -t26;
            auto t30 = -t27;
            auto t31 = -t28;
            auto t32 = t8 + t26 + t27 + t28;
            auto t33 = t32 * t32;
            auto t34 = t32 * t32 * t32;
            auto t37 = t18 * t23 * t32 * 2;
            auto t38 = t18 * t24 * t32 * 2;
            auto t39 = t18 * t25 * t32 * 2;
            auto t35 = t18 * t33;
            auto t36 = t19 * t34;
            auto t40 = t19 * t23 * t33 * 3;
            auto t41 = t19 * t24 * t33 * 3;
            auto t42 = t19 * t25 * t33 * 3;
            auto t43 = t5 + t9 + t37 + t40;
            auto t44 = t6 + t10 + t38 + t41;
            auto t45 = t7 + t11 + t39 + t42;
            auto t46 = L2 + t29 + t30 + t31 + t35 + t36;
            auto t47 = 1 / t46;
            auto t48 = mu * t43 * t47;
            auto t49 = mu * t44 * t47;
            auto t50 = mu * t45 * t47;
            auto grad = zs::vec<T, 6>{-t48, -t49, -t50, t48, t49, t50};
            tempE.tuple(dim_c<6>, "hardG", i) = grad;
        }
    });
}
#endif

} // namespace zeno