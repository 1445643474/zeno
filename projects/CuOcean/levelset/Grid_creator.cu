#include "hip/hip_runtime.h"
#include "Structures.hpp"
#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/geometry/LevelSetUtils.tpp"
#include "zensim/geometry/SparseGrid.hpp"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/zpc_tpls/fmt/color.h"
#include "zensim/zpc_tpls/fmt/format.h"

#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>

namespace zeno {

struct ZSMakeSparseGrid : INode {
    void apply() override {
        float dx = get_param<float>("dx");
        if (has_input("Dx")) {
            dx = get_input2<float>("Dx");
        }
        int nx = get_input2<int>("nx");
        int ny = get_input2<int>("ny");
        int nz = get_input2<int>("nz");

        int nbx = float(nx + 7) / 8.f;
        int nby = float(ny + 7) / 8.f;
        int nbz = float(nz + 7) / 8.f;

        size_t numExpectedBlocks = nbx * nby * nbz;

        auto zsSPG = zeno::IObject::make<ZenoSparseGrid>();
        auto &spg = zsSPG->spg;
        spg = ZenoSparseGrid::spg_t{{{"sdf", 1}}, numExpectedBlocks, zs::memsrc_e::device, 0};
        spg.scale(dx);
        spg._background = dx;

        auto pol = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;
        using ivec3 = zs::vec<int, 3>;

        pol(zs::range(numExpectedBlocks),
            [table = zs::proxy<space>(spg._table), nbx, nby, nbz] __device__(int nb) mutable {
                int i = nb / (nby * nbz);
                nb -= i * (nby * nbz);
                int j = nb / nbz;
                int k = nb - j * nbz;
                table.insert(ivec3{i * 8, j * 8, k * 8});
            });

        ivec3 sphere_c{nx / 2, ny / 2, nz / 2};
        float sphere_r = 10.f;

        auto bcnt = spg.numBlocks();
        pol(zs::range(bcnt * 512), [spgv = zs::proxy<space>(spg), sphere_c, sphere_r] __device__(int cellno) mutable {
#if 0            
			int bno = cellno / 512;
            int cno = cellno & 511;
            auto bcoord = spgv._table._activeKeys[bno];
            auto cellid = RM_CVREF_T(spgv)::local_offset_to_coord(cno);
            auto ccoord = bcoord + cellid;
#endif
			auto icoord = spgv.iCoord(cellno);
            auto dx = spgv.voxelSize()[0]; // spgv._transform(0, 0);

            float dist2c = zs::sqrt(float(zs::sqr(icoord[0] - sphere_c[0]) + zs::sqr(icoord[1] - sphere_c[1]) +
                                          zs::sqr(icoord[2] - sphere_c[2])));

            //spgv("sdf", bno, cno) = ;
            spgv("sdf", icoord) = dist2c > sphere_r ? dx : -dx;
        });

        // spg.resize(numExpectedBlocks);

        spg.append_channels(pol, {{"v", 3}});

        set_output("Grid", zsSPG);
    }
};

ZENDEFNODE(ZSMakeSparseGrid, {
                                 /* inputs: */
                                 {"Dx", {"int", "nx", "128"}, {"int", "ny", "128"}, {"int", "nz", "128"}},
                                 /* outputs: */
                                 {"Grid"},
                                 /* params: */
                                 {{"float", "dx", "1.0"}},
                                 /* category: */
                                 {"Volume"}
                             });

} // namespace zeno