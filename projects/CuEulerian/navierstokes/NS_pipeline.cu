#include "hip/hip_runtime.h"
#include "Structures.hpp"
#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/geometry/LevelSetUtils.tpp"
#include "zensim/geometry/SparseGrid.hpp"
#include "zensim/geometry/VdbLevelSet.h"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/profile/CppTimers.hpp"
#include "zensim/zpc_tpls/fmt/color.h"
#include "zensim/zpc_tpls/fmt/format.h"

#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>

#include <zeno/VDBGrid.h>

#include "../scheme.hpp"
#include "../utils.cuh"

namespace zeno {

struct ZSVDBToNavierStokesGrid : INode {
    void apply() override {
        auto vdbgrid = get_input<VDBFloatGrid>("VDB");

        auto spg = zs::convert_floatgrid_to_sparse_grid(vdbgrid->m_grid, zs::MemoryHandle{zs::memsrc_e::device, 0});
        spg.append_channels(zs::cuda_exec(), {{"v0", 3},
                                              {"v1", 3},
                                              {"p0", 1},
                                              {"p1", 1},
                                              {"div_v", 1},
                                              {"rho", 1}, // smoke density
                                              {"T", 1},   // smoke temperature
                                              {"flux", 3}});
        spg._background = 0.f;

        auto zsSPG = std::make_shared<ZenoSparseGrid>();
        zsSPG->spg = std::move(spg);
        zsSPG->setMeta("v_cur", 0);
        zsSPG->setMeta("p_cur", 0);

        set_output("NSGrid", zsSPG);
    }
};

ZENDEFNODE(ZSVDBToNavierStokesGrid, {/* inputs: */
                                     {"VDB"},
                                     /* outputs: */
                                     {"NSGrid"},
                                     /* params: */
                                     {},
                                     /* category: */
                                     {"Eulerian"}});

struct ZSNavierStokesDt : INode {
    void apply() override {
        auto NSGrid = get_input<ZenoSparseGrid>("NSGrid");
        auto rho = get_input2<float>("Density");
        auto mu = get_input2<float>("Viscosity");

        auto &spg = NSGrid->spg;
        auto block_cnt = spg.numBlocks();
        auto dx = spg.voxelSize()[0];

        auto pol = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;

        size_t cell_cnt = block_cnt * spg.block_size;
        zs::Vector<float> res{spg.get_allocator(), count_warps(cell_cnt)};
        zs::memset(zs::mem_device, res.data(), 0, sizeof(float) * count_warps(cell_cnt));

        int &v_cur = NSGrid->readMeta<int &>("v_cur");

        // maximum velocity
        pol(zs::Collapse{block_cnt, spg.block_size},
            [spgv = zs::proxy<space>(spg), res = zs::proxy<space>(res), cell_cnt,
             vSrcTag = zs::SmallString{std::string("v") + std::to_string(v_cur)}] __device__(int blockno,
                                                                                             int cellno) mutable {
                float u = spgv.value(vSrcTag, 0, blockno, cellno);
                float v = spgv.value(vSrcTag, 1, blockno, cellno);
                float w = spgv.value(vSrcTag, 2, blockno, cellno);

                size_t cellno_glb = blockno * spgv.block_size + cellno;

                float v_mag = zs::abs(u) + zs::abs(v) + zs::abs(w);

                reduce_max(cellno_glb, cell_cnt, v_mag, res[cellno_glb / 32]);
            });
        float v_max = reduce(pol, res, thrust::maximum<float>{});

        // CFL dt
        const float CFL = 0.8f;
        float dt_v = CFL * dx / (v_max + 1e-10);

        // Viscosity dt
        float nu = mu / (rho + 1e-10); // kinematic viscosity
        int dim = 3;
        float dt_nu = CFL * dx * dx / ((2.f * dim * nu) + 1e-10);

        float dt = dt_v < dt_nu ? dt_v : dt_nu;

        fmt::print(fg(fmt::color::blue_violet), "CFL time step : {} sec\n", dt);

        set_output("dt", std::make_shared<NumericObject>(dt));
    }
};

ZENDEFNODE(ZSNavierStokesDt, {/* inputs: */
                              {"NSGrid", {"float", "Density", "1.0"}, {"float", "Viscosity", "0.0"}},
                              /* outputs: */
                              {"dt"},
                              /* params: */
                              {},
                              /* category: */
                              {"Eulerian"}});

struct ZSNSAdvectDiffuse : INode {
    void apply() override {
        auto NSGrid = get_input<ZenoSparseGrid>("NSGrid");
        auto rho = get_input2<float>("Density");
        auto mu = get_input2<float>("Viscosity");
        auto dt = get_input2<float>("dt");

        auto &spg = NSGrid->spg;
        auto block_cnt = spg.numBlocks();
        auto dx = spg.voxelSize()[0];

        auto pol = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;

        int &v_cur = NSGrid->readMeta<int &>("v_cur");

        // advection
        pol(zs::range(block_cnt * spg.block_size),
            [spgv = zs::proxy<space>(spg), dx, dt, vSrcTag = zs::SmallString{std::string("v") + std::to_string(v_cur)},
             vDstTag = zs::SmallString{std::string("v") + std::to_string(v_cur ^ 1)}] __device__(int cellno) mutable {
                auto icoord = spgv.iCoord(cellno);

                for (int ch = 0; ch < 3; ++ch) {
                    int x = ch;
                    int y = (ch + 1) % 3;
                    int z = (ch + 2) % 3;

                    const int stcl = 2; // stencil point in each side
                    float u_x[2 * stcl + 1], u_y[2 * stcl + 1], u_z[2 * stcl + 1];

                    for (int i = -stcl; i <= stcl; ++i) {
                        u_x[i + stcl] = spgv.value(vSrcTag, x, icoord + zs::vec<int, 3>(i, 0, 0));
                        u_y[i + stcl] = spgv.value(vSrcTag, x, icoord + zs::vec<int, 3>(0, i, 0));
                        u_z[i + stcl] = spgv.value(vSrcTag, x, icoord + zs::vec<int, 3>(0, 0, i));
                    }

                    float u_adv = spgv.value(vSrcTag, x, icoord);
                    float v_adv = spgv.value(vSrcTag, y, icoord);
                    float w_adv = spgv.value(vSrcTag, z, icoord);

                    float adv_term = 0.f;
                    int upwind = u_adv < 0 ? 1 : -1;
                    adv_term += u_adv * scheme::HJ_WENO3(u_x[2 - upwind], u_x[2], u_x[2 + upwind], u_x[2 + 2 * upwind],
                                                         u_adv, dx);
                    upwind = v_adv < 0 ? 1 : -1;
                    adv_term += v_adv * scheme::HJ_WENO3(u_y[2 - upwind], u_y[2], u_y[2 + upwind], u_y[2 + 2 * upwind],
                                                         v_adv, dx);
                    upwind = w_adv < 0 ? 1 : -1;
                    adv_term += w_adv * scheme::HJ_WENO3(u_z[2 - upwind], u_z[2], u_z[2 + upwind], u_z[2 + 2 * upwind],
                                                         w_adv, dx);

                    spgv(vDstTag, x, icoord) = u_adv - adv_term * dt;
                }
            });
        v_cur ^= 1;

        // diffusion
        pol(zs::range(block_cnt * spg.block_size),
            [spgv = zs::proxy<space>(spg), dx, dt, rho, mu,
             vSrcTag = zs::SmallString{std::string("v") + std::to_string(v_cur)},
             vDstTag = zs::SmallString{std::string("v") + std::to_string(v_cur ^ 1)}] __device__(int cellno) mutable {
                auto icoord = spgv.iCoord(cellno);

                for (int ch = 0; ch < 3; ++ch) {
                    const int stcl = 1; // stencil point in each side
                    float u_x[2 * stcl + 1], u_y[2 * stcl + 1], u_z[2 * stcl + 1];

                    for (int i = -stcl; i <= stcl; ++i) {
                        u_x[i + stcl] = spgv.value(vSrcTag, ch, icoord + zs::vec<int, 3>(i, 0, 0));
                        u_y[i + stcl] = spgv.value(vSrcTag, ch, icoord + zs::vec<int, 3>(0, i, 0));
                        u_z[i + stcl] = spgv.value(vSrcTag, ch, icoord + zs::vec<int, 3>(0, 0, i));
                    }

                    float u_xx = scheme::central_diff_2nd(u_x[0], u_x[1], u_x[2], dx);
                    float u_yy = scheme::central_diff_2nd(u_y[0], u_y[1], u_y[2], dx);
                    float u_zz = scheme::central_diff_2nd(u_z[0], u_z[1], u_z[2], dx);

                    float diff_term = mu / rho * (u_xx + u_yy + u_zz);

                    spgv(vDstTag, ch, icoord) = u_x[1] + diff_term * dt;
                }
            });
        v_cur ^= 1;
    }
};

ZENDEFNODE(ZSNSAdvectDiffuse, {/* inputs: */
                               {"NSGrid", "dt", {"float", "Density", "1.0"}, {"float", "Viscosity", "0.0"}},
                               /* outputs: */
                               {},
                               /* params: */
                               {},
                               /* category: */
                               {"Eulerian"}});

struct ZSNSExternalForce : INode {
    void apply() override {
        auto NSGrid = get_input<ZenoSparseGrid>("NSGrid");
        auto dt = get_input2<float>("dt");
        auto force = get_input2<zeno::vec3f>("Force");

        auto &spg = NSGrid->spg;
        auto block_cnt = spg.numBlocks();

        auto pol = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;

        int &v_cur = NSGrid->readMeta<int &>("v_cur");

        // add force (accelaration)
        pol(zs::Collapse{block_cnt, spg.block_size},
            [spgv = zs::proxy<space>(spg), force, dt,
             vSrcTag = zs::SmallString{std::string("v") + std::to_string(v_cur)}] __device__(int blockno,
                                                                                             int cellno) mutable {
                for (int ch = 0; ch < 3; ++ch)
                    spgv(vSrcTag, ch, blockno, cellno) += force[ch] * dt;
            });
    }
};

ZENDEFNODE(ZSNSExternalForce, {/* inputs: */
                               {"NSGrid", "dt", {"vec3f", "Force", "0, 0, 0"}},
                               /* outputs: */
                               {},
                               /* params: */
                               {},
                               /* category: */
                               {"Eulerian"}});

struct ZSNSPressureProject : INode {
    void apply() override {
        auto NSGrid = get_input<ZenoSparseGrid>("NSGrid");
        auto rho = get_input2<float>("Density");
        auto dt = get_input2<float>("dt");
        int nIter = get_input2<int>("iterations");

        auto &spg = NSGrid->spg;
        auto block_cnt = spg.numBlocks();
        auto dx = spg.voxelSize()[0];

        auto pol = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;

        int &v_cur = NSGrid->readMeta<int &>("v_cur");

        // velocity divergence
        pol(zs::range(block_cnt * spg.block_size),
            [spgv = zs::proxy<space>(spg), dx, dt,
             vSrcTag = zs::SmallString{std::string("v") + std::to_string(v_cur)}] __device__(int cellno) mutable {
                auto icoord = spgv.iCoord(cellno);

                float u_x[2], u_y[2], u_z[2];
                for (int i = 0; i <= 1; ++i) {
                    u_x[i] = spgv.value(vSrcTag, 0, icoord + zs::vec<int, 3>(i, 0, 0));
                    u_y[i] = spgv.value(vSrcTag, 1, icoord + zs::vec<int, 3>(0, i, 0));
                    u_z[i] = spgv.value(vSrcTag, 2, icoord + zs::vec<int, 3>(0, 0, i));
                }

                float div_term = ((u_x[1] - u_x[0]) + (u_y[1] - u_y[0]) + (u_z[1] - u_z[0])) / dx;

                spgv("div_v", icoord) = div_term;
            });

        const float dxSqrOverDt = dx * dx / dt;

        int &p_cur = NSGrid->readMeta<int &>("p_cur");

        // zs::CppTimer timer;
        // timer.tick();
        // pressure Poisson equation
        for (int iter = 0; iter < nIter; ++iter) {
#if 0
            // point Jacobi iteration
            pol(zs::range(block_cnt * spg.block_size),
                [spgv = zs::proxy<space>(spg), dxSqrOverDt, rho,
                 pSrcTag = zs::SmallString{std::string("p") + std::to_string(p_cur)},
                 pDstTag =
                     zs::SmallString{std::string("p") + std::to_string(p_cur ^ 1)}] __device__(int cellno) mutable {
                    auto icoord = spgv.iCoord(cellno);

                    float div = spgv.value("div_v", icoord);

                    const int stcl = 1; // stencil point in each side
                    float p_x[2 * stcl + 1], p_y[2 * stcl + 1], p_z[2 * stcl + 1];

                    for (int i = -stcl; i <= stcl; ++i) {
                        p_x[i + stcl] = spgv.value(pSrcTag, icoord + zs::vec<int, 3>(i, 0, 0));
                        p_y[i + stcl] = spgv.value(pSrcTag, icoord + zs::vec<int, 3>(0, i, 0));
                        p_z[i + stcl] = spgv.value(pSrcTag, icoord + zs::vec<int, 3>(0, 0, i));
                    }

                    float p_this =
                        -(div * dxSqrOverDt * rho - (p_x[0] + p_x[2] + p_y[0] + p_y[2] + p_z[0] + p_z[2])) / 6.f;

                    spgv(pDstTag, icoord) = p_this;
                });
            p_cur ^= 1;
#else
            // red-black SOR iteration
            const float sor = 1.2f; // over relaxation rate

            for (int clr = 0; clr != 2; ++clr) {

                pol(zs::range(block_cnt * 32), [spgv = zs::proxy<space>(spg), dxSqrOverDt, rho, clr,
                                                sor] __device__(int tid) mutable {
                    auto blockno = tid / 32;

                    auto bcoord = spgv._table._activeKeys[blockno];
                    if ((((bcoord[0] & 8) ^ (bcoord[1] & 8) ^ (bcoord[2] & 8)) >> 3) == clr)
                        return;

                    auto tile = zs::cg::tiled_partition<32>(zs::cg::this_thread_block());
                    for (int c_clr = 0; c_clr != 2; ++c_clr) {

                        for (int cno = tile.thread_rank(); cno < 256; cno += tile.num_threads()) {
                            auto cellno = (cno << 1) | c_clr;

                            auto ccoord = spgv.local_offset_to_coord(cellno);
                            auto icoord = bcoord + ccoord;

                            float div = spgv.value("div_v", blockno, cellno);

                            const int stcl = 1; // stencil point in each side
                            float p_x[2 * stcl + 1], p_y[2 * stcl + 1], p_z[2 * stcl + 1];

                            for (int i = -stcl; i <= stcl; ++i) {
                                p_x[i + stcl] = spgv.value("p0", icoord + zs::vec<int, 3>(i, 0, 0));
                                p_y[i + stcl] = spgv.value("p0", icoord + zs::vec<int, 3>(0, i, 0));
                                p_z[i + stcl] = spgv.value("p0", icoord + zs::vec<int, 3>(0, 0, i));
                            }

                            float p_this =
                                (1.f - sor) * p_x[stcl] +
                                sor *
                                    ((p_x[0] + p_x[2] + p_y[0] + p_y[2] + p_z[0] + p_z[2]) - div * dxSqrOverDt * rho) /
                                    6.f;

                            spgv("p0", blockno, cellno) = p_this;
                        }
                    }
                });
            }
#endif
        }
        // timer.tock("jacobi/sor iterations");

        // pressure projection
        pol(zs::range(block_cnt * spg.block_size),
            [spgv = zs::proxy<space>(spg), dx, dt, rho,
             vSrcTag = zs::SmallString{std::string("v") + std::to_string(v_cur)},
             vDstTag = zs::SmallString{std::string("v") + std::to_string(v_cur ^ 1)},
             pSrcTag = zs::SmallString{std::string("p") + std::to_string(p_cur)}] __device__(int cellno) mutable {
                auto icoord = spgv.iCoord(cellno);
                float p_this = spgv.value(pSrcTag, icoord);

                for (int ch = 0; ch < 3; ++ch) {
                    float u = spgv.value(vSrcTag, ch, icoord);

                    zs::vec<int, 3> offset{0, 0, 0};
                    offset[ch] = -1;

                    float p_m = spgv.value(pSrcTag, icoord + offset);

                    u -= (p_this - p_m) / dx * dt / rho;

                    spgv(vDstTag, ch, icoord) = u;
                }
            });
        v_cur ^= 1;
    }
};

ZENDEFNODE(ZSNSPressureProject, {/* inputs: */
                                 {"NSGrid", "dt", {"int", "iterations", "10"}, {"float", "Density", "1.0"}},
                                 /* outputs: */
                                 {},
                                 /* params: */
                                 {},
                                 /* category: */
                                 {"Eulerian"}});

struct ZSNSNaiveSolidWall : INode {
    void apply() override {
        auto NSGrid = get_input<ZenoSparseGrid>("NSGrid");
        auto SolidSDF = get_input<ZenoSparseGrid>("SolidSDF");
        auto SolidVel = get_input<ZenoSparseGrid>("SolidVel");

        auto &sdf = SolidSDF->spg;
        auto &vel = SolidVel->spg;
        auto &spg = NSGrid->spg;
        auto block_cnt = spg.numBlocks();

        auto pol = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;

        int &v_cur = NSGrid->readMeta<int &>("v_cur");

        pol(zs::Collapse{block_cnt, spg.block_size},
            [spgv = zs::proxy<space>(spg), sdfv = zs::proxy<space>(sdf), velv = zs::proxy<space>(vel),
             vSrcTag = zs::SmallString{std::string("v") + std::to_string(v_cur)}] __device__(int blockno,
                                                                                             int cellno) mutable {
                auto wcoord = spgv.wCoord(blockno, cellno);
                auto solid_sdf = sdfv.wSample("sdf", wcoord);

                if (solid_sdf < 0) {
                    auto vel_s = velv.wStaggeredPack("v", wcoord);
                    auto block = spgv.block(blockno);
                    block.template tuple<3>(vSrcTag, cellno) = vel_s;
                }

                spgv("sdf", blockno, cellno) = solid_sdf;
            });
    }
};

ZENDEFNODE(ZSNSNaiveSolidWall, {/* inputs: */
                                {"NSGrid", "SolidSDF", "SolidVel"},
                                /* outputs: */
                                {},
                                /* params: */
                                {},
                                /* category: */
                                {"Eulerian"}});

struct ZSTracerAdvectDiffuse : INode {
    void compute(zs::CudaExecutionPolicy &pol, zs::SmallString tag, float diffuse, float dt, ZenoSparseGrid *NSGrid) {

        constexpr auto space = RM_CVREF_T(pol)::exec_tag::value;

        auto &spg = NSGrid->spg;
        auto block_cnt = spg.numBlocks();

        auto dx = spg.voxelSize()[0];

        int &v_cur = NSGrid->readMeta<int &>("v_cur");

        // Finite Volume Method (FVM)
        // numrtical flux of tracer
        pol(zs::Collapse{block_cnt, spg.block_size},
            [spgv = zs::proxy<space>(spg), diffuse, dx, tag,
             vSrcTag = zs::SmallString{std::string("v") + std::to_string(v_cur)}] __device__(int blockno,
                                                                                             int cellno) mutable {
                auto icoord = spgv.iCoord(blockno, cellno);

                const int stcl = 2; // stencil point in each side
                float trc[3][2 * stcl];

                // | i - 2 | i - 1 | i | i + 1 |
                for (int i = -stcl; i < stcl; ++i) {
                    trc[0][i + stcl] = spgv.value(tag, icoord + zs::vec<int, 3>(i, 0, 0));
                    trc[1][i + stcl] = spgv.value(tag, icoord + zs::vec<int, 3>(0, i, 0));
                    trc[2][i + stcl] = spgv.value(tag, icoord + zs::vec<int, 3>(0, 0, i));
                }

                float u_adv[3];
                for (int ch = 0; ch < 3; ++ch)
                    u_adv[ch] = spgv.value(vSrcTag, ch, icoord);

                // approximate value at i - 1/2
                float flux[3];
                for (int ch = 0; ch < 3; ++ch) {
                    // convection flux
                    if (u_adv[ch] < 0)
                        flux[ch] = u_adv[ch] * scheme::TVD_MUSCL3(trc[ch][1], trc[ch][2], trc[ch][3]);
                    else
                        flux[ch] = u_adv[ch] * scheme::TVD_MUSCL3(trc[ch][2], trc[ch][1], trc[ch][0]);

                    // diffusion flux
                    flux[ch] -= diffuse * (trc[ch][2] - trc[ch][1]) / dx;
                }

                for (int ch = 0; ch < 3; ++ch) {
                    spgv("flux", ch, blockno, cellno) = flux[ch];
                }
            });

        // time integration of tracer
        pol(zs::Collapse{block_cnt, spg.block_size},
            [spgv = zs::proxy<space>(spg), dx, dt, tag] __device__(int blockno, int cellno) mutable {
                auto icoord = spgv.iCoord(blockno, cellno);

                float flux[3][2];
                for (int ch = 0; ch < 3; ++ch) {
                    zs::vec<int, 3> offset{0, 0, 0};
                    offset[ch] = 1;

                    flux[ch][0] = spgv.value("flux", ch, icoord);
                    flux[ch][1] = spgv.value("flux", ch, icoord + offset);
                }

                float dtrc = 0;
                for (int ch = 0; ch < 3; ++ch) {
                    dtrc += (flux[ch][0] - flux[ch][1]) / dx;
                }
                dtrc *= dt;

                spgv(tag, blockno, cellno) += dtrc;
            });
    }

    void apply() override {
        auto NSGrid = get_input<ZenoSparseGrid>("NSGrid");
        auto diffuse = get_input2<float>("Diffusion");
        auto dt = get_input2<float>("dt");

        auto pol = zs::cuda_exec();
        ///
        if (get_input2<bool>("Density"))
            compute(pol, "rho", diffuse, dt, NSGrid.get());
        if (get_input2<bool>("Temperature"))
            compute(pol, "T", diffuse, dt, NSGrid.get());
    }
};

ZENDEFNODE(ZSTracerAdvectDiffuse,
           {/* inputs: */
            {"NSGrid", "dt", {"float", "Diffusion", "0.0"}, {"bool", "Density", "1"}, {"bool", "Temperature", "1"}},
            /* outputs: */
            {},
            /* params: */
            {},
            /* category: */
            {"Eulerian"}});

struct ZSTracerEmission : INode {
    void compute(zs::CudaExecutionPolicy &pol, zs::SmallString tag, ZenoSparseGrid *NSGrid, ZenoSparseGrid *EmitSDF) {

        constexpr auto space = RM_CVREF_T(pol)::exec_tag::value;

        auto &spg = NSGrid->spg;
        auto &sdf = EmitSDF->spg;

        auto block_cnt = spg.numBlocks();

        auto dx = spg.voxelSize()[0];

        pol(zs::Collapse{block_cnt, spg.block_size}, [spgv = zs::proxy<space>(spg), sdfv = zs::proxy<space>(sdf), dx,
                                                      tag] __device__(int blockno, int cellno) mutable {
            auto wcoord = spgv.wCoord(blockno, cellno);
            auto emit_sdf = sdfv.wSample("sdf", wcoord);

            if (emit_sdf <= 1.5f * dx) {
                // fix me: naive emission
                spgv(tag, blockno, cellno) = 1.0;
            }
        });
    }

    void apply() override {
        auto NSGrid = get_input<ZenoSparseGrid>("NSGrid");
        auto EmitSDF = get_input<ZenoSparseGrid>("EmitterSDF");

        auto pol = zs::cuda_exec();

        if (get_input2<bool>("Density"))
            compute(pol, "rho", NSGrid.get(), EmitSDF.get());
        if (get_input2<bool>("Temperature"))
            compute(pol, "T", NSGrid.get(), EmitSDF.get());
    }
};

ZENDEFNODE(ZSTracerEmission, {/* inputs: */
                              {"NSGrid", "EmitterSDF", {"bool", "Density", "1"}, {"bool", "Temperature", "1"}},
                              /* outputs: */
                              {},
                              /* params: */
                              {},
                              /* category: */
                              {"Eulerian"}});

struct ZSSmokeBuoyancy : INode {
    void apply() override {
        auto NSGrid = get_input<ZenoSparseGrid>("NSGrid");
        auto dt = get_input2<float>("dt");
        auto gravity = get_input2<zeno::vec3f>("Gravity");
        auto alpha = get_input2<float>("DensityCoef");
        auto beta = get_input2<float>("TemperatureCoef");
        auto T_amb = get_input2<float>("AmbientTemp");
        auto T_ref = get_input2<float>("ReferenceTemp");

        auto &spg = NSGrid->spg;
        auto block_cnt = spg.numBlocks();

        auto pol = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;

        int &v_cur = NSGrid->readMeta<int &>("v_cur");

        // add force (accelaration)
        pol(zs::Collapse{block_cnt, spg.block_size},
            [spgv = zs::proxy<space>(spg), dt, gravity, alpha, beta, T_amb, T_ref,
             vSrcTag = zs::SmallString{std::string("v") + std::to_string(v_cur)}] __device__(int blockno,
                                                                                             int cellno) mutable {
                auto icoord = spgv.iCoord(blockno, cellno);

                float rho_this = spgv.value("rho", blockno, cellno);
                float T_this = spgv.value("T", blockno, cellno);

                for (int ch = 0; ch < 3; ++ch) {
                    zs::vec<int, 3> offset{0, 0, 0};
                    offset[ch] = -1;

                    float rho_face = spgv.value("rho", icoord + icoord);
                    float T_face = spgv.value("T", icoord + icoord);

                    rho_face = 0.5f * (rho_this + rho_face);
                    T_face = 0.5f * (T_this + T_face);

                    float G_scale = alpha * rho_face - beta * (T_face * T_ref - T_amb);

                    spgv(vSrcTag, ch, blockno, cellno) += G_scale * gravity[ch] * dt;
                }
            });
    }
};

ZENDEFNODE(ZSSmokeBuoyancy, {/* inputs: */
                             {"NSGrid",
                              "dt",
                              {"vec3f", "Gravity", "0, -9.8, 0"},
                              {"float", "DensityCoef", "0.0"},
                              {"float", "TemperatureCoef", "0.0"},
                              {"float", "AmbientTemp", "273.0"},
                              {"float", "ReferenceTemp", "3000.0"}},
                             /* outputs: */
                             {},
                             /* params: */
                             {},
                             /* category: */
                             {"Eulerian"}});

struct ZSExtendSparseGrid : INode {

    template <typename PredT> void refit(ZenoSparseGrid *nsgridPtr, std::string tag, PredT pred) {
        using namespace zs;
        static constexpr auto space = execspace_e::cuda;
        namespace cg = ::cooperative_groups;
        auto pol = cuda_exec();
        auto &spg = nsgridPtr->getSparseGrid();
        // make sure spg.block_size % 32 == 0

        auto nbs = spg.numBlocks();
        using Ti = RM_CVREF_T(nbs);

        Vector<Ti> marks{spg.get_allocator(), nbs + 1}, offsets{spg.get_allocator(), nbs + 1};
        marks.reset(0);

        pol(range(nbs * 32), [spg = proxy<space>(spg), tagOffset = spg.getPropertyOffset(tag),
                              marks = proxy<space>(marks), pred] __device__(std::size_t i) mutable {
            auto tile = cg::tiled_partition<32>(cg::this_thread_block());
            auto bno = i / 32;
            auto cellno = tile.thread_rank();

            while (cellno < spg.block_size) {
                if (tile.ballot(pred(spg(tagOffset, bno, cellno))))
                    break;
                cellno += 32;
            }
            if (tile.thread_rank() == 0 && cellno < spg.block_size)
                marks[bno] = 1;
        });

        exclusive_scan(pol, std::begin(marks), std::end(marks), std::begin(offsets));
        auto newNbs = offsets.getVal(nbs);
        fmt::print("compacting {} blocks to {} active blocks.\n", nbs, newNbs);

        /// @brief compact active blocks
        // grid
        auto &grid = spg._grid;
        auto dstgrid = grid;
        // table
        auto &table = spg._table;
        table.reset(false);
        table._cnt.setVal(newNbs);
        auto &keys = table._activeKeys;
        auto newKeys = keys;

        pol(range(nbs * spg._table.bucket_size),
            [grid = proxy<space>(grid), dstgrid = proxy<space>(dstgrid), marks = proxy<space>(marks),
             newKeys = proxy<space>(newKeys), keys = proxy<space>(keys), table = proxy<space>(table),
             offsets = proxy<space>(offsets),
             bs_c = wrapv<RM_CVREF_T(spg)::block_size>{}] __device__(std::size_t i) mutable {
                constexpr auto block_size = decltype(bs_c)::value;
                constexpr auto bucket_size = RM_CVREF_T(table)::bucket_size;
                static_assert(block_size % bucket_size == 0, "block_size should be a multiple of bucket_size");
                auto tile = cg::tiled_partition<bucket_size>(cg::this_thread_block());
                auto bno = i / bucket_size;
                if (marks[bno] == 0)
                    return;
                auto dstBno = offsets[bno];
                // grid
                for (auto cellno = tile.thread_rank(); cellno < block_size; cellno += bucket_size) {
                    for (int chn = 0; chn != grid.numChannels(); ++chn)
                        dstgrid(chn, dstBno, cellno) = grid(chn, bno, cellno);
                }
                // table
                auto bcoord = keys[bno];
                table.tile_insert(tile, bcoord, dstBno, false); // do not enqueue key, hence set false
                if (tile.thread_rank() == 0)
                    newKeys[dstBno] = bcoord;
            });
        grid = std::move(dstgrid);
        keys = std::move(newKeys);
    }

    template <typename PredT>
    void extend(ZenoSparseGrid *nsgridPtr, std::string tag, std::size_t &nbsOffset, PredT pred) {
        using namespace zs;
        static constexpr auto space = execspace_e::cuda;
        namespace cg = ::cooperative_groups;
        auto pol = cuda_exec();
        auto &spg = nsgridPtr->getSparseGrid();
        // make sure spg.block_size % 32 == 0

        auto nbs = spg.numBlocks() - nbsOffset;
        if (nbs == 0)
            return;
        // worst case is that all candidate blocks activate all surrounding neighbors
        spg.resize(pol, nbs * 26 + nbsOffset);

        // zeno::log_info("currently {} blocks (offset {}), resizing to {}\n", nbsOffset + nbs, nbsOffset,
        //                nbs * 26 + nbsOffset);

        if (!spg._grid.hasProperty(tag))
            throw std::runtime_error(fmt::format("property [{}] not exist!", tag));

        pol(range(nbs * spg._table.bucket_size), [spg = proxy<space>(spg), tagOffset = spg.getPropertyOffset(tag),
                                                  nbsOffset, pred] __device__(std::size_t i) mutable {
            auto tile = cg::tiled_partition<RM_CVREF_T(spg._table)::bucket_size>(cg::this_thread_block());
            auto bno = i / spg._table.bucket_size + nbsOffset;
            auto cellno = tile.thread_rank();
            // searching for active voxels within this block

            while (cellno < spg.block_size) {
                if (tile.ballot(pred(spg(tagOffset, bno, cellno))))
                    break;
                cellno += spg._table.bucket_size;
            }
            if (cellno < spg.block_size) {
                auto bcoord = spg.iCoord(bno, 0);
                for (auto loc : ndrange<3>(3)) {
                    auto dir = make_vec<int>(loc) - 1;
                    // spg._table.insert(bcoord + dir * spg.side_length);
                    spg._table.tile_insert(tile, bcoord + dir * spg.side_length, RM_CVREF_T(spg._table)::sentinel_v,
                                           true);
                }
            }
        });
        // [ nbsOffset | nbsOffset + nbs | spg.numBlocks() ]
        nbsOffset += nbs;
        auto newNbs = spg.numBlocks();
        newNbs -= nbsOffset;
        if (newNbs > 0)
            zs::memset(mem_device, (void *)spg._grid.tileOffset(nbsOffset), 0,
                       (std::size_t)newNbs * spg._grid.tileBytes());

        if (tag == "sdf")
            pol(range(newNbs * spg.block_size),
                [dx = spg.voxelSize()[0], spg = proxy<space>(spg), sdfOffset = spg.getPropertyOffset("sdf"),
                 blockOffset = nbsOffset * spg.block_size] __device__(std::size_t cellno) mutable {
                    spg(sdfOffset, blockOffset + cellno) = 3 * dx;
                });
    }

    void apply() override {
        auto zsSPG = get_input<ZenoSparseGrid>("NSGrid");
        auto tag = get_input2<std::string>("Attribute");
        auto nlayers = get_input2<int>("layers");
        auto needRefit = get_input2<bool>("refit");

        std::size_t nbs = 0;
        int opt = 0;
        if (tag == "rho")
            opt = 1;
        else if (tag == "sdf")
            opt = 2;

        if (needRefit && opt != 0) {
            if (opt == 1)
                refit(zsSPG.get(), tag,
                      [] __device__(float v) -> bool { return v > zs::limits<float>::epsilon() * 10; });
            else if (opt == 2)
                refit(zsSPG.get(), tag,
                      [dx = zsSPG->getSparseGrid().voxelSize()[0]] __device__(float v) -> bool { return v < 2 * dx; });
            opt = 0;
        }

        while (nlayers-- > 0) {
            if (opt == 0)
                extend(zsSPG.get(), tag, nbs, [] __device__(float v) { return true; });
            else if (opt == 1)
                extend(zsSPG.get(), tag, nbs,
                       [] __device__(float v) -> bool { return v > zs::limits<float>::epsilon() * 10; });
            else if (opt == 2)
                extend(zsSPG.get(), tag, nbs,
                       [dx = zsSPG->getSparseGrid().voxelSize()[0]] __device__(float v) -> bool { return v < 2 * dx; });
            opt = 0; // always active since
        }

        set_output("NSGrid", zsSPG);
    }
};

ZENDEFNODE(ZSExtendSparseGrid,
           {/* inputs: */
            {"NSGrid", {"enum rho sdf", "Attribute", "rho"}, {"bool", "refit", "1"}, {"int", "layers", "2"}},
            /* outputs: */
            {"NSGrid"},
            /* params: */
            {},
            /* category: */
            {"Eulerian"}});

} // namespace zeno