#include "hip/hip_runtime.h"
#include "Structures.hpp"
#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/geometry/LevelSetUtils.tpp"
#include "zensim/geometry/SparseGrid.hpp"
#include "zensim/geometry/VdbLevelSet.h"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/profile/CppTimers.hpp"
#include "zensim/zpc_tpls/fmt/color.h"
#include "zensim/zpc_tpls/fmt/format.h"

#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>

#include <zeno/VDBGrid.h>

#include "../scheme.hpp"
#include "../utils.cuh"

namespace zeno {

struct ZSVDBToNavierStokesGrid : INode {
    void apply() override {
        auto vdbgrid = get_input<VDBFloatGrid>("VDB");

        auto spg = zs::convert_floatgrid_to_sparse_grid(vdbgrid->m_grid, zs::MemoryHandle{zs::memsrc_e::device, 0});
        spg.append_channels(zs::cuda_exec(), {
                                                 {"v0", 3},
                                                 {"v1", 3},
                                                 {"p0", 1},
                                                 {"p1", 1},
                                                 {"div_v", 1},
                                                 {"rho0", 1}, // smoke density
                                                 {"rho1", 1},
                                                 {"T0", 1}, // smoke temperature
                                                 {"T1", 1},
                                                 {"tmp", 3} // FVM, BFECC
                                             });
        spg._background = 0.f;

        auto NSGrid = std::make_shared<ZenoSparseGrid>();
        NSGrid->spg = std::move(spg);
        NSGrid->setMeta("v_cur", 0);
        NSGrid->setMeta("p_cur", 0);
        NSGrid->setMeta("rho_cur", 0);
        NSGrid->setMeta("T_cur", 0);

        set_output("NSGrid", NSGrid);
    }
};

ZENDEFNODE(ZSVDBToNavierStokesGrid, {/* inputs: */
                                     {"VDB"},
                                     /* outputs: */
                                     {"NSGrid"},
                                     /* params: */
                                     {},
                                     /* category: */
                                     {"Eulerian"}});

struct ZSNavierStokesDt : INode {
    void apply() override {
        auto NSGrid = get_input<ZenoSparseGrid>("NSGrid");
        auto rho = get_input2<float>("Density");
        auto mu = get_input2<float>("Viscosity");

        auto &spg = NSGrid->spg;
        auto block_cnt = spg.numBlocks();
        auto dx = spg.voxelSize()[0];

        auto pol = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;

        size_t cell_cnt = block_cnt * spg.block_size;
        zs::Vector<float> res{spg.get_allocator(), count_warps(cell_cnt)};
        zs::memset(zs::mem_device, res.data(), 0, sizeof(float) * count_warps(cell_cnt));

        // maximum velocity
        pol(zs::Collapse{block_cnt, spg.block_size},
            [spgv = zs::proxy<space>(spg), res = zs::proxy<space>(res), cell_cnt,
             vSrcTag = src_tag(NSGrid, "v")] __device__(int blockno, int cellno) mutable {
                float u = spgv.value(vSrcTag, 0, blockno, cellno);
                float v = spgv.value(vSrcTag, 1, blockno, cellno);
                float w = spgv.value(vSrcTag, 2, blockno, cellno);

                size_t cellno_glb = blockno * spgv.block_size + cellno;

                float v_mag = zs::abs(u) + zs::abs(v) + zs::abs(w);

                reduce_max(cellno_glb, cell_cnt, v_mag, res[cellno_glb / 32]);
            });
        float v_max = reduce(pol, res, thrust::maximum<float>{});

        // CFL dt
        float dt_v = dx / (v_max + 1e-10);

        // Viscosity dt
        float nu = mu / (rho + 1e-10); // kinematic viscosity
        int dim = 3;
        float dt_nu = dx * dx / ((2.f * dim * nu) + 1e-10);

        float dt = dt_v < dt_nu ? dt_v : dt_nu;

        fmt::print(fg(fmt::color::blue_violet), "Maximum velocity : {}\n", v_max);
        fmt::print(fg(fmt::color::blue_violet), "CFL time step : {} sec\n", dt);

        set_output("dt", std::make_shared<NumericObject>(dt));
    }
};

ZENDEFNODE(ZSNavierStokesDt, {/* inputs: */
                              {"NSGrid", {"float", "Density", "1.0"}, {"float", "Viscosity", "0.0"}},
                              /* outputs: */
                              {"dt"},
                              /* params: */
                              {},
                              /* category: */
                              {"Eulerian"}});

struct ZSNSAdvectDiffuse : INode {
    void apply() override {
        auto NSGrid = get_input<ZenoSparseGrid>("NSGrid");
        auto rho = get_input2<float>("Density");
        auto mu = get_input2<float>("Viscosity");
        auto dt = get_input2<float>("dt");
        auto scheme = get_input2<std::string>("Scheme");

        auto &spg = NSGrid->spg;
        auto block_cnt = spg.numBlocks();
        auto dx = spg.voxelSize()[0];

        auto pol = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;

        if (scheme == "Semi-Lagrangian") {
            // Semi-Lagrangian advection (1st order)
            pol(zs::Collapse{block_cnt, spg.block_size},
                [spgv = zs::proxy<space>(spg), dx, dt, vSrcTag = src_tag(NSGrid, "v"),
                 vDstTag = dst_tag(NSGrid, "v")] __device__(int blockno, int cellno) mutable {
                    auto icoord = spgv.iCoord(blockno, cellno);
                    auto wcoord = spgv.indexToWorld(icoord);

                    for (int ch = 0; ch < 3; ++ch) {
                        zs::vec<float, 3> u_adv;
                        u_adv[0] = spgv.iStaggeredCellSample(vSrcTag, 0, icoord, ch);
                        u_adv[1] = spgv.iStaggeredCellSample(vSrcTag, 1, icoord, ch);
                        u_adv[2] = spgv.iStaggeredCellSample(vSrcTag, 2, icoord, ch);

                        auto wcoord_face = spgv.wStaggeredCoord(blockno, cellno, ch);

                        float u_sl = spgv.wStaggeredSample(vSrcTag, ch, wcoord_face - u_adv * dt);

                        spgv(vDstTag, ch, blockno, cellno) = u_sl;
                    }
                });
        } else if (scheme == "BFECC") {
            // Back and Forth Error Compensation and Correction (BFECC)
            pol(zs::Collapse{block_cnt, spg.block_size},
                [spgv = zs::proxy<space>(spg), dx, dt, vSrcTag = src_tag(NSGrid, "v"),
                 vDstTag = dst_tag(NSGrid, "v")] __device__(int blockno, int cellno) mutable {
                    auto icoord = spgv.iCoord(blockno, cellno);
                    auto wcoord = spgv.indexToWorld(icoord);

                    for (int ch = 0; ch < 3; ++ch) {
                        zs::vec<float, 3> u_adv;
                        u_adv[0] = spgv.iStaggeredCellSample(vSrcTag, 0, icoord, ch);
                        u_adv[1] = spgv.iStaggeredCellSample(vSrcTag, 1, icoord, ch);
                        u_adv[2] = spgv.iStaggeredCellSample(vSrcTag, 2, icoord, ch);

                        auto wcoord_face = spgv.wStaggeredCoord(blockno, cellno, ch);

                        float u_sl = spgv.wStaggeredSample(vSrcTag, ch, wcoord_face - u_adv * dt);

                        spgv(vDstTag, ch, blockno, cellno) = u_sl;
                    }
                });
            pol(zs::Collapse{block_cnt, spg.block_size},
                [spgv = zs::proxy<space>(spg), dx, dt, advSrcTag = src_tag(NSGrid, "v"), vSrcTag = dst_tag(NSGrid, "v"),
                 vDstTag = zs::SmallString{"tmp"}] __device__(int blockno, int cellno) mutable {
                    auto icoord = spgv.iCoord(blockno, cellno);
                    auto wcoord = spgv.indexToWorld(icoord);

                    for (int ch = 0; ch < 3; ++ch) {
                        zs::vec<float, 3> u_adv;
                        u_adv[0] = spgv.iStaggeredCellSample(advSrcTag, 0, icoord, ch);
                        u_adv[1] = spgv.iStaggeredCellSample(advSrcTag, 1, icoord, ch);
                        u_adv[2] = spgv.iStaggeredCellSample(advSrcTag, 2, icoord, ch);

                        auto wcoord_face = spgv.wStaggeredCoord(blockno, cellno, ch);

                        float u_sl = spgv.wStaggeredSample(vSrcTag, ch, wcoord_face + u_adv * dt);

                        spgv(vDstTag, ch, blockno, cellno) = u_adv[ch] + (u_adv[ch] - u_sl) / 2.f;
                    }
                });
            pol(zs::Collapse{block_cnt, spg.block_size},
                [spgv = zs::proxy<space>(spg), dx, dt, advSrcTag = src_tag(NSGrid, "v"),
                 vSrcTag = zs::SmallString{"tmp"},
                 vDstTag = dst_tag(NSGrid, "v")] __device__(int blockno, int cellno) mutable {
                    auto icoord = spgv.iCoord(blockno, cellno);
                    auto wcoord = spgv.indexToWorld(icoord);

                    for (int ch = 0; ch < 3; ++ch) {
                        zs::vec<float, 3> u_adv;
                        u_adv[0] = spgv.iStaggeredCellSample(advSrcTag, 0, icoord, ch);
                        u_adv[1] = spgv.iStaggeredCellSample(advSrcTag, 1, icoord, ch);
                        u_adv[2] = spgv.iStaggeredCellSample(advSrcTag, 2, icoord, ch);

                        auto wcoord_face = spgv.wStaggeredCoord(blockno, cellno, ch);
                        auto wcoord_face_src = wcoord_face - u_adv * dt;

                        float u_sl = spgv.wStaggeredSample(vSrcTag, ch, wcoord_face_src);

                        // clamp
                        auto arena = spgv.wArena(wcoord_face_src, ch);
                        auto sl_mi = arena.minimum(advSrcTag, ch);
                        auto sl_ma = arena.maximum(advSrcTag, ch);
                        if (u_sl > sl_ma || u_sl < sl_mi) {
                            u_sl = spgv.wStaggeredSample(advSrcTag, ch, wcoord_face_src);
                        }

                        spgv(vDstTag, ch, blockno, cellno) = u_sl;
                    }
                });
        } else if (scheme == "Finite-Difference") {
            // advection
            pol(zs::Collapse{block_cnt, spg.block_size},
                [spgv = zs::proxy<space>(spg), dx, dt, vSrcTag = src_tag(NSGrid, "v"),
                 vDstTag = dst_tag(NSGrid, "v")] __device__(int blockno, int cellno) mutable {
                    auto icoord = spgv.iCoord(blockno, cellno);

                    for (int ch = 0; ch < 3; ++ch) {
                        zs::vec<float, 3> u_adv;
                        u_adv[0] = spgv.iStaggeredCellSample(vSrcTag, 0, icoord, ch);
                        u_adv[1] = spgv.iStaggeredCellSample(vSrcTag, 1, icoord, ch);
                        u_adv[2] = spgv.iStaggeredCellSample(vSrcTag, 2, icoord, ch);

                        int x = ch;
                        int y = (ch + 1) % 3;
                        int z = (ch + 2) % 3;

                        const int stcl = 2; // stencil point in each side
                        float u_x[2 * stcl + 1], u_y[2 * stcl + 1], u_z[2 * stcl + 1];

                        zs::vec<int, 3> offset;

                        for (int i = -stcl; i <= stcl; ++i) {
                            offset = zs::vec<int, 3>::zeros();
                            offset[x] = i;
                            u_x[i + stcl] = spgv.value(vSrcTag, x, icoord + offset);

                            offset = zs::vec<int, 3>::zeros();
                            offset[y] = i;
                            u_y[i + stcl] = spgv.value(vSrcTag, x, icoord + offset);

                            offset = zs::vec<int, 3>::zeros();
                            offset[z] = i;
                            u_z[i + stcl] = spgv.value(vSrcTag, x, icoord + offset);
                        }

                        float adv_term = 0.f;
                        int upwind = u_adv[x] < 0 ? 1 : -1;
                        adv_term += u_adv[x] * scheme::HJ_WENO3(u_x[2 - upwind], u_x[2], u_x[2 + upwind],
                                                                u_x[2 + 2 * upwind], u_adv[x], dx);
                        upwind = u_adv[y] < 0 ? 1 : -1;
                        adv_term += u_adv[y] * scheme::HJ_WENO3(u_y[2 - upwind], u_y[2], u_y[2 + upwind],
                                                                u_y[2 + 2 * upwind], u_adv[y], dx);
                        upwind = u_adv[z] < 0 ? 1 : -1;
                        adv_term += u_adv[z] * scheme::HJ_WENO3(u_z[2 - upwind], u_z[2], u_z[2 + upwind],
                                                                u_z[2 + 2 * upwind], u_adv[z], dx);

                        spgv(vDstTag, ch, blockno, cellno) = u_adv[x] - adv_term * dt;
                    }
                });
        } else {
            throw std::runtime_error(fmt::format("Advection scheme [{}] not found!", scheme));
        }

        update_cur(NSGrid, "v");

        if (mu > 0) {
            // diffusion
            pol(zs::range(block_cnt * spg.block_size),
                [spgv = zs::proxy<space>(spg), dx, dt, rho, mu, vSrcTag = src_tag(NSGrid, "v"),
                 vDstTag = dst_tag(NSGrid, "v")] __device__(int cellno) mutable {
                    auto icoord = spgv.iCoord(cellno);

                    for (int ch = 0; ch < 3; ++ch) {
                        const int stcl = 1; // stencil point in each side
                        float u_x[2 * stcl + 1], u_y[2 * stcl + 1], u_z[2 * stcl + 1];

                        for (int i = -stcl; i <= stcl; ++i) {
                            u_x[i + stcl] = spgv.value(vSrcTag, ch, icoord + zs::vec<int, 3>(i, 0, 0));
                            u_y[i + stcl] = spgv.value(vSrcTag, ch, icoord + zs::vec<int, 3>(0, i, 0));
                            u_z[i + stcl] = spgv.value(vSrcTag, ch, icoord + zs::vec<int, 3>(0, 0, i));
                        }

                        float u_xx = scheme::central_diff_2nd(u_x[0], u_x[1], u_x[2], dx);
                        float u_yy = scheme::central_diff_2nd(u_y[0], u_y[1], u_y[2], dx);
                        float u_zz = scheme::central_diff_2nd(u_z[0], u_z[1], u_z[2], dx);

                        float diff_term = mu / rho * (u_xx + u_yy + u_zz);

                        spgv(vDstTag, ch, icoord) = u_x[1] + diff_term * dt;
                    }
                });
            update_cur(NSGrid, "v");
        }

        set_output("NSGrid", NSGrid);
    }
};

ZENDEFNODE(ZSNSAdvectDiffuse, {/* inputs: */
                               {"NSGrid",
                                "dt",
                                {"float", "Density", "1.0"},
                                {"float", "Viscosity", "0.0"},
                                {"enum Finite-Difference Semi-Lagrangian BFECC", "Scheme", "Finite-Difference"}},
                               /* outputs: */
                               {"NSGrid"},
                               /* params: */
                               {},
                               /* category: */
                               {"Eulerian"}});

struct ZSNSExternalForce : INode {
    void apply() override {
        auto NSGrid = get_input<ZenoSparseGrid>("NSGrid");
        auto dt = get_input2<float>("dt");
        auto force = get_input2<zeno::vec3f>("Force");

        auto &spg = NSGrid->spg;
        auto block_cnt = spg.numBlocks();

        auto pol = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;

        // add force (accelaration)
        pol(zs::Collapse{block_cnt, spg.block_size},
            [spgv = zs::proxy<space>(spg), force, dt, vSrcTag = src_tag(NSGrid, "v")] __device__(int blockno,
                                                                                                 int cellno) mutable {
                for (int ch = 0; ch < 3; ++ch)
                    spgv(vSrcTag, ch, blockno, cellno) += force[ch] * dt;
            });

        set_output("NSGrid", NSGrid);
    }
};

ZENDEFNODE(ZSNSExternalForce, {/* inputs: */
                               {"NSGrid", "dt", {"vec3f", "Force", "0, 0, 0"}},
                               /* outputs: */
                               {"NSGrid"},
                               /* params: */
                               {},
                               /* category: */
                               {"Eulerian"}});

struct ZSNSPressureProject : INode {
    void apply() override {
        auto NSGrid = get_input<ZenoSparseGrid>("NSGrid");
        auto rho = get_input2<float>("Density");
        auto dt = get_input2<float>("dt");
        int nIter = get_input2<int>("iterations");

        auto &spg = NSGrid->spg;
        auto block_cnt = spg.numBlocks();
        auto dx = spg.voxelSize()[0];

        auto pol = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;

        // velocity divergence
        pol(zs::range(block_cnt * spg.block_size),
            [spgv = zs::proxy<space>(spg), dx, dt, vSrcTag = src_tag(NSGrid, "v")] __device__(int cellno) mutable {
                auto icoord = spgv.iCoord(cellno);

                float u_x[2], u_y[2], u_z[2];
                for (int i = 0; i <= 1; ++i) {
                    u_x[i] = spgv.value(vSrcTag, 0, icoord + zs::vec<int, 3>(i, 0, 0));
                    u_y[i] = spgv.value(vSrcTag, 1, icoord + zs::vec<int, 3>(0, i, 0));
                    u_z[i] = spgv.value(vSrcTag, 2, icoord + zs::vec<int, 3>(0, 0, i));
                }

                float div_term = ((u_x[1] - u_x[0]) + (u_y[1] - u_y[0]) + (u_z[1] - u_z[0])) / dx;

                spgv("div_v", icoord) = div_term;
            });

        const float dxSqrOverDt = dx * dx / dt;

        // zs::CppTimer timer;
        // timer.tick();
        // pressure Poisson equation
        for (int iter = 0; iter < nIter; ++iter) {
#if 0
            // point Jacobi iteration
            pol(zs::range(block_cnt * spg.block_size),
                [spgv = zs::proxy<space>(spg), dxSqrOverDt, rho,
                 pSrcTag = src_tag(NSGrid, "p"),
                 pDstTag =
                     dst_tag(NSGrid, "p")] __device__(int cellno) mutable {
                    auto icoord = spgv.iCoord(cellno);

                    float div = spgv.value("div_v", icoord);

                    const int stcl = 1; // stencil point in each side
                    float p_x[2 * stcl + 1], p_y[2 * stcl + 1], p_z[2 * stcl + 1];

                    for (int i = -stcl; i <= stcl; ++i) {
                        p_x[i + stcl] = spgv.value(pSrcTag, icoord + zs::vec<int, 3>(i, 0, 0));
                        p_y[i + stcl] = spgv.value(pSrcTag, icoord + zs::vec<int, 3>(0, i, 0));
                        p_z[i + stcl] = spgv.value(pSrcTag, icoord + zs::vec<int, 3>(0, 0, i));
                    }

                    float p_this =
                        -(div * dxSqrOverDt * rho - (p_x[0] + p_x[2] + p_y[0] + p_y[2] + p_z[0] + p_z[2])) / 6.f;

                    spgv(pDstTag, icoord) = p_this;
                });
            update_cur(NSGrid, "p");
#else
            // red-black SOR iteration
            const float sor = 1.0f; // over relaxation rate

            for (int clr = 0; clr != 2; ++clr) {

                pol(zs::range(block_cnt * 32), [spgv = zs::proxy<space>(spg), dxSqrOverDt, rho, clr, sor,
                                                pSrcTag = src_tag(NSGrid, "p")] __device__(int tid) mutable {
                    auto blockno = tid / 32;

                    auto bcoord = spgv._table._activeKeys[blockno];
                    if ((((bcoord[0] & 8) ^ (bcoord[1] & 8) ^ (bcoord[2] & 8)) >> 3) == clr)
                        return;

                    auto tile = zs::cg::tiled_partition<32>(zs::cg::this_thread_block());
                    for (int c_clr = 0; c_clr != 2; ++c_clr) {

                        for (int cno = tile.thread_rank(); cno < 256; cno += tile.num_threads()) {
                            auto cellno = (cno << 1) | c_clr;

                            auto ccoord = spgv.local_offset_to_coord(cellno);
                            auto icoord = bcoord + ccoord;

                            float div = spgv.value("div_v", blockno, cellno);

                            const int stcl = 1; // stencil point in each side
                            float p_x[2 * stcl + 1], p_y[2 * stcl + 1], p_z[2 * stcl + 1];

                            for (int i = -stcl; i <= stcl; ++i) {
                                p_x[i + stcl] = spgv.value(pSrcTag, icoord + zs::vec<int, 3>(i, 0, 0));
                                p_y[i + stcl] = spgv.value(pSrcTag, icoord + zs::vec<int, 3>(0, i, 0));
                                p_z[i + stcl] = spgv.value(pSrcTag, icoord + zs::vec<int, 3>(0, 0, i));
                            }

                            float p_this =
                                (1.f - sor) * p_x[stcl] +
                                sor *
                                    ((p_x[0] + p_x[2] + p_y[0] + p_y[2] + p_z[0] + p_z[2]) - div * dxSqrOverDt * rho) /
                                    6.f;

                            spgv(pSrcTag, blockno, cellno) = p_this;
                        }
                    }
                });
            }
#endif
        }
        // timer.tock("jacobi/sor iterations");

        // pressure projection
        pol(zs::range(block_cnt * spg.block_size),
            [spgv = zs::proxy<space>(spg), dx, dt, rho, vSrcTag = src_tag(NSGrid, "v"), vDstTag = dst_tag(NSGrid, "v"),
             pSrcTag = src_tag(NSGrid, "p")] __device__(int cellno) mutable {
                auto icoord = spgv.iCoord(cellno);
                float p_this = spgv.value(pSrcTag, icoord);

                for (int ch = 0; ch < 3; ++ch) {
                    float u = spgv.value(vSrcTag, ch, icoord);

                    zs::vec<int, 3> offset{0, 0, 0};
                    offset[ch] = -1;

                    float p_m = spgv.value(pSrcTag, icoord + offset);

                    u -= (p_this - p_m) / dx * dt / rho;

                    spgv(vDstTag, ch, icoord) = u;
                }
            });
        update_cur(NSGrid, "v");

        set_output("NSGrid", NSGrid);
    }
};

ZENDEFNODE(ZSNSPressureProject, {/* inputs: */
                                 {"NSGrid", "dt", {"int", "iterations", "10"}, {"float", "Density", "1.0"}},
                                 /* outputs: */
                                 {"NSGrid"},
                                 /* params: */
                                 {},
                                 /* category: */
                                 {"Eulerian"}});

struct ZSNSNaiveSolidWall : INode {
    void apply() override {
        auto NSGrid = get_input<ZenoSparseGrid>("NSGrid");
        auto SolidSDF = get_input<ZenoSparseGrid>("SolidSDF");
        auto SolidVel = get_input<ZenoSparseGrid>("SolidVel");

        auto &sdf = SolidSDF->spg;
        auto &vel = SolidVel->spg;
        auto &spg = NSGrid->spg;
        auto block_cnt = spg.numBlocks();

        auto pol = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;

        pol(zs::Collapse{block_cnt, spg.block_size},
            [spgv = zs::proxy<space>(spg), sdfv = zs::proxy<space>(sdf), velv = zs::proxy<space>(vel),
             vSrcTag = src_tag(NSGrid, "v")] __device__(int blockno, int cellno) mutable {
                auto wcoord = spgv.wCoord(blockno, cellno);
                auto solid_sdf = sdfv.wSample("sdf", wcoord);

                if (solid_sdf < 0) {
                    auto vel_s = velv.wStaggeredPack("v", wcoord);
                    auto block = spgv.block(blockno);
                    block.template tuple<3>(vSrcTag, cellno) = vel_s;
                }

                spgv("sdf", blockno, cellno) = solid_sdf;
            });

        set_output("NSGrid", NSGrid);
    }
};

ZENDEFNODE(ZSNSNaiveSolidWall, {/* inputs: */
                                {"NSGrid", "SolidSDF", "SolidVel"},
                                /* outputs: */
                                {"NSGrid"},
                                /* params: */
                                {},
                                /* category: */
                                {"Eulerian"}});

struct ZSTracerAdvectDiffuse : INode {
    void compute(zs::CudaExecutionPolicy &pol, zs::SmallString tag, float diffuse, float dt, std::string scheme,
                 ZenoSparseGrid *NSGrid) {

        constexpr auto space = RM_CVREF_T(pol)::exec_tag::value;

        auto &spg = NSGrid->spg;
        auto block_cnt = spg.numBlocks();

        auto dx = spg.voxelSize()[0];

        if (scheme == "Semi-Lagrangian") {
            // Semi-Lagrangian advection (1st order)
            pol(zs::Collapse{block_cnt, spg.block_size},
                [spgv = zs::proxy<space>(spg), dx, dt, vSrcTag = src_tag(NSGrid, "v"), trcSrcTag = src_tag(NSGrid, tag),
                 trcDstTag = dst_tag(NSGrid, tag)] __device__(int blockno, int cellno) mutable {
                    auto icoord = spgv.iCoord(blockno, cellno);
                    auto wcoord = spgv.indexToWorld(icoord);

                    auto u_adv = spgv.iStaggeredPack(vSrcTag, icoord);
                    float trc_sl = spgv.wSample(trcSrcTag, wcoord - u_adv * dt);

                    spgv(trcDstTag, blockno, cellno) = trc_sl;
                });

            update_cur(NSGrid, tag);
        } else if (scheme == "BFECC") {
            // Back and Forth Error Compensation and Correction (BFECC)
            pol(zs::Collapse{block_cnt, spg.block_size},
                [spgv = zs::proxy<space>(spg), dx, dt, vSrcTag = src_tag(NSGrid, "v"), trcSrcTag = src_tag(NSGrid, tag),
                 trcDstTag = dst_tag(NSGrid, tag)] __device__(int blockno, int cellno) mutable {
                    auto icoord = spgv.iCoord(blockno, cellno);
                    auto wcoord = spgv.indexToWorld(icoord);

                    auto u_adv = spgv.iStaggeredPack(vSrcTag, icoord);
                    float trc_sl = spgv.wSample(trcSrcTag, wcoord - u_adv * dt);

                    spgv(trcDstTag, blockno, cellno) = trc_sl;
                });
            pol(zs::Collapse{block_cnt, spg.block_size},
                [spgv = zs::proxy<space>(spg), dx, dt, vSrcTag = src_tag(NSGrid, "v"), trcTag = src_tag(NSGrid, tag),
                 trcSrcTag = dst_tag(NSGrid, tag),
                 trcDstTag = zs::SmallString{"tmp"}] __device__(int blockno, int cellno) mutable {
                    auto icoord = spgv.iCoord(blockno, cellno);
                    auto wcoord = spgv.indexToWorld(icoord);

                    auto u_adv = spgv.iStaggeredPack(vSrcTag, icoord);
                    float trc_sl = spgv.wSample(trcSrcTag, wcoord + u_adv * dt);
                    float trc_n = spgv.value(trcTag, blockno, cellno);

                    spgv(trcDstTag, blockno, cellno) = trc_n + (trc_n - trc_sl) / 2.f;
                });
            pol(zs::Collapse{block_cnt, spg.block_size},
                [spgv = zs::proxy<space>(spg), dx, dt, vSrcTag = src_tag(NSGrid, "v"), trcTag = src_tag(NSGrid, tag),
                 trcSrcTag = zs::SmallString{"tmp"},
                 trcDstTag = dst_tag(NSGrid, tag)] __device__(int blockno, int cellno) mutable {
                    auto icoord = spgv.iCoord(blockno, cellno);
                    auto wcoord = spgv.indexToWorld(icoord);

                    auto u_adv = spgv.iStaggeredPack(vSrcTag, icoord);
                    auto wcoord_src = wcoord - u_adv * dt;

                    float trc_sl = spgv.wSample(trcSrcTag, wcoord_src);

                    // clamp
                    auto arena = spgv.wArena(wcoord_src);
                    auto sl_mi = arena.minimum(trcTag);
                    auto sl_ma = arena.maximum(trcTag);
                    if (trc_sl > sl_ma || trc_sl < sl_mi) {
                        trc_sl = spgv.wSample(trcTag, wcoord_src);
                    }

                    spgv(trcDstTag, blockno, cellno) = trc_sl;
                });

            update_cur(NSGrid, tag);
        } else if (scheme == "Finite-Volume") {
            // Finite Volume Method (FVM)
            // numrtical flux of tracer
            pol(zs::Collapse{block_cnt, spg.block_size},
                [spgv = zs::proxy<space>(spg), diffuse, dx, tag = src_tag(NSGrid, tag),
                 vSrcTag = src_tag(NSGrid, "v")] __device__(int blockno, int cellno) mutable {
                    auto icoord = spgv.iCoord(blockno, cellno);

                    const int stcl = 2; // stencil point in each side
                    float trc[3][2 * stcl];

                    // | i - 2 | i - 1 | i | i + 1 |
                    for (int i = -stcl; i < stcl; ++i) {
                        trc[0][i + stcl] = spgv.value(tag, icoord + zs::vec<int, 3>(i, 0, 0));
                        trc[1][i + stcl] = spgv.value(tag, icoord + zs::vec<int, 3>(0, i, 0));
                        trc[2][i + stcl] = spgv.value(tag, icoord + zs::vec<int, 3>(0, 0, i));
                    }

                    float u_adv[3];
                    for (int ch = 0; ch < 3; ++ch)
                        u_adv[ch] = spgv.value(vSrcTag, ch, icoord);

                    // approximate value at i - 1/2
                    float flux[3];
                    for (int ch = 0; ch < 3; ++ch) {
                        // convection flux
                        if (u_adv[ch] < 0)
                            flux[ch] = u_adv[ch] * scheme::TVD_MUSCL3(trc[ch][1], trc[ch][2], trc[ch][3]);
                        else
                            flux[ch] = u_adv[ch] * scheme::TVD_MUSCL3(trc[ch][2], trc[ch][1], trc[ch][0]);

                        // diffusion flux
                        flux[ch] -= diffuse * (trc[ch][2] - trc[ch][1]) / dx;
                    }

                    for (int ch = 0; ch < 3; ++ch) {
                        spgv("tmp", ch, blockno, cellno) = flux[ch];
                    }
                });

            // time integration of tracer
            pol(zs::Collapse{block_cnt, spg.block_size},
                [spgv = zs::proxy<space>(spg), dx, dt, tag = src_tag(NSGrid, tag)] __device__(int blockno,
                                                                                              int cellno) mutable {
                    auto icoord = spgv.iCoord(blockno, cellno);

                    float flux[3][2];
                    for (int ch = 0; ch < 3; ++ch) {
                        zs::vec<int, 3> offset{0, 0, 0};
                        offset[ch] = 1;

                        flux[ch][0] = spgv.value("tmp", ch, icoord);
                        flux[ch][1] = spgv.value("tmp", ch, icoord + offset);
                    }

                    float dtrc = 0;
                    for (int ch = 0; ch < 3; ++ch) {
                        dtrc += (flux[ch][0] - flux[ch][1]) / dx;
                    }
                    dtrc *= dt;

                    spgv(tag, blockno, cellno) += dtrc;
                });
        } else {
            throw std::runtime_error(fmt::format("Advection scheme [{}] not found!", scheme));
        }
    }

    void apply() override {
        auto NSGrid = get_input<ZenoSparseGrid>("NSGrid");
        auto diffuse = get_input2<float>("Diffusion");
        auto dt = get_input2<float>("dt");
        auto scheme = get_input2<std::string>("Scheme");

        auto pol = zs::cuda_exec();
        ///
        if (get_input2<bool>("Density"))
            compute(pol, "rho", diffuse, dt, scheme, NSGrid.get());
        if (get_input2<bool>("Temperature"))
            compute(pol, "T", diffuse, dt, scheme, NSGrid.get());

        set_output("NSGrid", NSGrid);
    }
};

ZENDEFNODE(ZSTracerAdvectDiffuse, {/* inputs: */
                                   {"NSGrid",
                                    "dt",
                                    {"float", "Diffusion", "0.0"},
                                    {"bool", "Density", "1"},
                                    {"bool", "Temperature", "1"},
                                    {"enum Finite-Volume Semi-Lagrangian BFECC", "Scheme", "Finite-Volume"}},
                                   /* outputs: */
                                   {"NSGrid"},
                                   /* params: */
                                   {},
                                   /* category: */
                                   {"Eulerian"}});

struct ZSTracerEmission : INode {
    void compute(zs::CudaExecutionPolicy &pol, zs::SmallString tag, ZenoSparseGrid *NSGrid, ZenoSparseGrid *EmitSDF) {

        constexpr auto space = RM_CVREF_T(pol)::exec_tag::value;

        auto &spg = NSGrid->spg;
        auto &sdf = EmitSDF->spg;

        auto block_cnt = spg.numBlocks();

        auto dx = spg.voxelSize()[0];

        pol(zs::Collapse{block_cnt, spg.block_size},
            [spgv = zs::proxy<space>(spg), sdfv = zs::proxy<space>(sdf), dx,
             tag = src_tag(NSGrid, tag)] __device__(int blockno, int cellno) mutable {
                auto wcoord = spgv.wCoord(blockno, cellno);
                auto emit_sdf = sdfv.wSample("sdf", wcoord);

                if (emit_sdf <= 1.5f * dx) {
                    // fix me: naive emission
                    spgv(tag, blockno, cellno) = 1.0;
                }
            });
    }

    void apply() override {
        auto NSGrid = get_input<ZenoSparseGrid>("NSGrid");
        auto EmitSDF = get_input<ZenoSparseGrid>("EmitterSDF");

        auto pol = zs::cuda_exec();

        if (get_input2<bool>("Density"))
            compute(pol, "rho", NSGrid.get(), EmitSDF.get());
        if (get_input2<bool>("Temperature"))
            compute(pol, "T", NSGrid.get(), EmitSDF.get());

        set_output("NSGrid", NSGrid);
    }
};

ZENDEFNODE(ZSTracerEmission, {/* inputs: */
                              {"NSGrid", "EmitterSDF", {"bool", "Density", "1"}, {"bool", "Temperature", "1"}},
                              /* outputs: */
                              {"NSGrid"},
                              /* params: */
                              {},
                              /* category: */
                              {"Eulerian"}});

struct ZSSmokeBuoyancy : INode {
    void apply() override {
        auto NSGrid = get_input<ZenoSparseGrid>("NSGrid");
        auto dt = get_input2<float>("dt");
        auto gravity = get_input2<zeno::vec3f>("Gravity");
        auto alpha = get_input2<float>("DensityCoef");
        auto beta = get_input2<float>("TemperatureCoef");

        auto &spg = NSGrid->spg;
        auto block_cnt = spg.numBlocks();

        auto pol = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;

        // add force (accelaration)
        pol(zs::Collapse{block_cnt, spg.block_size},
            [spgv = zs::proxy<space>(spg), dt, alpha, beta, gravity = zs::vec<float, 3>::from_array(gravity),
             vSrcTag = src_tag(NSGrid, "v"), rhoSrcTag = src_tag(NSGrid, "rho"),
             TSrcTag = src_tag(NSGrid, "T")] __device__(int blockno, int cellno) mutable {
                auto icoord = spgv.iCoord(blockno, cellno);

                float rho_this = spgv.value(rhoSrcTag, blockno, cellno);
                float T_this = spgv.value(TSrcTag, blockno, cellno);

                for (int ch = 0; ch < 3; ++ch) {
                    zs::vec<int, 3> offset{0, 0, 0};
                    offset[ch] = -1;

                    float rho_face = spgv.value(rhoSrcTag, icoord + offset);
                    float T_face = spgv.value(TSrcTag, icoord + offset);

                    rho_face = 0.5f * (rho_this + rho_face);
                    T_face = 0.5f * (T_this + T_face);

                    float G_scale = alpha * rho_face - beta * T_face;

                    spgv(vSrcTag, ch, blockno, cellno) += G_scale * gravity[ch] * dt;
                }
            });

        set_output("NSGrid", NSGrid);
    }
};

ZENDEFNODE(ZSSmokeBuoyancy, {/* inputs: */
                             {"NSGrid",
                              "dt",
                              {"vec3f", "Gravity", "0, -9.8, 0"},
                              {"float", "DensityCoef", "0.0"},
                              {"float", "TemperatureCoef", "0.0"}},
                             /* outputs: */
                             {"NSGrid"},
                             /* params: */
                             {},
                             /* category: */
                             {"Eulerian"}});

struct ZSExtendSparseGrid : INode {

    template <typename PredT> void refit(ZenoSparseGrid *nsgridPtr, zs::SmallString tag, PredT pred) {
        using namespace zs;
        static constexpr auto space = execspace_e::cuda;
        namespace cg = ::cooperative_groups;
        auto pol = cuda_exec();
        auto &spg = nsgridPtr->getSparseGrid();
        // make sure spg.block_size % 32 == 0

        auto nbs = spg.numBlocks();
        using Ti = RM_CVREF_T(nbs);

        Vector<Ti> marks{spg.get_allocator(), nbs + 1}, offsets{spg.get_allocator(), nbs + 1};
        marks.reset(0);

        pol(range(nbs * 32), [spg = proxy<space>(spg), tagOffset = spg.getPropertyOffset(tag),
                              marks = proxy<space>(marks), pred] __device__(std::size_t i) mutable {
            auto tile = cg::tiled_partition<32>(cg::this_thread_block());
            auto bno = i / 32;
            auto cellno = tile.thread_rank();

            while (cellno < spg.block_size) {
                if (tile.ballot(pred(spg(tagOffset, bno, cellno))))
                    break;
                cellno += 32;
            }
            if (tile.thread_rank() == 0 && cellno < spg.block_size)
                marks[bno] = 1;
        });

        exclusive_scan(pol, std::begin(marks), std::end(marks), std::begin(offsets));
        auto newNbs = offsets.getVal(nbs);
        fmt::print("compacting {} blocks to {} active blocks.\n", nbs, newNbs);

        /// @brief compact active blocks
        // grid
        auto &grid = spg._grid;
        auto dstgrid = grid;
        // table
        auto &table = spg._table;
        table.reset(false);
        table._cnt.setVal(newNbs);
        auto &keys = table._activeKeys;
        auto newKeys = keys;

        pol(range(nbs * spg._table.bucket_size),
            [grid = proxy<space>(grid), dstgrid = proxy<space>(dstgrid), marks = proxy<space>(marks),
             newKeys = proxy<space>(newKeys), keys = proxy<space>(keys), table = proxy<space>(table),
             offsets = proxy<space>(offsets),
             bs_c = wrapv<RM_CVREF_T(spg)::block_size>{}] __device__(std::size_t i) mutable {
                constexpr auto block_size = decltype(bs_c)::value;
                constexpr auto bucket_size = RM_CVREF_T(table)::bucket_size;
                static_assert(block_size % bucket_size == 0, "block_size should be a multiple of bucket_size");
                auto tile = cg::tiled_partition<bucket_size>(cg::this_thread_block());
                auto bno = i / bucket_size;
                if (marks[bno] == 0)
                    return;
                auto dstBno = offsets[bno];
                // grid
                for (auto cellno = tile.thread_rank(); cellno < block_size; cellno += bucket_size) {
                    for (int chn = 0; chn != grid.numChannels(); ++chn)
                        dstgrid(chn, dstBno, cellno) = grid(chn, bno, cellno);
                }
                // table
                auto bcoord = keys[bno];
                table.tile_insert(tile, bcoord, dstBno, false); // do not enqueue key, hence set false
                if (tile.thread_rank() == 0)
                    newKeys[dstBno] = bcoord;
            });
        grid = std::move(dstgrid);
        keys = std::move(newKeys);
    }

    template <typename PredT>
    void extend(ZenoSparseGrid *nsgridPtr, zs::SmallString tag, std::size_t &nbsOffset, PredT pred) {
        using namespace zs;
        static constexpr auto space = execspace_e::cuda;
        namespace cg = ::cooperative_groups;
        auto pol = cuda_exec();
        auto &spg = nsgridPtr->getSparseGrid();
        // make sure spg.block_size % 32 == 0

        auto nbs = spg.numBlocks() - nbsOffset;
        if (nbs == 0)
            return;
        // worst case is that all candidate blocks activate all surrounding neighbors
        spg.resize(pol, nbs * 26 + nbsOffset);

        // zeno::log_info("currently {} blocks (offset {}), resizing to {}\n", nbsOffset + nbs, nbsOffset,
        //                nbs * 26 + nbsOffset);

        if (!spg._grid.hasProperty(tag))
            throw std::runtime_error(fmt::format("property [{}] not exist!", tag.asString()));

        pol(range(nbs * spg._table.bucket_size), [spg = proxy<space>(spg), tagOffset = spg.getPropertyOffset(tag),
                                                  nbsOffset, pred] __device__(std::size_t i) mutable {
            auto tile = cg::tiled_partition<RM_CVREF_T(spg._table)::bucket_size>(cg::this_thread_block());
            auto bno = i / spg._table.bucket_size + nbsOffset;
            auto cellno = tile.thread_rank();
            // searching for active voxels within this block

            while (cellno < spg.block_size) {
                if (tile.ballot(pred(spg(tagOffset, bno, cellno))))
                    break;
                cellno += spg._table.bucket_size;
            }
            if (cellno < spg.block_size) {
                auto bcoord = spg.iCoord(bno, 0);
                for (auto loc : ndrange<3>(3)) {
                    auto dir = make_vec<int>(loc) - 1;
                    // spg._table.insert(bcoord + dir * spg.side_length);
                    spg._table.tile_insert(tile, bcoord + dir * spg.side_length, RM_CVREF_T(spg._table)::sentinel_v,
                                           true);
                }
            }
        });
        // [ nbsOffset | nbsOffset + nbs | spg.numBlocks() ]
        nbsOffset += nbs;
        auto newNbs = spg.numBlocks();
        newNbs -= nbsOffset;
        if (newNbs > 0)
            zs::memset(mem_device, (void *)spg._grid.tileOffset(nbsOffset), 0,
                       (std::size_t)newNbs * spg._grid.tileBytes());

        if (tag == "sdf")
            pol(range(newNbs * spg.block_size),
                [dx = spg.voxelSize()[0], spg = proxy<space>(spg), sdfOffset = spg.getPropertyOffset("sdf"),
                 blockOffset = nbsOffset * spg.block_size] __device__(std::size_t cellno) mutable {
                    spg(sdfOffset, blockOffset + cellno) = 3 * dx;
                });
    }

    void apply() override {
        auto zsSPG = get_input<ZenoSparseGrid>("NSGrid");
        auto tag = get_input2<std::string>("Attribute");
        auto nlayers = get_input2<int>("layers");
        auto needRefit = get_input2<bool>("refit");

        std::size_t nbs = 0;
        int opt = 0;
        if (tag == "rho")
            opt = 1;
        else if (tag == "sdf")
            opt = 2;

        if (needRefit && opt != 0) {
            if (opt == 1)
                refit(zsSPG.get(), src_tag(zsSPG, tag),
                      [] __device__(float v) -> bool { return v > zs::limits<float>::epsilon() * 10; });
            else if (opt == 2)
                refit(zsSPG.get(), src_tag(zsSPG, tag),
                      [dx = zsSPG->getSparseGrid().voxelSize()[0]] __device__(float v) -> bool { return v < 2 * dx; });
            opt = 0;
        }

        while (nlayers-- > 0) {
            if (opt == 0)
                extend(zsSPG.get(), src_tag(zsSPG, tag), nbs, [] __device__(float v) { return true; });
            else if (opt == 1)
                extend(zsSPG.get(), src_tag(zsSPG, tag), nbs,
                       [] __device__(float v) -> bool { return v > zs::limits<float>::epsilon() * 10; });
            else if (opt == 2)
                extend(zsSPG.get(), src_tag(zsSPG, tag), nbs,
                       [dx = zsSPG->getSparseGrid().voxelSize()[0]] __device__(float v) -> bool { return v < 2 * dx; });
            opt = 0; // always active since
        }

        set_output("NSGrid", zsSPG);
    }
};

ZENDEFNODE(ZSExtendSparseGrid,
           {/* inputs: */
            {"NSGrid", {"enum rho sdf", "Attribute", "rho"}, {"bool", "refit", "1"}, {"int", "layers", "2"}},
            /* outputs: */
            {"NSGrid"},
            /* params: */
            {},
            /* category: */
            {"Eulerian"}});

struct ZSMaintainSparseGrid : INode {
    template <typename PredT> void maintain(ZenoSparseGrid *nsgridPtr, zs::SmallString tag, PredT pred, int nlayers) {
        using namespace zs;
        static constexpr auto space = execspace_e::cuda;
        namespace cg = ::cooperative_groups;
        auto pol = cuda_exec();
        auto &spg = nsgridPtr->getSparseGrid();

        if (!spg._grid.hasProperty(tag))
            throw std::runtime_error(fmt::format("property [{}] not exist!", tag.asString()));

        auto nbs = spg.numBlocks();
        using Ti = RM_CVREF_T(nbs);

        Vector<Ti> marks{spg.get_allocator(), nbs + 1}, offsets{spg.get_allocator(), nbs + 1};
        marks.reset(0);

        static_assert(RM_CVREF_T(spg)::block_size % 32 == 0, "block size should be a multiple of 32.");

        /// @brief mark active block entries
        pol(range(nbs * 32), [spg = proxy<space>(spg), tagOffset = spg.getPropertyOffset(tag),
                              marks = proxy<space>(marks), pred] __device__(std::size_t i) mutable {
            auto tile = cg::tiled_partition<32>(cg::this_thread_block());
            auto bno = i / 32;
            auto cellno = tile.thread_rank();

            while (cellno < spg.block_size) {
                if (tile.ballot(pred(spg(tagOffset, bno, cellno))))
                    break;
                cellno += 32;
            }
            if (tile.thread_rank() == 0 && cellno < spg.block_size)
                marks[bno] = 1;
        });

        exclusive_scan(pol, std::begin(marks), std::end(marks), std::begin(offsets));
        auto newNbs = offsets.getVal(nbs);
        fmt::print("compacting {} blocks to {} active blocks.\n", nbs, newNbs);

        /// @brief compact active block entries
        // table
        auto &table = spg._table;
        table.reset(false);
        table._cnt.setVal(newNbs);
        // backup previous block entries, nbs is the previous count of blocks
        auto prevKeys = table._activeKeys;
        auto &keys = table._activeKeys;

        pol(range(nbs * spg._table.bucket_size),
            [marks = proxy<space>(marks), newKeys = proxy<space>(keys), keys = proxy<space>(prevKeys),
             table = proxy<space>(table), offsets = proxy<space>(offsets),
             bs_c = wrapv<RM_CVREF_T(spg)::block_size>{}] __device__(std::size_t i) mutable {
                constexpr auto block_size = decltype(bs_c)::value;
                constexpr auto bucket_size = RM_CVREF_T(table)::bucket_size;
                static_assert(block_size % bucket_size == 0, "block_size should be a multiple of bucket_size");
                auto tile = cg::tiled_partition<bucket_size>(cg::this_thread_block());
                auto bno = i / bucket_size;
                if (marks[bno] == 0)
                    return;
                auto dstBno = offsets[bno];
                // table
                auto bcoord = keys[bno];
                table.tile_insert(tile, bcoord, dstBno, false); // do not enqueue key, hence set false
                if (tile.thread_rank() == 0)
                    newKeys[dstBno] = bcoord;
            });

        // grid
        /// @note backup the grid ahead
        auto &grid = spg._grid;
        auto prevGrid = grid;

        /// @brief iteratively expand the active domain
        Ti nbsOffset = 0;
        while (nlayers-- > 0 && newNbs > 0) {
            // reserve enough memory for expanded grid and table
            spg.resize(pol, newNbs * 27 + nbsOffset);
            // extend one layer
            pol(range(newNbs * spg._table.bucket_size),
                [spg = proxy<space>(spg), tagOffset = spg.getPropertyOffset(tag),
                 nbsOffset] __device__(std::size_t i) mutable {
                    auto tile = cg::tiled_partition<RM_CVREF_T(spg._table)::bucket_size>(cg::this_thread_block());
                    auto bno = i / spg._table.bucket_size + nbsOffset;
                    auto bcoord = spg.iCoord(bno, 0);
                    for (auto loc : ndrange<3>(3)) {
                        auto dir = make_vec<int>(loc) - 1;
                        // spg._table.insert(bcoord + dir * spg.side_length);
                        spg._table.tile_insert(tile, bcoord + dir * spg.side_length, RM_CVREF_T(spg._table)::sentinel_v,
                                               true);
                    }
                });
            // slide the window
            nbsOffset += newNbs;
            newNbs = spg.numBlocks() - nbsOffset;

            // initialize newly added blocks
            if (newNbs > 0) {
                zs::memset(mem_device, (void *)spg._grid.tileOffset(nbsOffset), 0,
                           (std::size_t)newNbs * spg._grid.tileBytes());

                if (tag == "sdf") {
                    // special treatment for "sdf" property
                    pol(range(newNbs * spg.block_size),
                        [dx = spg.voxelSize()[0], spg = proxy<space>(spg), sdfOffset = spg.getPropertyOffset("sdf"),
                         blockOffset = nbsOffset * spg.block_size] __device__(std::size_t cellno) mutable {
                            spg(sdfOffset, blockOffset + cellno) = 3 * dx;
                        });
                }
            }
        }

        /// @brief relocate original grid data to the new sparse grid
        pol(range(nbs * spg._table.bucket_size), [grid = proxy<space>(prevGrid), spg = proxy<space>(spg),
                                                  keys = proxy<space>(prevKeys)] __device__(std::size_t i) mutable {
            constexpr auto bucket_size = RM_CVREF_T(table)::bucket_size;
            auto tile = cg::tiled_partition<bucket_size>(cg::this_thread_block());
            auto bno = i / bucket_size;
            auto bcoord = keys[bno];
            auto dstBno = spg._table.tile_query(tile, bcoord);
            if (dstBno == spg._table.sentinel_v)
                return;
            // table
            for (auto cellno = tile.thread_rank(); cellno < spg.block_size; cellno += bucket_size) {
                for (int chn = 0; chn != grid.numChannels(); ++chn)
                    spg._grid(chn, dstBno, cellno) = grid(chn, bno, cellno);
            }
        });
    }

    void apply() override {
        auto zsSPG = get_input<ZenoSparseGrid>("NSGrid");
        auto tag = get_input2<std::string>("Attribute");
        auto nlayers = get_input2<int>("layers");
        auto needRefit = get_input2<bool>("refit");

        int opt = 0;
        if (needRefit) {
            if (tag == "rho")
                opt = 1;
            else if (tag == "sdf")
                opt = 2;
        }

        if (opt == 0)
            maintain(
                zsSPG.get(), src_tag(zsSPG, tag), [] __device__(float v) { return true; }, nlayers);
        else if (opt == 1)
            maintain(
                zsSPG.get(), src_tag(zsSPG, tag),
                [] __device__(float v) -> bool { return v > zs::limits<float>::epsilon() * 10; }, nlayers);
        else if (opt == 2)
            maintain(
                zsSPG.get(), src_tag(zsSPG, tag),
                [dx = zsSPG->getSparseGrid().voxelSize()[0]] __device__(float v) -> bool { return v < 2 * dx; },
                nlayers);

        set_output("NSGrid", zsSPG);
    }
};

ZENDEFNODE(ZSMaintainSparseGrid,
           {/* inputs: */
            {"NSGrid", {"enum rho sdf", "Attribute", "rho"}, {"bool", "refit", "1"}, {"int", "layers", "2"}},
            /* outputs: */
            {"NSGrid"},
            /* params: */
            {},
            /* category: */
            {"Eulerian"}});

} // namespace zeno