#include "hip/hip_runtime.h"
#include "Structures.hpp"
#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/profile/CppTimers.hpp"
#include "zensim/zpc_tpls/fmt/color.h"
#include "zensim/zpc_tpls/fmt/format.h"

#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>

#include <zeno/utils/log.h>

#include "../utils.cuh"
#include "Noise.cuh"

namespace zeno {

struct ZSGridPerlinNoise : INode {
    virtual void apply() override {
        auto zsSPG = get_input<ZenoSparseGrid>("SparseGrid");
        auto attrTag = get_input2<std::string>("GridAttribute");
        auto frequency = get_input2<vec3f>("Frequency");
        auto offset = get_input2<vec3f>("Offset");
        auto roughness = get_input2<float>("Roughness");
        auto turbulence = get_input2<int>("Turbulence");
        auto amplitude = get_input2<float>("Amplitude");
        auto attenuation = get_input2<float>("Attenuation");
        auto mean = get_input2<float>("MeanNoise");

        auto tag = src_tag(zsSPG, attrTag);

        auto &spg = zsSPG->spg;
        auto block_cnt = spg.numBlocks();

        if (!spg.hasProperty(tag))
            throw std::runtime_error(fmt::format("GridAttribute [{}] doesn't exist!", tag.asString()));
        const int nchns = spg.getPropertySize(tag);

        auto pol = zs::cuda_exec();
        constexpr auto space = zs::execspace_e::cuda;

        zs::Vector<int> flag{1, zs::memsrc_e::um};
        pol(zs::Collapse{block_cnt, spg.block_size},
            [spgv = zs::proxy<space>(spg), tag, nchns, frequency = zs::vec<float, 3>::from_array(frequency),
             offset = zs::vec<float, 3>::from_array(offset), roughness, turbulence, amplitude, attenuation,
             mean] __device__(int blockno, int cellno) mutable {
                auto wcoord = spgv.wCoord(blockno, cellno);
                auto pp = frequency * wcoord - offset;

                float scale = amplitude;

                if (nchns == 3) {
                    // fractal Brownian motion
                    auto fbm = zs::vec<float, 3>::uniform(0);
                    for (int i = 0; i < turbulence; ++i, pp *= 2.f, scale *= roughness) {
                        zs::vec<float, 3> pln{ZSPerlinNoise1::perlin(pp[0], pp[1], pp[2]),
                                              ZSPerlinNoise1::perlin(pp[1], pp[2], pp[0]),
                                              ZSPerlinNoise1::perlin(pp[2], pp[0], pp[1])};
                        fbm += scale * pln;
                    }
                    auto noise = zs::vec<float, 3>{zs::pow(fbm[0], attenuation), zs::pow(fbm[1], attenuation),
                                                   zs::pow(fbm[2], attenuation)} +
                                 mean;

                    spgv._grid.tuple(zs::dim_c<3>, tag, blockno * spgv.block_size + cellno) =
                        spgv._grid.pack(zs::dim_c<3>, tag, blockno * spgv.block_size + cellno) + noise;
                } else if (nchns == 1) {
                    float fbm = 0;
                    for (int i = 0; i < turbulence; ++i, pp *= 2.f, scale *= roughness) {
                        float pln = ZSPerlinNoise1::perlin(pp[0], pp[1], pp[2]);
                        fbm += scale * pln;
                    }
                    auto noise = zs::pow(fbm, attenuation) + mean;

                    spgv(tag, blockno, cellno) += noise;
                }
            });

        set_output("SparseGrid", zsSPG);
    }
};

ZENDEFNODE(ZSGridPerlinNoise, {/* inputs: */
                               {"SparseGrid",
                                {"string", "GridAttribute", "v"},
                                {"vec3f", "Frequency", "1, 1, 1"},
                                {"vec3f", "Offset", "0, 0, 0"},
                                {"float", "Roughness", "0.5"},
                                {"int", "Turbulence", "4"},
                                {"float", "Amplitude", "1.0"},
                                {"float", "Attenuation", "1.0"},
                                {"float", "MeanNoise", "0"}},
                               /* outputs: */
                               {"SparseGrid"},
                               /* params: */
                               {},
                               /* category: */
                               {"Eulerian"}});

} // namespace zeno