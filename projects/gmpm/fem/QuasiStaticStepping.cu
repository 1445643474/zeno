#include "hip/hip_runtime.h"
#include "../Structures.hpp"
#include "zensim/Logger.hpp"
#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/geometry/PoissonDisk.hpp"
#include "zensim/geometry/VdbLevelSet.h"
#include "zensim/geometry/VdbSampler.h"
#include "zensim/io/MeshIO.hpp"
#include "zensim/math/bit/Bits.h"
#include "zensim/types/Property.h"
#include <atomic>
#include <zeno/VDBGrid.h>
#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>
#include <zeno/types/StringObject.h>

namespace zeno {
struct QuasiStaticStepping : INode {
  using T = float;
  using dtiles_t = zs::TileVector<T,32>;
  using tiles_t = typename ZenoParticles::particles_t;
  using vec3 = zs::vec<T, 3>;
  using mat3 = zs::vec<T, 3, 3>;
  struct FEMSystem {
    template <typename Pol, typename Model>
    T energy(Pol &pol, const Model &model,const zeno::vec<3,T>& g, const zs::SmallString tag, dtiles_t& vtemp) {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      Vector<T> res{verts.get_allocator(), 1};
      res.setVal(0);
    //   elastic potential
      pol(range(eles.size()), [verts = proxy<space>({}, verts),
                               eles = proxy<space>({}, eles),
                               vtemp = proxy<space>({}, vtemp),
                               res = proxy<space>(res), tag, model = model] 
                               ZS_LAMBDA (int ei) mutable {
        auto DmInv = eles.pack<3, 3>("IB", ei);
        auto inds = eles.pack<4>("inds", ei).reinterpret_bits<int>();
        vec3 xs[4] = {vtemp.pack<3>(tag, inds[0]), vtemp.pack<3>(tag, inds[1]),
                      vtemp.pack<3>(tag, inds[2]), vtemp.pack<3>(tag, inds[3])};
        mat3 F{};
        {
          auto x1x0 = xs[1] - xs[0];
          auto x2x0 = xs[2] - xs[0];
          auto x3x0 = xs[3] - xs[0];
          auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1],
                         x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
          F = Ds * DmInv;
        }
        auto psi = model.psi(F);
        auto vole = eles("vol", ei);

        atomic_add(exec_cuda, &res[0], vole * psi);
      });
    // gravity potential (TO DO, using per-element computation to speed up)
      pol(range(verts.size()),
            [verts = proxy<space>({},verts),vtemp = proxy<space>({},vtemp),res = proxy<space>(res),tag,g = vec3::from_array(g)]
            ZS_LAMBDA (int vi) mutable {
                auto m = verts("m",vi);
                auto v0 = vtemp.pack<3>(tag,vi);
                auto gpsi = -m * v0.dot(g); 
                atomic_add(exec_cuda, &res[0], gpsi);
      });
      return res.getVal();
    }

    template <typename Pol> void project(Pol &pol, const zs::SmallString tag, dtiles_t& vtemp) {
      using namespace zs;
      constexpr execspace_e space = execspace_e::cuda;
      // projection
      pol(zs::range(verts.size()),
          [vtemp = proxy<space>({}, vtemp), verts = proxy<space>({}, verts),
           tag] ZS_LAMBDA(int vi) mutable {
            if (verts("x", 1, vi) > 0.5)
              vtemp.tuple<3>(tag, vi) = vec3::zeros();
          });
    }


    template <typename Model>
    void computeGradientAndHessian(zs::CudaExecutionPolicy& cudaPol,
                                            const Model& model,
                                            const zeno::vec<3,T>& g,
                                            dtiles_t& vtemp,
                                            dtiles_t& etemp) {
        using namespace zs;
        constexpr auto space = execspace_e::cuda;
        cudaPol(zs::range(eles.size()), [vtemp = proxy<space>({}, vtemp),
                                        etemp = proxy<space>({}, etemp),
                                        verts = proxy<space>({}, verts),
                                        eles = proxy<space>({}, eles), model] ZS_LAMBDA (int ei) mutable {
            auto DmInv = eles.pack<3, 3>("IB", ei);
            auto dFdX = dFdXMatrix(DmInv);
            auto inds = eles.pack<4>("inds", ei).reinterpret_bits<int>();
            vec3 xs[4] = {vtemp.pack<3>("xn", inds[0]), vtemp.pack<3>("xn", inds[1]),
                            vtemp.pack<3>("xn", inds[2]), vtemp.pack<3>("xn", inds[3])};
            mat3 F{};
            {
                auto x1x0 = xs[1] - xs[0];
                auto x2x0 = xs[2] - xs[0];
                auto x3x0 = xs[3] - xs[0];
                auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1],
                            x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
                F = Ds * DmInv;
            }
            auto P = model.first_piola(F);
            auto vole = eles("vol", ei);
            auto vecP = flatten(P);
            auto dFdXT = dFdX.transpose();
            auto vfdt = -vole * (dFdXT * vecP);

            for (int i = 0; i != 4; ++i) {
                auto vi = inds[i];
                for (int d = 0; d != 3; ++d)
                atomic_add(exec_cuda, &vtemp("grad", d, vi), vfdt(i * 3 + d));
            }

            auto Hq = model.first_piola_derivative(F, true_c);
            auto H = dFdXT * Hq * dFdX * vole;

            etemp.tuple<12 * 12>("He", ei) = H;
        });

        cudaPol(zs::range(verts.size()),[   vtemp = proxy<space>({},vtemp),
                                            verts = proxy<space>({},verts),
                                            g = vec3::from_array(g)] ZS_LAMBDA (int vi) mutable {
            auto m = verts("m",vi);
            vtemp.tuple<3>("grad",vi) = vtemp.pack<3>("grad",vi) + m * g;
        });
    }
    template <typename Pol>
    void precondition(Pol &pol, const zs::SmallString srcTag,
                      const zs::SmallString dstTag,dtiles_t& vtemp) {
      using namespace zs;
      constexpr execspace_e space = execspace_e::cuda;
      // precondition
      pol(zs::range(verts.size()),
          [vtemp = proxy<space>({}, vtemp), verts = proxy<space>({}, verts),
           srcTag, dstTag] ZS_LAMBDA(int vi) mutable {
            vtemp.tuple<3>(dstTag, vi) =
                vtemp.pack<3, 3>("P", vi) * vtemp.pack<3>(srcTag, vi);
            // vtemp.tuple<3>(dstTag, vi) = vtemp.pack<3>(srcTag, vi);
          });
    }
    template <typename Pol>
    void multiply(Pol &pol, const zs::SmallString dxTag,
                  const zs::SmallString bTag,
                  dtiles_t& vtemp,
                  const dtiles_t& etemp) {
      using namespace zs;
      constexpr execspace_e space = execspace_e::cuda;
      constexpr auto execTag = wrapv<space>{};
      const auto numVerts = verts.size();
      const auto numEles = eles.size();
      // dx -> b
      pol(range(numVerts),
          [execTag, vtemp = proxy<space>({}, vtemp), bTag] ZS_LAMBDA(
              int vi) mutable { vtemp.tuple<3>(bTag, vi) = vec3::zeros(); });
      // elastic energy
      pol(range(numEles), [execTag, etemp = proxy<space>({}, etemp),
                           vtemp = proxy<space>({}, vtemp),
                           eles = proxy<space>({}, eles), dxTag, bTag] ZS_LAMBDA(int ei) mutable {
        constexpr int dim = 3;
        constexpr auto dimp1 = dim + 1;
        auto inds = eles.pack<dimp1>("inds", ei).reinterpret_bits<int>();
        zs::vec<T, dimp1 * dim> temp{};
        for (int vi = 0; vi != dimp1; ++vi)
          for (int d = 0; d != dim; ++d) {
            temp[vi * dim + d] = vtemp(dxTag, d, inds[vi]);
          }
        auto He = etemp.pack<dim * dimp1, dim * dimp1>("He", ei);

        temp = He * temp;

        for (int vi = 0; vi != dimp1; ++vi)
          for (int d = 0; d != dim; ++d) {
            atomic_add(execTag, &vtemp(bTag, d, inds[vi]), temp[vi * dim + d]);
          }
      });
    }

    FEMSystem(const tiles_t &verts, const tiles_t &eles)
        : verts{verts}, eles{eles}{}

    const tiles_t &verts;
    const tiles_t &eles;
    // dtiles_t &vtemp;
    // dtiles_t &etemp; 
  };

  template<int pack_dim = 3>
  T dot(zs::CudaExecutionPolicy &cudaPol, dtiles_t &vertData,
        const zs::SmallString tag0, const zs::SmallString tag1) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    Vector<T> res{vertData.get_allocator(), 1};
    res.setVal(0);
    cudaPol(range(vertData.size()),
            [data = proxy<space>({}, vertData), res = proxy<space>(res), tag0,
             tag1] __device__(int pi) mutable {
              auto v0 = data.pack<pack_dim>(tag0, pi);
              auto v1 = data.pack<pack_dim>(tag1, pi);
              atomic_add(exec_cuda, res.data(), v0.dot(v1));
            });
    return res.getVal();
  }
  T infNorm(zs::CudaExecutionPolicy &cudaPol, dtiles_t &vertData,
            const zs::SmallString tag = "dir") {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    Vector<T> res{vertData.get_allocator(), 1};
    res.setVal(0);
    cudaPol(range(vertData.size()),
            [data = proxy<space>({}, vertData), res = proxy<space>(res),
             tag] __device__(int pi) mutable {
              auto v = data.pack<3>(tag, pi);
              atomic_max(exec_cuda, res.data(), v.abs().max());
            });
    return res.getVal();
  }
  T avgForceRes(zs::CudaExecutionPolicy &cudaPol,const tiles_t &verts, dtiles_t &vertData,const zs::SmallString tag,const zeno::vec<3,T>& g) {
        using namespace zs;
        constexpr auto space = execspace_e::cuda;
        Vector<T> res{vertData.get_allocator(), 1};
        res.setVal(0);
        T gn = vec3::from_array(g).norm();
        cudaPol(range(vertData.size()),[data = proxy<space>({},vertData),verts = proxy<space>({},verts),tag,gn,res = proxy<space>(res)]
                ZS_LAMBDA(int vi) mutable {
                    auto ag = data.pack<3>(tag,vi).norm()/verts("m",vi)/gn;
                    atomic_add(exec_cuda,res.data(),ag);
                });
        return res.getVal()/verts.size();
  }

  void apply() override {
    using namespace zs;
    auto zstets = get_input<ZenoParticles>("ZSParticles");
    auto gravity = get_input<zeno::NumericObject>("gravity")->get<zeno::vec<3,T>>();
    auto armijo = get_param<float>("armijo");
    auto curvature = get_param<float>("wolfe");
    auto cg_res = get_param<float>("cg_res");
    auto btl_res = get_param<float>("btl_res");
    auto models = zstets->getModel();
    auto& verts = zstets->getParticles();
    auto& eles = zstets->getQuadraturePoints();

    static dtiles_t vtemp{verts.get_allocator(),
                          {{"grad", 3},
                           {"P", 9},
                           {"dir", 3},
                           {"xn", 3},
                           {"xn0", 3},
                           {"xtilde", 3},
                           {"temp", 3},
                           {"r", 3},
                           {"p", 3},
                           {"q", 3}},
                          verts.size()};
    static dtiles_t etemp{eles.get_allocator(), {{"He", 12 * 12}}, eles.size()};
    vtemp.resize(verts.size());
    etemp.resize(eles.size());

    FEMSystem A{verts,eles};

    constexpr auto space = execspace_e::cuda;
    auto cudaPol = cuda_exec();

    // use the previous simulation result as initial guess
    cudaPol(zs::range(vtemp.size()),
              [vtemp = proxy<space>({},vtemp), verts = proxy<space>({},verts)]
                  __device__(int i) mutable{
                auto x = verts.pack<3>("x",i);
                vtemp.tuple<3>("xtilde",i) = x;
    });

    cudaPol(zs::range(verts.size()),
            [vtemp = proxy<space>({}, vtemp),
             verts = proxy<space>({}, verts)] __device__(int vi) mutable {
              auto x = verts.pack<3>("x", vi);
              vtemp.tuple<3>("xn", vi) = x;
            });


    for(int newtonIter = 0;newtonIter != 100;++newtonIter){
      cudaPol(zs::range(vtemp.size()),
            [vtemp = proxy<space>({},vtemp),verts = proxy<space>({},verts)]
              __device__(int i) mutable {
                vtemp.tuple<3>("grad",i) = vec3{0,0,0};
      });
    //   fmt::print("COMPUTE GRADIENT AND HESSIAN\n",newtonIter);
    //   fmt::print("gravity_n:{}\n",gravity)
      match([&](auto &elasticModel) {
        A.computeGradientAndHessian(cudaPol, elasticModel, gravity,vtemp,etemp);
      })(models.getElasticModel());

    //   T Hn = dot<144>(cudaPol,etemp,"He","He");
    //   fmt::print("Hn:{}\n",Hn);

    //   fmt::print("prepare Preconditioner \n",newtonIter);
  //  Prepare Preconditioning
      cudaPol(zs::range(vtemp.size()),
          [vtemp = proxy<space>({}, vtemp),
            verts = proxy<space>({}, verts)] ZS_LAMBDA (int vi) mutable {
                vtemp.tuple<9>("P", vi) = mat3::zeros();
      });

    

      cudaPol(zs::range(eles.size()),
                [vtemp = proxy<space>({},vtemp),etemp = proxy<space>({},etemp),eles = proxy<space>({},eles)]
                  ZS_LAMBDA (int ei) mutable {
                    constexpr int dim = 3;
                    constexpr auto dimp1 = dim + 1;
                    auto inds = 
                        eles.template pack<dimp1>("inds",ei).template reinterpret_bits<int>();
                    auto He = etemp.pack<dim * dimp1,dim * dimp1>("He",ei);
                    for (int vi = 0; vi != dimp1; ++vi) {
                    #if 1
                      for (int i = 0; i != dim; ++i)
                        for (int j = 0; j != dim; ++j) {
                          atomic_add(exec_cuda, &vtemp("P", i * dim + j, inds[vi]),
                                    He(vi * dim + i, vi * dim + j));
                        }
                    #else
                      for (int j = 0; j != dim; ++j) {
                          atomic_add(exec_cuda, &vtemp("P", j * dim + j, inds[vi]),
                                    He(vi * dim + j, vi * dim + j));
                      }
                    #endif
                    }
      });


    // fmt::print("FOUND NON_SPD P\n");
    // cudaPol(zs::range(vtemp.size()),
    //     [vtemp = proxy<space>({},vtemp)] ZS_LAMBDA(int vi){
    //         auto P = vtemp.pack<3,3>("P",vi);
    //         if(vi == 4966){
    //             // if(P(0,0) < 0 || P(1,1) < 0 || P(2,2) < 0) {
    //                 printf("NON_SPD_P<%d> : \n%f\t%f\t%f\n%f\t%f\t%f\n%f\t%f\t%f\n",vi,
    //                     (float)P(0,0),(float)P(0,1),(float)P(0,2),(float)P(1,0),(float)P(1,1),(float)P(1,2),(float)P(2,0),(float)P(2,1),(float)P(2,2)
    //                 );
    //             // }
    //         }
    //     });

    //   T Pn = dot<9>(cudaPol,vtemp,"P","P");
    //   fmt::print("P_n:{}\n",Pn);

      cudaPol(zs::range(vtemp.size()),
              [vtemp = proxy<space>({},vtemp)] __device__(int i) mutable {
                vtemp.tuple<9>("P",i) = inverse(vtemp.pack<3,3>("P",i));
      });

        // fmt::print("FOUND NON_SPD PINV\n");
        // cudaPol(zs::range(vtemp.size()),
        //     [vtemp = proxy<space>({},vtemp)] ZS_LAMBDA(int vi){
        //         auto P = vtemp.pack<3,3>("P",vi);
        //         // if(vi == 4966){
        //             if(P(0,0) < 0 || P(1,1) < 0 || P(2,2) < 0) {
        //                 printf("NON_SPD_PINV<%d> : \n%f\t%f\t%f\n%f\t%f\t%f\n%f\t%f\t%f\n",vi,
        //                     (float)P(0,0),(float)P(0,1),(float)P(0,2),(float)P(1,0),(float)P(1,1),(float)P(1,2),(float)P(2,0),(float)P(2,1),(float)P(2,2)
        //                 );
        //             }
        //         // }
        //     });

    //   Pn = dot<9>(cudaPol,vtemp,"P","P");
    //   fmt::print("Piv_n:{}\n",Pn);

    //   fmt::print("Solve Ax = b using PCG \n",newtonIter);

      // if the grad is too small, return the result
      // Solve equation using PCG
      {
        // solve for A dir = grad;
        cudaPol(zs::range(vtemp.size()),
                [vtemp = proxy<space>({}, vtemp)] __device__(int i) mutable {
                  vtemp.tuple<3>("dir", i) = vec3::zeros();
                });
        // {
        //     auto dirD = dot(cudaPol, vtemp, "dir", "dir");
        //     fmt::print("dir norm: {}\n", dirD);
        //     auto tmp = dot(cudaPol, vtemp, "grad", "grad");
        //     fmt::print("grad norm: {}\n", tmp);
        // }
        // temp = A * dir
        A.multiply(cudaPol, "dir", "temp",vtemp,etemp);
        // auto AdNorm = dot(cudaPol,vtemp,"temp","temp");
        // fmt::print("AdNorm: {}\n",AdNorm);
        // r = grad - temp
        cudaPol(zs::range(vtemp.size()),
                [vtemp = proxy<space>({}, vtemp)] __device__(int i) mutable {
                  vtemp.tuple<3>("r", i) =
                      vtemp.pack<3>("grad", i) - vtemp.pack<3>("temp", i);
                });
        A.project(cudaPol, "r",vtemp);
        A.precondition(cudaPol, "r", "q",vtemp); // q has the unit of length
        cudaPol(zs::range(vtemp.size()),
            [vtemp = proxy<space>({}, vtemp)] __device__(int i) mutable {
                vtemp.tuple<3>("p", i) = vtemp.pack<3>("q", i);
        });



        T zTrk = dot(cudaPol,vtemp,"r","q");

        if(std::isnan(zTrk)){
            T rn = std::sqrt(dot(cudaPol,vtemp,"r","r"));
            T qn = std::sqrt(dot(cudaPol,vtemp,"q","q"));
            T gn = std::sqrt(dot(cudaPol,vtemp,"grad","grad"));
            T Pn = std::sqrt(dot<9>(cudaPol,vtemp,"P","P"));

            fmt::print("NAN zTrk Detected r: {} q: {}, gn:{} Pn:{}\n",rn,qn,gn,Pn);
            throw std::runtime_error("NAN zTrk");
        }

        // if(zTrk < 1e-12){
        //     T rn = std::sqrt(dot(cudaPol,vtemp,"r","r"));
        //     T qn = std::sqrt(dot(cudaPol,vtemp,"q","q"));
        //     fmt::print("\t# newton optimizer ends in {} iters with zTrk {} and grad {}\n",
        //     newtonIter, zTrk, infNorm(cudaPol, vtemp, "grad"));
        //     break;
        // }
        if(zTrk < 0){
            T rn = std::sqrt(dot(cudaPol,vtemp,"r","r"));
            T qn = std::sqrt(dot(cudaPol,vtemp,"q","q"));
            fmt::print("\t# invalid zTrk found in {} iters with zTrk {} and r {} and q {}\n",
                newtonIter, zTrk, infNorm(cudaPol, vtemp, "grad"),rn,qn);

            fmt::print("FOUND NON_SPD P\n");
            cudaPol(zs::range(vtemp.size()),
                [vtemp = proxy<space>({},vtemp)] ZS_LAMBDA(int vi){
                    auto P = vtemp.pack<3,3>("P",vi);
                    if(P(0,0) < 0 || P(1,1) < 0 || P(2,2) < 0) {
                        printf("NON_SPD_P<%d> : \n%f\t%f\t%f\n%f\t%f\t%f\n%f\t%f\t%f\n",
                            P(0,0),P(0,1),P(0,2),P(1,0),P(1,1),P(1,2),P(2,0),P(2,1),P(2,2)
                        );
                    }
                });

            

            throw std::runtime_error("INVALID zTrk");
        }
        auto residualPreconditionedNorm = std::sqrt(zTrk);
        // auto localTol = std::min(0.5 * residualPreconditionedNorm, 1.0);
        auto localTol = 0.1 * residualPreconditionedNorm;
        // if(newtonIter < 10)
        //     localTol = 0.5 * residualPreconditionedNorm;
        int iter = 0;
        for (; iter != 1000; ++iter) {
          if (iter % 200 == 0)
            fmt::print("cg iter: {}, norm: {} zTrk: {} localTol: {}\n", iter,
                        residualPreconditionedNorm,zTrk,localTol);
          
            if(zTrk < 0){
                T rn = std::sqrt(dot(cudaPol,vtemp,"r","r"));
                T qn = std::sqrt(dot(cudaPol,vtemp,"q","q"));
                fmt::print("\t# invalid zTrk found in {} iters with zTrk {} and r {} and q {}\n",
                    iter, zTrk,rn,qn);


                throw std::runtime_error("INVALID zTrk");
            }

          if (residualPreconditionedNorm <= localTol){ // this termination criterion is dimensionless
            // T dg = dot(cudaPol,vtemp,"grad","dir");
            // if(dg > 0)
                fmt::print("finish with cg iter: {}, norm: {} zTrk: {}\n", iter,
                            residualPreconditionedNorm,zTrk);
          
                break;
          }
          A.multiply(cudaPol, "p", "temp",vtemp,etemp);
          A.project(cudaPol, "temp",vtemp);

          T alpha = zTrk / dot(cudaPol, vtemp, "temp", "p");
          cudaPol(range(verts.size()), [verts = proxy<space>({}, verts),
                                        vtemp = proxy<space>({}, vtemp),
                                        alpha] ZS_LAMBDA(int vi) mutable {
            vtemp.tuple<3>("dir", vi) =
                vtemp.pack<3>("dir", vi) + alpha * vtemp.pack<3>("p", vi);
            vtemp.tuple<3>("r", vi) =
                vtemp.pack<3>("r", vi) - alpha * vtemp.pack<3>("temp", vi);
          });

          A.precondition(cudaPol, "r", "q",vtemp);
          auto zTrkLast = zTrk;
          zTrk = dot(cudaPol, vtemp, "q", "r");
          auto beta = zTrk / zTrkLast;
          cudaPol(range(verts.size()), [vtemp = proxy<space>({}, vtemp),
                                        beta] ZS_LAMBDA(int vi) mutable {
            vtemp.tuple<3>("p", vi) =
                vtemp.pack<3>("q", vi) + beta * vtemp.pack<3>("p", vi);
          });
          residualPreconditionedNorm = std::sqrt(zTrk);
        } // end cg step
        fmt::print("FINISH SOLVING PCG with cg_iter = {}\n",iter);  
      }    
    // in case
      A.project(cudaPol,"dir",vtemp);
      A.project(cudaPol,"grad",vtemp);
      T res = infNorm(cudaPol, vtemp, "dir");// this norm is independent of descriterization

    //   fmt::print("NEWTON_ITER<{}> with gradn: {} and dirn: {}\n",newtonIter,gradn,res);

      if (res < 1e-2) {
        T gradn = avgForceRes(cudaPol,verts,vtemp,"grad",gravity);
        // infNorm(cudaPol, vtemp, "grad")/(infNorm(cudaPol,eles,));
        fmt::print("\t# newton optimizer reach desired resolution in {} iters with residual {} and grad {}\n",
                   newtonIter, res, gradn);
        break;
      }


      T dg = dot(cudaPol,vtemp,"grad","dir");
      if(fabs(dg) < btl_res){
        T gradn = avgForceRes(cudaPol,verts,vtemp,"grad",gravity);
        fmt::print("\t# newton optimizer reach stagnation point in {} iters with residual {} and grad {}\n",
        newtonIter, res, gradn);
        break;
      }
      if(dg < 0){
          T gradn = std::sqrt(dot(cudaPol,vtemp,"grad","grad"));
          T dirn = std::sqrt(dot(cudaPol,vtemp,"dir","dir"));
          fmt::print("invalid dg = {} grad = {} dir = {}\n",dg);
          throw std::runtime_error("INVALID DESCENT DIRECTION");
      }

      fmt::print("DO LINE SEARCH\n");
      // line search
      T alpha = 1.;
      cudaPol(zs::range(vtemp.size()),
              [vtemp = proxy<space>({}, vtemp)] __device__(int i) mutable {
                vtemp.tuple<3>("xn0", i) = vtemp.pack<3>("xn", i);
              });
      T E0;
      match([&](auto &elasticModel) {
        E0 = A.energy(cudaPol, elasticModel,gravity, "xn0",vtemp);
      })(models.getElasticModel());


      dg = -dg;

      T E{E0};
    //   Backtracking Linesearch
      int max_line_search = 10;
      int line_search = 0;
      std::vector<T> armijo_buffer(max_line_search);
      do {
        cudaPol(zs::range(vtemp.size()), [vtemp = proxy<space>({}, vtemp),
                                          alpha] __device__(int i) mutable {
          vtemp.tuple<3>("xn", i) =
              vtemp.pack<3>("xn0", i) + alpha * vtemp.pack<3>("dir", i);
        });
        match([&](auto &elasticModel) {
          E = A.energy(cudaPol, elasticModel,gravity, "xn",vtemp);
        })(models.getElasticModel());
        // fmt::print("E: {} at alpha {}. E0 {}\n", E, alpha, E0);
        // fmt::print("Armijo : {} < {}\n",(E - E0)/alpha,dg);
        armijo_buffer[line_search] = (E - E0)/alpha;
        // test Armojo condition
        if (E - E0 < armijo * dg * alpha)
          break;
        alpha /= 2;
        ++line_search;
      } while (line_search < max_line_search);

    //   fmt::print("FINISH LINE SEARCH WITH LINE_SEARCH = {}\n",line_search);

      if(line_search == max_line_search){
          fmt::print("LINE_SEARCH_EXCEED:\n");
          for(size_t i = 0;i != max_line_search;++i)
            fmt::print("AB[{}]\t = {} dg = {}\n",i,armijo_buffer[i],dg);
      }

      cudaPol(zs::range(vtemp.size()), [vtemp = proxy<space>({}, vtemp),
                                        alpha] __device__(int i) mutable {
        vtemp.tuple<3>("xn", i) =
            vtemp.pack<3>("xn0", i) + alpha * vtemp.pack<3>("dir", i);
      });
    
    }

    cudaPol(zs::range(verts.size()),
            [vtemp = proxy<space>({}, vtemp), verts = proxy<space>({}, verts)] __device__(int vi) mutable {
              auto newX = vtemp.pack<3>("xn", vi);
              verts.tuple<3>("x", vi) = newX;
            });


    set_output("ZSParticles", std::move(zstets));
  }
};

ZENDEFNODE(QuasiStaticStepping, {{"ZSParticles","gravity"},
                                  {"ZSParticles"},
                                  {{"float","armijo","0.1"},{"float","wolfe","0.9"},{"float","cg_res","0.1"},{"float","btl_res","0.0001"}},
                                  {"FEM"}});

}