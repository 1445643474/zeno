#include "hip/hip_runtime.h"
#include "../Utils.hpp"
#include "Structures.hpp"

#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/geometry/VdbSampler.h"
#include "zensim/io/ParticleIO.hpp"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/tpls/fmt/color.h"
#include "zensim/tpls/fmt/format.h"
#include <zeno/types/DictObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>

namespace zeno {

struct ComputeParticleVolume : INode {
  void apply() override {
    auto parObjPtrs = RETRIEVE_OBJECT_PTRS(ZenoParticles, "ZSParticles");
    auto zsgrid = get_input<ZenoGrid>("ZSGrid");
    auto &grid = zsgrid->get();

    auto buckets = std::make_shared<ZenoIndexBuckets>();
    auto &ibs = buckets->get();

    using namespace zs;
    auto cudaPol = cuda_exec().device(0);
    bool first = true;

    for (auto &&parObjPtr : parObjPtrs)
      if (parObjPtr->category == ZenoParticles::mpm) {
        auto &pars = parObjPtr->getParticles();
        spatial_hashing(cudaPol, pars, grid.dx, ibs, first, true);
        first = false;
      }

    for (auto &&parObjPtr : parObjPtrs) {
      if (parObjPtr->category != ZenoParticles::mpm)
        continue;
      auto &pars = parObjPtr->getParticles();
      cudaPol(range(pars.size()),
              [pars = proxy<execspace_e::cuda>({}, pars),
               ibs = proxy<execspace_e::cuda>(ibs),
               density = parObjPtr->getModel().density,
               cellVol =
                   grid.dx * grid.dx * grid.dx] __device__(size_t pi) mutable {
                auto pos = pars.template pack<3>("pos", pi);
                auto coord = ibs.bucketCoord(pos);
                const auto bucketNo = ibs.table.query(coord);
                // bucketNo should be > 0
                const auto cnt = ibs.counts[bucketNo];
                const auto vol = cellVol / cnt;
                pars("vol", pi) = vol;
                pars("mass", pi) = density * vol;
              });
    }
    set_output("ZSParticles", get_input("ZSParticles"));
  }
};

ZENDEFNODE(ComputeParticleVolume,
           {
               {{"ZenoParticles", "ZSParticles"}, {"ZenoGrid", "ZSGrid"}},
               {{"ZenoParticles", "ZSParticles"}},
               {},
               {"MPM"},
           });

struct PushOutZSParticles : INode {
  template <typename LsView>
  void pushout(zs::CudaExecutionPolicy &cudaPol,
               typename ZenoParticles::particles_t &pars, LsView lsv) {
    using namespace zs;
    cudaPol(range(pars.size()), [pars = proxy<execspace_e::cuda>({}, pars), lsv,
                                 eps = limits<float>::epsilon() *
                                       128] __device__(size_t pi) mutable {
      auto x = pars.pack<3>("pos", pi);
      bool updated = false;
      int cnt = 5;
      for (auto sd = lsv.getSignedDistance(x); sd < 0 && cnt--;) {
        auto diff = x.zeros();
        for (int i = 0; i != 3; i++) {
          auto v1 = x;
          auto v2 = x;
          v1[i] = x[i] + eps;
          v2[i] = x[i] - eps;
          diff[i] = (lsv.getSignedDistance(v1) - lsv.getSignedDistance(v2)) /
                    (eps + eps);
        }
        if (math::near_zero(diff.l2NormSqr()))
          break;
        auto n = diff.normalized();
        x -= n * sd;
        auto newSd = lsv.getSignedDistance(x);
        if (newSd < sd ||
            zs::abs(newSd - sd) < -sd) // new position should be no deeper
          break;
        updated = true;
        sd = newSd;
      }
      if (updated)
        pars.tuple<3>("pos", pi) = x;
    });
  }
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing PushOutZSParticles\n");
    auto parObjPtrs = RETRIEVE_OBJECT_PTRS(ZenoParticles, "ZSParticles");
    using namespace zs;
    auto cudaPol = cuda_exec().device(0);
    auto zsls = get_input<ZenoLevelSet>("ZSLevelSet");

    for (auto &&parObjPtr : parObjPtrs) {
      auto &pars = parObjPtr->getParticles();
      using basic_ls_t = typename ZenoLevelSet::basic_ls_t;
      using sdf_vel_ls_t = typename ZenoLevelSet::sdf_vel_ls_t;
      using transition_ls_t = typename ZenoLevelSet::transition_ls_t;
      match(
          [&](basic_ls_t &ls) {
            match([&](const auto &lsPtr) {
              auto lsv = get_level_set_view<execspace_e::cuda>(lsPtr);
              pushout(cudaPol, pars, lsv);
            })(ls._ls);
          },
          [&](sdf_vel_ls_t &ls) {
            match([&](auto lsv) {
              pushout(cudaPol, pars, SdfVelFieldView{lsv});
            })(ls.template getView<execspace_e::cuda>());
          },
          [&](transition_ls_t &ls) {
            auto [fieldViewSrc, fieldViewDst] =
                ls.template getView<zs::execspace_e::cuda>();
            match(
                [&](auto fvSrc, auto fvDst)
                    -> std::enable_if_t<
                        is_same_v<RM_CVREF_T(fvSrc), RM_CVREF_T(fvDst)>> {
                  pushout(cudaPol, pars,
                          TransitionLevelSetView{SdfVelFieldView{fvSrc},
                                                 SdfVelFieldView{fvDst},
                                                 ls._stepDt, ls._alpha});
                },
                [](...) {})(fieldViewSrc, fieldViewDst);
          })(zsls->getLevelSet());
    }

    fmt::print(fg(fmt::color::cyan), "done executing PushOutZSParticles\n");
    set_output("ZSParticles", get_input("ZSParticles"));
  }
};
ZENDEFNODE(PushOutZSParticles, {
                                   {"ZSParticles", "ZSLevelSet"},
                                   {"ZSParticles"},
                                   {},
                                   {"MPM"},
                               });

} // namespace zeno
