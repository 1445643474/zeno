#include "hip/hip_runtime.h"
#include "../Structures.hpp"
#include "../Utils.hpp"

#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/geometry/VdbSampler.h"
#include "zensim/io/ParticleIO.hpp"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/tpls/fmt/color.h"
#include "zensim/tpls/fmt/format.h"
#include <zeno/types/DictObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>

namespace zeno {

struct ConfigConstitutiveModel : INode {
  void apply() override {
    auto out = std::make_shared<ZenoConstitutiveModel>();

    float dx = get_input2<float>("dx");

    // volume
    out->volume = dx * dx * dx / get_input2<float>("ppc");
    out->dx = dx;

    // density
    out->density = get_input2<float>("density");

    // constitutive models
    auto params = has_input("params") ? get_input<DictObject>("params")
                                      : std::make_shared<DictObject>();
    float E = get_input2<float>("E");

    float nu = get_input2<float>("nu");

    auto typeStr = get_input2<std::string>("type");
    // elastic model
    auto &model = out->getElasticModel();

    if (typeStr == "fcr")
      model = zs::FixedCorotated<float>{E, nu};
    else if (typeStr == "nhk")
      model = zs::NeoHookean<float>{E, nu};
    else if (typeStr == "stvk")
      model = zs::StvkWithHencky<float>{E, nu};
    else
      throw std::runtime_error(fmt::format(
          "unrecognized (isotropic) elastic model [{}]\n", typeStr));

    // aniso elastic model
    const auto get_arg = [&params](const char *const tag, auto type) {
      using T = typename RM_CVREF_T(type)::type;
      std::optional<T> ret{};
      if (auto it = params->lut.find(tag); it != params->lut.end())
        ret = safe_any_cast<T>(it->second);
      return ret;
    };
    auto anisoTypeStr = get_input2<std::string>("aniso");
    if (anisoTypeStr == "arap") { // a (fiber direction)
      float strength = get_arg("strength", zs::wrapt<float>{}).value_or(10.f);
      out->getAnisoElasticModel() = zs::AnisotropicArap<float>{E, nu, strength};
    } else
      out->getAnisoElasticModel() = std::monostate{};

    // plastic model
    auto plasticTypeStr = get_input2<std::string>("plasticity");
    if (plasticTypeStr == "nadp") {
      model = zs::StvkWithHencky<float>{E, nu};
      float fa = get_arg("friction_angle", zs::wrapt<float>{}).value_or(35.f);
      out->getPlasticModel() = zs::NonAssociativeDruckerPrager<float>{fa};
    } else if (plasticTypeStr == "navm") {
      model = zs::StvkWithHencky<float>{E, nu};
      float ys = get_arg("yield_stress", zs::wrapt<float>{}).value_or(1e5f);
      out->getPlasticModel() = zs::NonAssociativeVonMises<float>{ys};
    } else if (plasticTypeStr == "nacc") { // logjp
      model = zs::StvkWithHencky<float>{E, nu};
      float fa = get_arg("friction_angle", zs::wrapt<float>{}).value_or(35.f);
      float beta = get_arg("beta", zs::wrapt<float>{}).value_or(2.f);
      float xi = get_arg("xi", zs::wrapt<float>{}).value_or(1.f);
      out->getPlasticModel() =
          zs::NonAssociativeCamClay<float>{fa, beta, xi, 3, true};
    } else
      out->getPlasticModel() = std::monostate{};

    set_output("ZSModel", out);
  }
};

ZENDEFNODE(ConfigConstitutiveModel,
           {
               {{"float", "dx", "0.1"},
                {"float", "ppc", "8"},
                {"float", "density", "1000"},
                {"string", "type", "fcr"},
                {"string", "aniso", "none"},
                {"string", "plasticity", "none"},
                {"float", "E", "10000"},
                {"float", "nu", "0.4"},
                {"DictObject:NumericObject", "params"}},
               {"ZSModel"},
               {},
               {"MPM"},
           });

struct ToZSParticles : INode {
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing ToZensimParticles\n");
    auto model = get_input<ZenoConstitutiveModel>("ZSModel");

    // primitive
    auto inParticles = get_input<PrimitiveObject>("prim");
    auto &obj = inParticles->attr<vec3f>("pos");
    vec3f *velsPtr{nullptr};
    if (inParticles->has_attr("vel"))
      velsPtr = inParticles->attr<vec3f>("vel").data();
    vec3f *nrmsPtr{nullptr};
    if (inParticles->has_attr("nrm"))
      nrmsPtr = inParticles->attr<vec3f>("nrm").data();
    auto &quads = inParticles->quads;
    auto &tris = inParticles->tris;
    auto &lines = inParticles->lines;

    auto outParticles = std::make_shared<ZenoParticles>();

    // primitive binding
    outParticles->prim = inParticles;
    // model
    outParticles->getModel() = *model;

    /// category, size
    std::size_t size{obj.size()};
    // (mesh）
    std::size_t eleSize{0};
    std::vector<float> dofVol{};
    std::vector<float> eleVol{};
    std::vector<vec3f> elePos{};
    std::vector<vec3f> eleVel{};
    std::vector<std::array<vec3f, 3>> eleD{};

    ZenoParticles::category_e category{ZenoParticles::mpm};
    bool bindMesh = get_input2<int>("category") != ZenoParticles::mpm;
    if (bindMesh) {
      if (quads.size()) {
        category = ZenoParticles::tet;
        eleSize = quads.size();
      } else if (tris.size()) {
        category = ZenoParticles::surface;
        eleSize = tris.size();
      } else if (lines.size()) {
        category = ZenoParticles::curve;
        eleSize = lines.size();
      } else
        throw std::runtime_error("unable to deduce primitive manifold type.");

      dofVol.resize(size, 0.f);

      eleVol.resize(eleSize);
      elePos.resize(eleSize);
      eleVel.resize(eleSize);
      eleD.resize(eleSize);
    }
    outParticles->category = category;

    // per vertex (node) vol, pos, vel
    using namespace zs;
    auto ompExec = zs::omp_exec();

    if (bindMesh) {
      switch (category) {
      // tet
      case ZenoParticles::tet: {
        const auto tetVol = [&obj](vec4i quad) {
          const auto &p0 = obj[quad[0]];
          auto s = cross(obj[quad[2]] - p0, obj[quad[1]] - p0);
          return std::abs(dot(s, obj[quad[3]] - p0)) / 6;
        };
        for (std::size_t i = 0; i != eleSize; ++i) {
          auto quad = quads[i];
          auto v = tetVol(quad);

          eleVol[i] = v;
          elePos[i] =
              (obj[quad[0]] + obj[quad[1]] + obj[quad[2]] + obj[quad[3]]) / 4;
          if (velsPtr)
            eleVel[i] = (velsPtr[quad[0]] + velsPtr[quad[1]] +
                         velsPtr[quad[2]] + velsPtr[quad[3]]) /
                        4;
          eleD[i][0] = obj[quad[1]] - obj[quad[0]];
          eleD[i][1] = obj[quad[2]] - obj[quad[0]];
          eleD[i][2] = obj[quad[3]] - obj[quad[0]];
          for (auto pi : quad)
            dofVol[pi] += v / 4;
        }
      } break;
      // surface
      case ZenoParticles::surface: {
        const auto triArea = [&obj](vec3i tri) {
          using TV3 = zs::vec<float, 3>;
          TV3 p0 = TV3{obj[tri[0]][0], obj[tri[0]][1], obj[tri[0]][2]};
          TV3 p1 = TV3{obj[tri[1]][0], obj[tri[1]][1], obj[tri[1]][2]};
          TV3 p2 = TV3{obj[tri[2]][0], obj[tri[2]][1], obj[tri[2]][2]};
          return (p1 - p0).cross(p2 - p0).norm() * 0.5f;
          // const auto &p0 = obj[tri[0]];
          // return length(cross(obj[tri[1]] - p0, obj[tri[2]] - p0)) * 0.5;
        };
        for (std::size_t i = 0; i != eleSize; ++i) {
          auto tri = tris[i];
          auto v = triArea(tri) * model->dx;
#if 0
          if (i <= 3) {
            for (auto pi : tri)
              fmt::print("vi[{}]: {}, {}, {}\n", pi, obj[pi][0], obj[pi][1],
                         obj[pi][2]);
            fmt::print("tri area: {}, volume: {}, dx: {}\n", triArea(tri), v,
                       model->dx);
            getchar();
          }
#endif
          eleVol[i] = v;
          elePos[i] = (obj[tri[0]] + obj[tri[1]] + obj[tri[2]]) / 3;
          if (velsPtr)
            eleVel[i] =
                (velsPtr[tri[0]] + velsPtr[tri[1]] + velsPtr[tri[2]]) / 3;
          eleD[i][0] = obj[tri[1]] - obj[tri[0]];
          eleD[i][1] = obj[tri[2]] - obj[tri[0]];
          eleD[i][2] = normalize(cross(eleD[i][0], eleD[i][1]));
          for (auto pi : tri)
            dofVol[pi] += v / 3;
        }
      } break;
      // curve
      case ZenoParticles::curve: {
        const auto lineLength = [&obj](vec2i line) {
          return length(obj[line[1]] - obj[line[0]]);
        };
        for (std::size_t i = 0; i != eleSize; ++i) {
          auto line = lines[i];
          auto v = lineLength(line) * model->dx * model->dx;
          eleVol[i] = v;
          elePos[i] = (obj[line[0]] + obj[line[1]]) / 2;
          if (velsPtr)
            eleVel[i] = (velsPtr[line[0]] + velsPtr[line[1]]) / 2;
          eleD[i][0] = obj[line[1]] - obj[line[0]];
          if (auto n = cross(vec3f{0, 1, 0}, eleD[i][0]);
              lengthSquared(n) > zs::limits<float>::epsilon() * 128) {
            eleD[i][1] = normalize(n);
          } else
            eleD[i][1] = normalize(cross(vec3f{1, 0, 0}, eleD[i][0]));
          eleD[i][2] = normalize(cross(eleD[i][0], eleD[i][1]));
          for (auto pi : line)
            dofVol[pi] += v / 2;
        }
      } break;
      default:;
      } // end switch
    }   // end bindmesh

    // particles
    auto &pars = outParticles->getParticles(); // tilevector

    // attributes
    std::vector<zs::PropertyTag> tags{{"mass", 1}, {"pos", 3}, {"vel", 3},
                                      {"vol", 1},  {"C", 9},   {"vms", 1}};
    std::vector<zs::PropertyTag> eleTags{
        {"mass", 1}, {"pos", 3},  {"vel", 3},
        {"vol", 1},  {"C", 9},    {"F", 9},
        {"d", 9},    {"Dinv", 9}, {"inds", (int)category + 1}};

    const bool hasLogJp = model->hasLogJp();
    const bool hasOrientation = model->hasOrientation();
    const bool hasF = model->hasF();

    if (hasF)
      tags.emplace_back(zs::PropertyTag{"F", 9});
    else {
      tags.emplace_back(zs::PropertyTag{"J", 1});
      if (category != ZenoParticles::mpm)
        throw std::runtime_error(
            "mesh particles should not use the 'J' attribute.");
    }

    if (hasOrientation) {
      tags.emplace_back(zs::PropertyTag{"a", 3});
      if (category != ZenoParticles::mpm)
        //
        ;
    }

    if (hasLogJp) {
      tags.emplace_back(zs::PropertyTag{"logJp", 1});
      if (category != ZenoParticles::mpm)
        //
        ;
    }

    // prim attrib tags
    std::vector<zs::PropertyTag> auxAttribs{};
    for (auto &&[key, arr] : inParticles->verts.attrs) {
      const auto checkDuplication = [&tags](const std::string &name) {
        for (std::size_t i = 0; i != tags.size(); ++i)
          if (tags[i].name == name.data())
            return true;
        return false;
      };
      if (checkDuplication(key))
        continue;
      const auto &k{key};
      match(
          [&k, &auxAttribs](const std::vector<vec3f> &vals) {
            auxAttribs.push_back(PropertyTag{k, 3});
          },
          [&k, &auxAttribs](const std::vector<float> &vals) {
            auxAttribs.push_back(PropertyTag{k, 1});
          },
          [&k, &auxAttribs](const std::vector<vec3i> &vals) {},
          [&k, &auxAttribs](const std::vector<int> &vals) {},
          [](...) {
            throw std::runtime_error(
                "what the heck is this type of attribute!");
          })(arr);
    }
    tags.insert(std::end(tags), std::begin(auxAttribs), std::end(auxAttribs));

    fmt::print(
        "{} elements in process. pending {} particles with these attributes.\n",
        eleSize, size);
    for (auto tag : tags)
      fmt::print("tag: [{}, {}]\n", tag.name, tag.numChannels);

    {
      pars = typename ZenoParticles::particles_t{tags, size, memsrc_e::host};
      ompExec(zs::range(size), [pars = proxy<execspace_e::host>({}, pars),
                                hasLogJp, hasOrientation, hasF, &model, &obj,
                                velsPtr, nrmsPtr, &dofVol, category,
                                &inParticles, &auxAttribs](size_t pi) mutable {
        using vec3 = zs::vec<float, 3>;
        using mat3 = zs::vec<float, 3, 3>;

        // volume, mass
        float vol = category == ZenoParticles::mpm ? model->volume : dofVol[pi];
        pars("vol", pi) = vol;
        pars("mass", pi) = vol * model->density;

        // pos
        pars.tuple<3>("pos", pi) = obj[pi];

        // vel
        if (velsPtr != nullptr)
          pars.tuple<3>("vel", pi) = velsPtr[pi];
        else
          pars.tuple<3>("vel", pi) = vec3::zeros();

        // deformation
        if (hasF)
          pars.tuple<9>("F", pi) = mat3::identity();
        else
          pars("J", pi) = 1.;

        // apic transfer
        pars.tuple<9>("C", pi) = mat3::zeros();

        // orientation
        if (hasOrientation) {
          if (nrmsPtr != nullptr) {
            const auto n_ = nrmsPtr[pi];
            const auto n = vec3{n_[0], n_[1], n_[2]};
            constexpr auto up = vec3{0, 1, 0};
            if (!parallel(n, up)) {
              auto side = cross(up, n);
              auto a = cross(side, n);
              pars.tuple<3>("a", pi) = a;
            } else
              pars.tuple<3>("a", pi) = vec3{0, 0, 1};
          } else
            pars.tuple<3>("a", pi) = vec3::zeros();
        }

        // plasticity
        if (hasLogJp)
          pars("logJp", pi) = -0.04;
        pars("vms", pi) = 0; // vms

        // additional attributes
        for (auto &prop : auxAttribs) {
          if (prop.numChannels == 3)
            pars.tuple<3>(prop.name, pi) =
                inParticles->attr<vec3f>(std::string{prop.name})[pi];
          else
            pars(prop.name, pi) =
                inParticles->attr<float>(std::string{prop.name})[pi];
        }
      });

      pars = pars.clone({memsrc_e::um, 0});
    }
    if (bindMesh) {
      outParticles->elements =
          typename ZenoParticles::particles_t{eleTags, eleSize, memsrc_e::host};
      auto &eles = outParticles->getQuadraturePoints(); // tilevector
      ompExec(zs::range(eleSize),
              [eles = proxy<execspace_e::host>({}, eles), &model, velsPtr,
               nrmsPtr, &eleVol, &elePos, &eleVel, &eleD, category, &quads,
               &tris, &lines](size_t ei) mutable {
                using vec3 = zs::vec<float, 3>;
                using mat3 = zs::vec<float, 3, 3>;
                // vol, mass
                eles("vol", ei) = eleVol[ei];
                eles("mass", ei) = eleVol[ei] * model->density;

                // pos
                eles.tuple<3>("pos", ei) = elePos[ei];

                // vel
                if (velsPtr != nullptr)
                  eles.tuple<3>("vel", ei) = eleVel[ei];
                else
                  eles.tuple<3>("vel", ei) = vec3::zeros();

                // deformation
                const auto &D = eleD[ei]; // [col]
                auto Dmat = mat3{D[0][0], D[1][0], D[2][0], D[0][1], D[1][1],
                                 D[2][1], D[0][2], D[1][2], D[2][2]};
                // could qr decomp here first (tech doc)
                eles.tuple<9>("d", ei) = Dmat;
                eles.tuple<9>("Dinv", ei) = zs::inverse(Dmat);
                eles.tuple<9>("F", ei) = mat3::identity();

                // apic transfer
                eles.tuple<9>("C", ei) = mat3::zeros();

                // plasticity

                // element-vertex indices
                if (category == ZenoParticles::tet) {
                  const auto &quad = quads[ei];
                  for (int i = 0; i != 4; ++i) {
                    eles("inds", i, ei) = quad[i];
                  }
                } else if (category == ZenoParticles::surface) {
                  const auto &tri = tris[ei];
                  for (int i = 0; i != 3; ++i) {
                    eles("inds", i, ei) = tri[i];
                  }
                } else if (category == ZenoParticles::curve) {
                  const auto &line = lines[ei];
                  for (int i = 0; i != 2; ++i) {
                    eles("inds", i, ei) = line[i];
                  }
                }
              });
      eles = eles.clone({memsrc_e::um, 0});
    }

    fmt::print(fg(fmt::color::cyan), "done executing ToZensimParticles\n");
    set_output("ZSParticles", outParticles);
  }
};

ZENDEFNODE(ToZSParticles, {
                              {"ZSModel", "prim", {"int", "category", "0"}},
                              {"ZSParticles"},
                              {},
                              {"MPM"},
                          });

/// this requires further polishing
struct UpdatePrimitiveFromZSParticles : INode {
  void apply() override {
    fmt::print(fg(fmt::color::green),
               "begin executing UpdatePrimitiveFromZSParticles\n");

    auto parObjPtrs = RETRIEVE_OBJECT_PTRS(ZenoParticles, "ZSParticles");

    using namespace zs;
    auto ompExec = zs::omp_exec();

    for (auto &&parObjPtr : parObjPtrs) {
      auto &pars = parObjPtr->getParticles();
      if (parObjPtr->prim.get() == nullptr)
        continue;

      // const auto category = parObjPtr->category;
      auto &pos = parObjPtr->prim->attr<vec3f>("pos");
      auto size = pos.size(); // in case zsparticle-mesh is refined
      vec3f *velsPtr{nullptr};
      if (parObjPtr->prim->has_attr("vel"))
        velsPtr = parObjPtr->prim->attr<vec3f>("vel").data();

      // currently only write back pos and vel (if has)
      ompExec(range(size),
              [&, pars = proxy<execspace_e::host>({}, pars)](std::size_t pi) {
                pos[pi] = pars.array<3>("pos", pi);
                if (velsPtr != nullptr)
                  velsPtr[pi] = pars.array<3>("vel", pi);
              });
    }

    fmt::print(fg(fmt::color::cyan),
               "done executing UpdatePrimitiveFromZSParticles\n");
    set_output("ZSParticles", get_input("ZSParticles"));
  }
};

ZENDEFNODE(UpdatePrimitiveFromZSParticles, {
                                               {"ZSParticles"},
                                               {"ZSParticles"},
                                               {},
                                               {"MPM"},
                                           });

struct MakeZSPartition : INode {
  void apply() override {
    auto partition = std::make_shared<ZenoPartition>();
    partition->get() =
        typename ZenoPartition::table_t{(std::size_t)1, zs::memsrc_e::um, 0};
    set_output("ZSPartition", partition);
  }
};
ZENDEFNODE(MakeZSPartition, {
                                {},
                                {"ZSPartition"},
                                {},
                                {"MPM"},
                            });

struct MakeZSGrid : INode {
  void apply() override {
    auto dx = get_input2<float>("dx");

    std::vector<zs::PropertyTag> tags{{"m", 1}, {"v", 3}};

    auto grid = std::make_shared<ZenoGrid>();
    grid->transferScheme = get_input2<std::string>("transfer");
    // default is "apic"
    if (grid->transferScheme == "flip")
      tags.emplace_back(zs::PropertyTag{"vdiff", 3});
    else if (grid->transferScheme == "apic")
      ;
    else
      throw std::runtime_error(fmt::format(
          "unrecognized transfer scheme [{}]\n", grid->transferScheme));

    grid->get() = typename ZenoGrid::grid_t{tags, dx, 1, zs::memsrc_e::um, 0};

    using traits = zs::grid_traits<typename ZenoGrid::grid_t>;
    fmt::print("grid of dx [{}], side_length [{}], block_size [{}]\n",
               grid->get().dx, traits::side_length, traits::block_size);
    set_output("ZSGrid", grid);
  }
};
ZENDEFNODE(MakeZSGrid,
           {
               {{"float", "dx", "0.1"}, {"string", "transfer", "apic"}},
               {"ZSGrid"},
               {},
               {"MPM"},
           });

struct MakeZSLevelSet : INode {
  void apply() override {
    auto dx = get_input2<float>("dx");

    std::vector<zs::PropertyTag> tags{{"sdf", 1}};

    auto ls = std::make_shared<ZenoLevelSet>();
    ls->transferScheme = get_input2<std::string>("transfer");
    auto cateStr = get_input2<std::string>("category");

    // default is "cellcentered"
    if (cateStr == "staggered")
      tags.emplace_back(zs::PropertyTag{"vel", 3});
    // default is "flip"
    if (ls->transferScheme == "flip")
      tags.emplace_back(zs::PropertyTag{"vdiff", 3});
    else if (ls->transferScheme == "apic")
      ;
    else
      throw std::runtime_error(fmt::format(
          "unrecognized transfer scheme [{}]\n", ls->transferScheme));

    if (cateStr == "collocated")
      ls->getLevelSet() =
          typename ZenoLevelSet::template spls_t<zs::grid_e::collocated>{
              tags, dx, 1, zs::memsrc_e::um, 0};
    else if (cateStr == "cellcentered")
      ls->getLevelSet() =
          typename ZenoLevelSet::template spls_t<zs::grid_e::cellcentered>{
              tags, dx, 1, zs::memsrc_e::um, 0};
    else if (cateStr == "staggered")
      ls->getLevelSet() =
          typename ZenoLevelSet::template spls_t<zs::grid_e::staggered>{
              tags, dx, 1, zs::memsrc_e::um, 0};
    else
      throw std::runtime_error(
          fmt::format("unknown levelset (grid) category [{}].", cateStr));

    zs::match([](const auto &lsPtr) {
      if constexpr (zs::is_spls_v<typename RM_CVREF_T(lsPtr)::element_type>) {
        using spls_t = typename RM_CVREF_T(lsPtr)::element_type;
        fmt::print(
            "levelset [{}] of dx [{}, {}], side_length [{}], block_size [{}]\n",
            spls_t::category, lsPtr->_i2wShat(0, 0), lsPtr->_grid.dx,
            spls_t::side_length, spls_t::block_size);
      } else {
        throw std::runtime_error(
            fmt::format("invalid levelset [{}] initialized in basicls.",
                        zs::get_var_type_str(lsPtr)));
      }
    })(ls->getBasicLevelSet()._ls);
    set_output("ZSLevelSet", std::move(ls));
  }
};
ZENDEFNODE(MakeZSLevelSet, {
                               {{"float", "dx", "0.1"},
                                {"string", "transfer", "flip"},
                                {"string", "category", "cellcentered"}},
                               {"ZSLevelSet"},
                               {},
                               {"SOP"},
                           });

struct ToZSBoundary : INode {
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing ToZSBoundary\n");
    auto boundary = std::make_shared<ZenoBoundary>();

    auto type = get_param<std::string>("type");
    auto queryType = [&type]() -> zs::collider_e {
      if (type == "sticky" || type == "Sticky")
        return zs::collider_e::Sticky;
      else if (type == "slip" || type == "Slip")
        return zs::collider_e::Slip;
      else if (type == "separate" || type == "Separate")
        return zs::collider_e::Separate;
      return zs::collider_e::Sticky;
    };

    boundary->zsls = get_input<ZenoLevelSet>("ZSLevelSet");

    boundary->type = queryType();

    // translation
    if (has_input("translation")) {
      auto b = get_input<NumericObject>("translation")->get<vec3f>();
      boundary->b = zs::vec<float, 3>{b[0], b[1], b[2]};
    }
    if (has_input("translation_rate")) {
      auto dbdt = get_input<NumericObject>("translation_rate")->get<vec3f>();
      boundary->dbdt = zs::vec<float, 3>{dbdt[0], dbdt[1], dbdt[2]};
      // fmt::print("dbdt assigned as {}, {}, {}\n", boundary->dbdt[0],
      //            boundary->dbdt[1], boundary->dbdt[2]);
    }
    // scale
    if (has_input("scale")) {
      auto s = get_input<NumericObject>("scale")->get<float>();
      boundary->s = s;
    }
    if (has_input("scale_rate")) {
      auto dsdt = get_input<NumericObject>("scale_rate")->get<float>();
      boundary->dsdt = dsdt;
    }
    // rotation
    if (has_input("ypr_angles")) {
      auto yprAngles = get_input<NumericObject>("ypr_angles")->get<vec3f>();
      auto rot = zs::Rotation<float, 3>{yprAngles[0], yprAngles[1],
                                        yprAngles[2], zs::degree_c, zs::ypr_c};
      boundary->R = rot;
    }
    { boundary->omega = zs::AngularVelocity<float, 3>{}; }

    fmt::print(fg(fmt::color::cyan), "done executing ToZSBoundary\n");
    set_output("ZSBoundary", boundary);
  }
};
ZENDEFNODE(ToZSBoundary, {
                             {"ZSLevelSet", "translation", "translation_rate",
                              "scale", "scale_rate", "ypr_angles"},
                             {"ZSBoundary"},
                             {{"string", "type", "sticky"}},
                             {"MPM"},
                         });

struct StepZSBoundary : INode {
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing StepZSBoundary\n");

    auto boundary = get_input<ZenoBoundary>("ZSBoundary");
    auto dt = get_input2<float>("dt");

    // auto oldB = boundary->b;

    boundary->s += boundary->dsdt * dt;
    boundary->b += boundary->dbdt * dt;

#if 0
    auto b = boundary->b;
    auto dbdt = boundary->dbdt;
    auto delta = dbdt * dt;
    fmt::print("({}, {}, {}) + ({}, {}, {}) * {} -> ({}, {}, {})\n", oldB[0],
               oldB[1], oldB[2], dbdt[0], dbdt[1], dbdt[2], dt, delta[0],
               delta[1], delta[2]);
#endif

    fmt::print(fg(fmt::color::cyan), "done executing StepZSBoundary\n");
    set_output("ZSBoundary", boundary);
  }
};
ZENDEFNODE(StepZSBoundary, {
                               {"ZSBoundary", {"float", "dt", "0"}},
                               {"ZSBoundary"},
                               {},
                               {"MPM"},
                           });

/// conversion

struct ZSParticlesToPrimitiveObject : INode {
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing "
                                      "ZSParticlesToPrimitiveObject\n");
    auto &zspars = get_input<ZenoParticles>("ZSParticles")->getParticles();
    const auto size = zspars.size();

    auto prim = std::make_shared<PrimitiveObject>();
    prim->resize(size);

    using namespace zs;
    auto cudaExec = cuda_exec().device(0);

    static_assert(sizeof(zs::vec<float, 3>) == sizeof(zeno::vec3f),
                  "zeno::vec3f != zs::vec<float, 3>");
    for (auto &&prop : zspars.getPropertyTags()) {
      if (prop.numChannels == 3) {
        zs::Vector<zs::vec<float, 3>> dst{size, memsrc_e::device, 0};
        cudaExec(zs::range(size),
                 [zspars = zs::proxy<execspace_e::cuda>({}, zspars),
                  dst = zs::proxy<execspace_e::cuda>(dst),
                  name = prop.name] __device__(size_t pi) mutable {
                   dst[pi] = zspars.pack<3>(name, pi);
                 });
        copy(zs::mem_device,
             prim->add_attr<zeno::vec3f>(prop.name.asString()).data(),
             dst.data(), sizeof(zeno::vec3f) * size);
      } else if (prop.numChannels == 1) {
        zs::Vector<float> dst{size, memsrc_e::device, 0};
        cudaExec(zs::range(size),
                 [zspars = zs::proxy<execspace_e::cuda>({}, zspars),
                  dst = zs::proxy<execspace_e::cuda>(dst),
                  name = prop.name] __device__(size_t pi) mutable {
                   dst[pi] = zspars(name, pi);
                 });
        copy(zs::mem_device, prim->add_attr<float>(prop.name.asString()).data(),
             dst.data(), sizeof(float) * size);
      }
    }
    fmt::print(fg(fmt::color::cyan), "done executing "
                                     "ZSParticlesToPrimitiveObject\n");
    set_output("prim", prim);
  }
};

ZENDEFNODE(ZSParticlesToPrimitiveObject, {
                                             {"ZSParticles"},
                                             {"prim"},
                                             {},
                                             {"MPM"},
                                         });

struct WriteZSParticles : zeno::INode {
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing WriteZSParticles\n");
    auto &pars = get_input<ZenoParticles>("ZSParticles")->getParticles();
    auto path = get_param<std::string>("path");
    auto cudaExec = zs::cuda_exec().device(0);
    zs::Vector<zs::vec<float, 3>> pos{pars.size(), zs::memsrc_e::um, 0};
    zs::Vector<float> vms{pars.size(), zs::memsrc_e::um, 0};
    cudaExec(zs::range(pars.size()),
             [pos = zs::proxy<zs::execspace_e::cuda>(pos),
              vms = zs::proxy<zs::execspace_e::cuda>(vms),
              pars = zs::proxy<zs::execspace_e::cuda>(
                  {}, pars)] __device__(size_t pi) mutable {
               pos[pi] = pars.pack<3>("pos", pi);
               vms[pi] = pars("vms", pi);
             });
    std::vector<std::array<float, 3>> posOut(pars.size());
    std::vector<float> vmsOut(pars.size());
    copy(zs::mem_device, posOut.data(), pos.data(),
         sizeof(zeno::vec3f) * pars.size());
    copy(zs::mem_device, vmsOut.data(), vms.data(),
         sizeof(float) * pars.size());

    zs::write_partio_with_stress<float, 3>(path, posOut, vmsOut);
    fmt::print(fg(fmt::color::cyan), "done executing WriteZSParticles\n");
  }
};

ZENDEFNODE(WriteZSParticles, {
                                 {"ZSParticles"},
                                 {},
                                 {{"string", "path", ""}},
                                 {"MPM"},
                             });

struct ComputeVonMises : INode {
  template <typename Model>
  void computeVms(zs::CudaExecutionPolicy &cudaPol, const Model &model,
                  typename ZenoParticles::particles_t &pars, int option) {
    using namespace zs;
    cudaPol(range(pars.size()), [pars = proxy<execspace_e::cuda>({}, pars),
                                 model, option] __device__(size_t pi) mutable {
      auto F = pars.pack<3, 3>("F", pi);
      auto [U, S, V] = math::svd(F);
      auto cauchy = model.dpsi_dsigma(S) * S / S.prod();

      auto diff = cauchy;
      for (int d = 0; d != 3; ++d)
        diff(d) -= cauchy((d + 1) % 3);

      auto vms = ::sqrt(diff.l2NormSqr() * 0.5f);
      pars("vms", pi) = option ? ::log10(vms + 1) : vms;
    });
  }
  void apply() override {
    fmt::print(fg(fmt::color::green), "begin executing ComputeVonMises\n");
    auto zspars = get_input<ZenoParticles>("ZSParticles");
    auto &pars = zspars->getParticles();
    auto model = zspars->getModel();
    auto option = get_param<int>("by_log1p(base10)");

    auto cudaExec = zs::cuda_exec().device(0);
    zs::match([&](auto &elasticModel) {
      computeVms(cudaExec, elasticModel, pars, option);
    })(model.getElasticModel());

    set_output("ZSParticles", std::move(zspars));
    fmt::print(fg(fmt::color::cyan), "done executing ComputeVonMises\n");
  }
};

ZENDEFNODE(ComputeVonMises, {
                                {"ZSParticles"},
                                {"ZSParticles"},
                                {{"int", "by_log1p(base10)", "1"}},
                                {"MPM"},
                            });

} // namespace zeno