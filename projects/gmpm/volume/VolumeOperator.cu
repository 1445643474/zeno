#include "../Utils.hpp"
#include "../mpm/Structures.hpp"

#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/io/ParticleIO.hpp"
#include "zensim/math/matrix/QRSVD.hpp"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/simulation/Utils.hpp"
#include "zensim/tpls/fmt/color.h"
#include "zensim/tpls/fmt/format.h"
#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>

namespace zeno {

struct ZSPartitionForZSParticles : INode {};

} // namespace zeno